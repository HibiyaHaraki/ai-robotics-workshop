#include "MWConvLayerImpl.hpp"
#include "MWCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWTargetNetworkImpl.hpp"
#include <cassert>
#include <cstdio>
 MWConvLayerImpl::MWConvLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int filt_H, int filt_W, int numGrps, int numChnls, int numFilts, int 
FshVHIJMRAhtQirYPlZd, int GDRXdUDklKFEYEfifhIH, int 
ClEhcJFlvGCgiavziIag, int CZNYmBcNFSZWvaCklqeM, int 
CufLFODQDXTAPyRqYodN, int DCdZnqpcBnvXVgEsLBnz, int 
AuqaQHxmPQSyYRemQvyX, int AzTsxYcYjIEJsGQbeYHm, const 
char* xHViLEwTujGGrPZZgmbF, const char* KCudOrFMfgCzUPMcdePX) : 
MWCNNLayerImpl(layer, ntwk_impl) , vpXxoeEhdEosLSsYXkNG(NULL) , JCaothdxtQBQxaaitLio(NULL) , 
TfsmDFpPPOscKZifVzSQ(NULL) , BHuHNDGoRwGRouCxeMbw(filt_H) , BLjrjqvCcCommiXWQLjs 
(filt_W) , BuyZFXzwOMxcePIbCLfl (numGrps) , BkwhtPQUCQKchmmimoXs (numChnls) , 
BlRIQPyqJZORKENzSdYf (numFilts) , 
AdmgfUbRAfzFeYHxSnQr(AuqaQHxmPQSyYRemQvyX) , 
AwZQzUhuWVLGrWgLHRuM(AzTsxYcYjIEJsGQbeYHm) , 
CTCbzQMDaLxINPbODdng(ClEhcJFlvGCgiavziIag) , 
CLOUhPjbgggWoXHTtmjC(CZNYmBcNFSZWvaCklqeM) , 
CpMjJjtGOeWOzwxpAAQP(CufLFODQDXTAPyRqYodN) , 
CqtPRJvHlGJFssiPzsOm(DCdZnqpcBnvXVgEsLBnz) , 
FrpxvsDMwwgbpqHXWxmN(FshVHIJMRAhtQirYPlZd) , 
FwLnexHgxHRquTKmNpoa(GDRXdUDklKFEYEfifhIH) , 
IbSWJNMuIiKbocfQKqXb((CTCbzQMDaLxINPbODdng != CLOUhPjbgggWoXHTtmjC) 
|| (CpMjJjtGOeWOzwxpAAQP != CqtPRJvHlGJFssiPzsOm)) { dMxIKDGTITyhdLqIHBLA = 
ntwk_impl; CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&NZjOkZPwLzQsdEVkwMcX)); 
CUDNN_CALL(hipdnnCreateFilterDescriptor(&QhTesEEIHwhNmHSeYbRR)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&JwxFdqOKggeawILBfGgg));  MWCNNLayer* 
convLayer = getLayer(); MWTensorBase* ipTensor = convLayer->getInputTensor(0); 
if (IbSWJNMuIiKbocfQKqXb) { TfsmDFpPPOscKZifVzSQ = new MWTensor<float>(-1, 
-1, -1, -1, -1, NULL, getLayer(), 0); if (!TfsmDFpPPOscKZifVzSQ) { 
MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&YCSvyQZBWMDYQXHtyVai)); } else { 
TfsmDFpPPOscKZifVzSQ = ipTensor; } assert(TfsmDFpPPOscKZifVzSQ != NULL); int 
NbunkIVaMPVYgAQHXXYd; int NldNILHvuQqQPSAHXxdT; if 
(IbSWJNMuIiKbocfQKqXb) { NbunkIVaMPVYgAQHXXYd = 0;  
NldNILHvuQqQPSAHXxdT = 0; } else { NbunkIVaMPVYgAQHXXYd = 
CTCbzQMDaLxINPbODdng; NldNILHvuQqQPSAHXxdT = CpMjJjtGOeWOzwxpAAQP; } 
bYBVtTnVUuGDUlaTmmHp = CTCbzQMDaLxINPbODdng; cQBKlCKXxecGPJrXBXdk = CpMjJjtGOeWOzwxpAAQP;
#if (CUDNN_MAJOR <= 5)
 { if ((AdmgfUbRAfzFeYHxSnQr != 1) && (AwZQzUhuWVLGrWgLHRuM != 1)){ 
printf("Dilated Convolution only supported for cuDNN 6 or greater "); throw 
std::runtime_error("Unsupported Dilation Factor"); } 
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(NZjOkZPwLzQsdEVkwMcX, 
NbunkIVaMPVYgAQHXXYd, NldNILHvuQqQPSAHXxdT, FrpxvsDMwwgbpqHXWxmN, 
FwLnexHgxHRquTKmNpoa, 1, 1, HIPDNN_CROSS_CORRELATION));  }
#else
 { CUDNN_CALL(hipdnnSetConvolution2dDescriptor(NZjOkZPwLzQsdEVkwMcX, 
NbunkIVaMPVYgAQHXXYd, NldNILHvuQqQPSAHXxdT, FrpxvsDMwwgbpqHXWxmN, 
FwLnexHgxHRquTKmNpoa, AdmgfUbRAfzFeYHxSnQr, AwZQzUhuWVLGrWgLHRuM, 
HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT)); }
#endif
#if (FP16_ENABLED == 1 && ( CUDNN_MAJOR > 7 || (CUDNN_MAJOR == 7 && CUDNN_MINOR >= 2) ))
 CUDNN_CALL(hipdnnSetConvolutionMathType(NZjOkZPwLzQsdEVkwMcX, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
#endif
 if (BuyZFXzwOMxcePIbCLfl > 1){ 
CUDNN_CALL(hipdnnSetConvolutionGroupCount(NZjOkZPwLzQsdEVkwMcX, 
BuyZFXzwOMxcePIbCLfl)); } int euppfEoiaoCTcVgRPVhA = 
BkwhtPQUCQKchmmimoXs*BuyZFXzwOMxcePIbCLfl; int fSbUUBgjKRbNXrHrlOLo = 
BlRIQPyqJZORKENzSdYf*BuyZFXzwOMxcePIbCLfl; 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(QhTesEEIHwhNmHSeYbRR, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, fSbUUBgjKRbNXrHrlOLo, 
euppfEoiaoCTcVgRPVhA/BuyZFXzwOMxcePIbCLfl, BHuHNDGoRwGRouCxeMbw, 
BLjrjqvCcCommiXWQLjs)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JwxFdqOKggeawILBfGgg, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, fSbUUBgjKRbNXrHrlOLo, 1, 1)); int weightSize = 
BkwhtPQUCQKchmmimoXs*fSbUUBgjKRbNXrHrlOLo*BHuHNDGoRwGRouCxeMbw*BLjrjqvCcCommiXWQLjs; 
CUDA_CALL(hipMalloc((void**)&vpXxoeEhdEosLSsYXkNG, sizeof(float)*weightSize)); 
CUDA_CALL(hipMalloc((void**)&JCaothdxtQBQxaaitLio, 
sizeof(float)*fSbUUBgjKRbNXrHrlOLo)); 
loadWeights(xHViLEwTujGGrPZZgmbF); loadBias(KCudOrFMfgCzUPMcdePX); 
createAndAddDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); } 
void MWConvLayerImpl::propagateSize() { MWTensorBase* ipTensor = 
getLayer()->getInputTensor(0); int inputH; int inputW; if 
(IbSWJNMuIiKbocfQKqXb) { inputH = ipTensor->getHeight() + 
CTCbzQMDaLxINPbODdng + CLOUhPjbgggWoXHTtmjC; inputW = ipTensor->getWidth() + 
CpMjJjtGOeWOzwxpAAQP + CqtPRJvHlGJFssiPzsOm; } else { inputH = 
ipTensor->getHeight(); inputW = ipTensor->getWidth(); } 
TfsmDFpPPOscKZifVzSQ->setHeight(inputH); TfsmDFpPPOscKZifVzSQ->setWidth(inputW); 
TfsmDFpPPOscKZifVzSQ->setChannels(ipTensor->getChannels()); 
TfsmDFpPPOscKZifVzSQ->setBatchSize(ipTensor->getBatchSize()); 
TfsmDFpPPOscKZifVzSQ->setSequenceLength(ipTensor->getSequenceLength()); 
assert(TfsmDFpPPOscKZifVzSQ->getSequenceLength() == 1); if 
(IbSWJNMuIiKbocfQKqXb) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(YCSvyQZBWMDYQXHtyVai, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, TfsmDFpPPOscKZifVzSQ->getBatchSize(), TfsmDFpPPOscKZifVzSQ->getChannels(), 
TfsmDFpPPOscKZifVzSQ->getHeight(), TfsmDFpPPOscKZifVzSQ->getWidth())); } else { 
YCSvyQZBWMDYQXHtyVai = MWCNNLayerImpl::getCuDNNDescriptor(TfsmDFpPPOscKZifVzSQ); } 
MWTensorBase* opTensor = getLayer()->getOutputTensor(0); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor));
#if (CUDNN_MAJOR < 7)
 { 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
YCSvyQZBWMDYQXHtyVai, QhTesEEIHwhNmHSeYbRR, NZjOkZPwLzQsdEVkwMcX, *desc, 
HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &NNhshzQGJHLSGjDiVerE)); }
#else
 { const int maxAlgoCount(3); int returnedAlgoCount(-1);  
hipdnnConvolutionFwdAlgoPerf_t perf_results[maxAlgoCount]; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
YCSvyQZBWMDYQXHtyVai, QhTesEEIHwhNmHSeYbRR, NZjOkZPwLzQsdEVkwMcX, *desc, 
maxAlgoCount, &returnedAlgoCount, perf_results)); NNhshzQGJHLSGjDiVerE = 
perf_results[0].algo; }
#endif
 size_t tnTPxeDjBsqLAPkJcPJX = 0; 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize( 
*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), YCSvyQZBWMDYQXHtyVai, 
QhTesEEIHwhNmHSeYbRR, NZjOkZPwLzQsdEVkwMcX, *desc, NNhshzQGJHLSGjDiVerE, 
&tnTPxeDjBsqLAPkJcPJX)); if (tnTPxeDjBsqLAPkJcPJX > 
*dMxIKDGTITyhdLqIHBLA->getProposedWorkSpaceSize()) { 
dMxIKDGTITyhdLqIHBLA->setProposedWorkSpaceSize(tnTPxeDjBsqLAPkJcPJX); } } void 
MWConvLayerImpl::allocate() { MWTensorBase* ipTensor = 
getLayer()->getInputTensor(0); if (IbSWJNMuIiKbocfQKqXb) { float* 
newInput; int inputH = ipTensor->getHeight() + CTCbzQMDaLxINPbODdng + 
CLOUhPjbgggWoXHTtmjC; int inputW = ipTensor->getWidth() + 
CpMjJjtGOeWOzwxpAAQP + CqtPRJvHlGJFssiPzsOm; int paddedSize = 
ipTensor->getBatchSize() * ipTensor->getChannels() * inputH * inputW; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float)*paddedSize)); 
CUDA_CALL(hipMemset(newInput, 0, sizeof(float)*paddedSize)); 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->setData(newInput); } } void 
MWConvLayerImpl::deallocate() { if (TfsmDFpPPOscKZifVzSQ != 
getLayer()->getInputTensor(0)) { assert(IbSWJNMuIiKbocfQKqXb); 
CUDA_FREE_CALL(static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData()); 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->setData((float*)NULL); } } void 
MWConvLayerImpl::predict() { MWCNNLayer* convLayer = getLayer(); MWTensorBase* 
ipTensorBase = convLayer->getInputTensor(); MWTensorBase* opTensorBase = 
convLayer->getOutputTensor(); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); if (TfsmDFpPPOscKZifVzSQ != 
convLayer->getInputTensor()) { 
CUDA_CALL(hipMemset(static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), 
0, sizeof(float)*TfsmDFpPPOscKZifVzSQ->getNumElements()));  
MWCNNLayerImpl::padInput(ipTensor->getData(), ipTensor->getHeight(), 
ipTensor->getWidth(), ipTensor->getChannels(), TfsmDFpPPOscKZifVzSQ->getHeight(), 
TfsmDFpPPOscKZifVzSQ->getWidth(), bYBVtTnVUuGDUlaTmmHp, cQBKlCKXxecGPJrXBXdk, 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), 
ipTensor->getNumElements()); } assert(opTensor->getData() != 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData()); 
hipdnnTensorDescriptor_t* desc = getDescriptor(opTensor->getSourcePortIndex()); 
assert(desc); 
CUDNN_CALL(hipdnnConvolutionForward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
getOnePtr(), YCSvyQZBWMDYQXHtyVai, 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), QhTesEEIHwhNmHSeYbRR, 
vpXxoeEhdEosLSsYXkNG, NZjOkZPwLzQsdEVkwMcX, NNhshzQGJHLSGjDiVerE, 
dMxIKDGTITyhdLqIHBLA->getWorkSpace(), 
*dMxIKDGTITyhdLqIHBLA->getAllocatedWorkSpaceSize(), getZeroPtr(), *desc, 
opTensor->getData())); 
CUDNN_CALL(hipdnnAddTensor(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), getOnePtr(), 
JwxFdqOKggeawILBfGgg, JCaothdxtQBQxaaitLio, getOnePtr(), *desc, opTensor->getData())); } 
void MWConvLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(NZjOkZPwLzQsdEVkwMcX)); 
CUDNN_CALL(hipdnnDestroyFilterDescriptor(QhTesEEIHwhNmHSeYbRR)); if 
(vpXxoeEhdEosLSsYXkNG) { CUDA_FREE_CALL(vpXxoeEhdEosLSsYXkNG); vpXxoeEhdEosLSsYXkNG = NULL; } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(JwxFdqOKggeawILBfGgg)); if 
(JCaothdxtQBQxaaitLio) { CUDA_FREE_CALL(JCaothdxtQBQxaaitLio); JCaothdxtQBQxaaitLio = NULL; } if 
(TfsmDFpPPOscKZifVzSQ != getLayer()->getInputTensor(0)) { 
assert(IbSWJNMuIiKbocfQKqXb); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(YCSvyQZBWMDYQXHtyVai)); } } void 
MWConvLayerImpl::loadWeights(const char* QVgVGfoCXYiYXzPhvVPX) {  FILE* 
QwUuNuQNtlPXrIwRNiSZ = MWCNNLayer::openBinaryFile(QVgVGfoCXYiYXzPhvVPX); 
assert(QwUuNuQNtlPXrIwRNiSZ); int dkLDkRwCBjeybwDHbKiE = 
BkwhtPQUCQKchmmimoXs*BlRIQPyqJZORKENzSdYf*BuyZFXzwOMxcePIbCLfl*BHuHNDGoRwGRouCxeMbw*BLjrjqvCcCommiXWQLjs; 
 float* LgxABSJPBXdCozJkFqTg = MALLOC_CALL(sizeof(float)*dkLDkRwCBjeybwDHbKiE); 
MWCNNLayer::call_fread(LgxABSJPBXdCozJkFqTg, sizeof(float), dkLDkRwCBjeybwDHbKiE, 
QwUuNuQNtlPXrIwRNiSZ, QVgVGfoCXYiYXzPhvVPX); CUDA_CALL(hipMemcpy(vpXxoeEhdEosLSsYXkNG, 
LgxABSJPBXdCozJkFqTg, sizeof(float)*dkLDkRwCBjeybwDHbKiE, hipMemcpyHostToDevice)); 
fclose(QwUuNuQNtlPXrIwRNiSZ); free(LgxABSJPBXdCozJkFqTg); } void 
MWConvLayerImpl::loadBias(const char* QVgVGfoCXYiYXzPhvVPX) { FILE* 
QwUuNuQNtlPXrIwRNiSZ = MWCNNLayer::openBinaryFile(QVgVGfoCXYiYXzPhvVPX);  
assert(QwUuNuQNtlPXrIwRNiSZ); int dkLDkRwCBjeybwDHbKiE = 
BlRIQPyqJZORKENzSdYf*BuyZFXzwOMxcePIbCLfl;  float* LgxABSJPBXdCozJkFqTg = 
MALLOC_CALL(sizeof(float)*dkLDkRwCBjeybwDHbKiE); 
MWCNNLayer::call_fread(LgxABSJPBXdCozJkFqTg, sizeof(float), dkLDkRwCBjeybwDHbKiE, 
QwUuNuQNtlPXrIwRNiSZ, QVgVGfoCXYiYXzPhvVPX); CUDA_CALL(hipMemcpy(JCaothdxtQBQxaaitLio, 
LgxABSJPBXdCozJkFqTg, sizeof(float)*dkLDkRwCBjeybwDHbKiE, hipMemcpyHostToDevice)); 
free(LgxABSJPBXdCozJkFqTg); fclose(QwUuNuQNtlPXrIwRNiSZ); } void 
MWConvLayerImpl::postSetup() { if (dMxIKDGTITyhdLqIHBLA->getAutoTune()) { 
getConvAlgoTuned(); } else { getConvAlgoWorkSpaceLimit(); } } void 
MWConvLayerImpl::getConvAlgoTuned() { MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnConvolutionFwdAlgoPerf_t 
perf_results[3]; hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); 
assert(desc); int returnedAlgoCount; 
CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithmEx(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
YCSvyQZBWMDYQXHtyVai, static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), 
QhTesEEIHwhNmHSeYbRR, vpXxoeEhdEosLSsYXkNG, NZjOkZPwLzQsdEVkwMcX, *desc, 
opTensor->getData(), 3, &returnedAlgoCount, &perf_results[0], 
dMxIKDGTITyhdLqIHBLA->getWorkSpace(), 
*dMxIKDGTITyhdLqIHBLA->getAllocatedWorkSpaceSize())); NNhshzQGJHLSGjDiVerE = 
perf_results[0].algo; } void MWConvLayerImpl::getConvAlgoWorkSpaceLimit() { 
hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); assert(desc);
#if (CUDNN_MAJOR < 8)
 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
YCSvyQZBWMDYQXHtyVai, QhTesEEIHwhNmHSeYbRR, NZjOkZPwLzQsdEVkwMcX, *desc, 
HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT, 
*dMxIKDGTITyhdLqIHBLA->getAllocatedWorkSpaceSize(), &NNhshzQGJHLSGjDiVerE));
#else
 int maxAlgoCount(-1); 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithmMaxCount(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
&maxAlgoCount)); int returnedAlgoCount(-1); 
std::vector<hipdnnConvolutionFwdAlgoPerf_t> perf_results(maxAlgoCount);  
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
YCSvyQZBWMDYQXHtyVai, QhTesEEIHwhNmHSeYbRR, NZjOkZPwLzQsdEVkwMcX, *desc, 
maxAlgoCount, &returnedAlgoCount, &perf_results[0])); 
hipdnnConvolutionFwdAlgoPerf_t nextFastest; bool algoFound(false); for (int i = 
0; i < returnedAlgoCount; ++i) { nextFastest = perf_results[i]; if 
(nextFastest.memory <= *dMxIKDGTITyhdLqIHBLA->getAllocatedWorkSpaceSize()) { 
NNhshzQGJHLSGjDiVerE = nextFastest.algo; algoFound = true; break; } } assert(algoFound);
#endif
 }