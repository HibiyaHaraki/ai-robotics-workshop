//
// File: line_detection_initialize.cu
//
// GPU Coder version                    : 2.1
// CUDA/C/C++ source code generated on  : 08-Apr-2021 15:32:32
//

// Include Files
#include "line_detection_initialize.h"
#include "line_detection.h"
#include "line_detection_data.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void line_detection_initialize()
{
  line_detection_init();
  hipGetLastError();
  isInitialized_line_detection = true;
}

//
// File trailer for line_detection_initialize.cu
//
// [EOF]
//
