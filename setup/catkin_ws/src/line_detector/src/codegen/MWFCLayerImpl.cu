#include "MWFCLayerImpl.hpp"
#include "MWCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWTargetNetworkImpl.hpp"
 MWFCLayerImpl::MWFCLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int XVcMnvCXvZpKICKIjgZi, int lsqeARVLtpJTWezgnTkg, const char* 
xHViLEwTujGGrPZZgmbF,  const char* KCudOrFMfgCzUPMcdePX) : 
MWCNNLayerImpl(layer, ntwk_impl)  , 
CCKWXUFWgrbBMjwfpOBN(XVcMnvCXvZpKICKIjgZi) , 
CDJtexcMbXMWAmnNZsNf(lsqeARVLtpJTWezgnTkg) , vpXxoeEhdEosLSsYXkNG(NULL) , 
wJyXsrUCMgxdIKVIJSyx(NULL) , JCaothdxtQBQxaaitLio(NULL) , 
xHiBGayUfxIpXKkCTDNU(false) { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&JwxFdqOKggeawILBfGgg)); 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); 
CUDA_CALL(hipMalloc((void**)&vpXxoeEhdEosLSsYXkNG, 
sizeof(float)*CCKWXUFWgrbBMjwfpOBN*CDJtexcMbXMWAmnNZsNf)); 
CUDA_CALL(hipMalloc((void**)&JCaothdxtQBQxaaitLio, 
sizeof(float)*CDJtexcMbXMWAmnNZsNf)); wJyXsrUCMgxdIKVIJSyx = 
MALLOC_CALL(sizeof(float)*CCKWXUFWgrbBMjwfpOBN*CDJtexcMbXMWAmnNZsNf); 
loadWeights(xHViLEwTujGGrPZZgmbF); loadBias(KCudOrFMfgCzUPMcdePX); } 
MWFCLayerImpl::~MWFCLayerImpl() { } void MWFCLayerImpl::propagateSize() { 
MWCNNLayer* fcLayer = getLayer(); MWTensorBase* opTensor = 
fcLayer->getOutputTensor(0); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor)); if 
(opTensor->getSequenceLength() == 1) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JwxFdqOKggeawILBfGgg, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, CDJtexcMbXMWAmnNZsNf, 1, 1)); } else { int dims[5] 
= {1, 1, CDJtexcMbXMWAmnNZsNf, 1, 1}; int strides[5]; 
MWTensorBase::getStrides(dims, 5, strides); 
CUDNN_CALL(hipdnnSetTensorNdDescriptor(JwxFdqOKggeawILBfGgg, HIPDNN_DATA_FLOAT, 5, 
dims, strides)); } } void MWFCLayerImpl::loadWeights(const char* 
QVgVGfoCXYiYXzPhvVPX) {  FILE* QwUuNuQNtlPXrIwRNiSZ = 
MWCNNLayer::openBinaryFile(QVgVGfoCXYiYXzPhvVPX); assert(QwUuNuQNtlPXrIwRNiSZ); int 
dkLDkRwCBjeybwDHbKiE = CCKWXUFWgrbBMjwfpOBN*CDJtexcMbXMWAmnNZsNf;  
MWCNNLayer::call_fread(wJyXsrUCMgxdIKVIJSyx, sizeof(float), dkLDkRwCBjeybwDHbKiE, 
QwUuNuQNtlPXrIwRNiSZ, QVgVGfoCXYiYXzPhvVPX); fclose(QwUuNuQNtlPXrIwRNiSZ); } void 
MWFCLayerImpl::prepareWeights() { if (!xHiBGayUfxIpXKkCTDNU) { int 
dkLDkRwCBjeybwDHbKiE = CCKWXUFWgrbBMjwfpOBN*CDJtexcMbXMWAmnNZsNf; 
MWCNNLayer* fcLayer = getLayer(); MWTensorBase* ipTensor = 
fcLayer->getInputTensor(0); if( ipTensor->getHeight() != 1 && 
ipTensor->getWidth() != 1 ) { float* LgxABSJPBXdCozJkFqTg = 
MALLOC_CALL(sizeof(float)*ipTensor->getHeight()*ipTensor->getWidth()); for(int 
k=0; k<dkLDkRwCBjeybwDHbKiE/ipTensor->getHeight()/ipTensor->getWidth(); k++) { 
for(int i=0; i<ipTensor->getHeight()*ipTensor->getWidth(); i++) 
LgxABSJPBXdCozJkFqTg[i]=wJyXsrUCMgxdIKVIJSyx[k*ipTensor->getHeight()*ipTensor->getWidth()+i]; 
for(int j=0; j<ipTensor->getHeight(); j++) for(int i=0; i<ipTensor->getWidth(); 
i++) 
wJyXsrUCMgxdIKVIJSyx[k*ipTensor->getHeight()*ipTensor->getWidth()+j*ipTensor->getWidth()+i]=LgxABSJPBXdCozJkFqTg[j+i*ipTensor->getHeight()]; 
} free(LgxABSJPBXdCozJkFqTg); } CUDA_CALL(hipMemcpy(vpXxoeEhdEosLSsYXkNG, 
wJyXsrUCMgxdIKVIJSyx, sizeof(float)*dkLDkRwCBjeybwDHbKiE, 
hipMemcpyHostToDevice)); free(wJyXsrUCMgxdIKVIJSyx); 
wJyXsrUCMgxdIKVIJSyx = NULL; xHiBGayUfxIpXKkCTDNU = true; } } void 
MWFCLayerImpl::loadBias(const char* QVgVGfoCXYiYXzPhvVPX) { MWCNNLayer* fcLayer 
= getLayer(); MWTensorBase* opTensor = fcLayer->getOutputTensor(0); FILE* 
QwUuNuQNtlPXrIwRNiSZ = MWCNNLayer::openBinaryFile(QVgVGfoCXYiYXzPhvVPX); 
assert(QwUuNuQNtlPXrIwRNiSZ); int dkLDkRwCBjeybwDHbKiE = CDJtexcMbXMWAmnNZsNf;  
float* LgxABSJPBXdCozJkFqTg = MALLOC_CALL(sizeof(float)*dkLDkRwCBjeybwDHbKiE); 
MWCNNLayer::call_fread(LgxABSJPBXdCozJkFqTg, sizeof(float), dkLDkRwCBjeybwDHbKiE, 
QwUuNuQNtlPXrIwRNiSZ, QVgVGfoCXYiYXzPhvVPX); CUDA_CALL(hipMemcpy(JCaothdxtQBQxaaitLio, 
LgxABSJPBXdCozJkFqTg, sizeof(float)*dkLDkRwCBjeybwDHbKiE, hipMemcpyHostToDevice)); 
free(LgxABSJPBXdCozJkFqTg); fclose(QwUuNuQNtlPXrIwRNiSZ); } void 
MWFCLayerImpl::postSetup() { prepareWeights(); } void MWFCLayerImpl::predict() 
{ MWCNNLayer* fcLayer = getLayer(); MWTensorBase* ipTensorBase = 
fcLayer->getInputTensor(0); MWTensorBase* opTensorBase = 
fcLayer->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); int numOutputRows = 
opTensor->getChannels(); int numOutputCols = 
ipTensor->getBatchSize()*ipTensor->getSequenceLength(); int innerDimension = 
ipTensor->getHeight()*ipTensor->getWidth()*ipTensor->getChannels(); int 
UVzBVEOIylFjkSgHwFMp=1; int UWAGLbDcvybdWBtshhsr=1; if(opTensor->getBatchSize() == 1 && 
opTensor->getSequenceLength() == 1) { CUDA_CALL(hipMemcpy(opTensor->getData(), 
JCaothdxtQBQxaaitLio, sizeof(float)*numOutputRows, hipMemcpyDeviceToDevice)); 
CUBLAS_CALL(hipblasSgemv(*dMxIKDGTITyhdLqIHBLA->getCublasHandle(), HIPBLAS_OP_T, 
innerDimension, numOutputRows, getOnePtr(), vpXxoeEhdEosLSsYXkNG, innerDimension, 
ipTensor->getData(), UVzBVEOIylFjkSgHwFMp, getOnePtr(), opTensor->getData(), 
UWAGLbDcvybdWBtshhsr)); } else { 
CUBLAS_CALL(hipblasSgemm(*dMxIKDGTITyhdLqIHBLA->getCublasHandle(), HIPBLAS_OP_T, 
HIPBLAS_OP_N, numOutputRows, numOutputCols, innerDimension, getOnePtr(), 
vpXxoeEhdEosLSsYXkNG, innerDimension, ipTensor->getData(), innerDimension, 
getZeroPtr(), opTensor->getData(), numOutputRows)); hipdnnTensorDescriptor_t* 
desc = getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
CUDNN_CALL(hipdnnAddTensor(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), getOnePtr(), 
JwxFdqOKggeawILBfGgg, JCaothdxtQBQxaaitLio, getOnePtr(), *desc, opTensor->getData())); } 
return; } void MWFCLayerImpl::cleanup() { if (vpXxoeEhdEosLSsYXkNG) { 
CUDA_FREE_CALL(vpXxoeEhdEosLSsYXkNG); vpXxoeEhdEosLSsYXkNG = NULL; } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(JwxFdqOKggeawILBfGgg)); if 
(JCaothdxtQBQxaaitLio) { CUDA_FREE_CALL(JCaothdxtQBQxaaitLio); JCaothdxtQBQxaaitLio = NULL; } }