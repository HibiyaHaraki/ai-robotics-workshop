#include "MWFusedConvReLULayerImpl.hpp"
#include "MWCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWTargetNetworkImpl.hpp"
#include <cassert>
#include <cstdio>
 MWFusedConvReLULayerImpl::MWFusedConvReLULayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int filt_H, int filt_W, int numGrps, int 
numChnls, int numFilts, int FshVHIJMRAhtQirYPlZd, int 
GDRXdUDklKFEYEfifhIH, int ClEhcJFlvGCgiavziIag, int 
CZNYmBcNFSZWvaCklqeM, int CufLFODQDXTAPyRqYodN, int 
DCdZnqpcBnvXVgEsLBnz, int AuqaQHxmPQSyYRemQvyX, int 
AzTsxYcYjIEJsGQbeYHm, int eybNKlJCSDUvsznWynwK, const char* 
xHViLEwTujGGrPZZgmbF, const char* KCudOrFMfgCzUPMcdePX) : 
MWCNNLayerImpl(layer, ntwk_impl) , vpXxoeEhdEosLSsYXkNG(NULL) , JCaothdxtQBQxaaitLio(NULL) , 
URgvgDXnZskIYGdtimcU(NULL) , XCnEVUzxqcNgsuUbRonz(NULL) , 
GFienSVKLlDQuZeqAdLC(NULL) , HoqiuUUuJnXGkfDodicJ(NULL) , 
BHuHNDGoRwGRouCxeMbw(filt_H) , BLjrjqvCcCommiXWQLjs (filt_W) , 
BuyZFXzwOMxcePIbCLfl (numGrps) , BkwhtPQUCQKchmmimoXs (numChnls) , 
BlRIQPyqJZORKENzSdYf (numFilts) , FrpxvsDMwwgbpqHXWxmN(FshVHIJMRAhtQirYPlZd) 
, FwLnexHgxHRquTKmNpoa(GDRXdUDklKFEYEfifhIH) , 
CTCbzQMDaLxINPbODdng(ClEhcJFlvGCgiavziIag) , 
CLOUhPjbgggWoXHTtmjC(CZNYmBcNFSZWvaCklqeM) , 
CpMjJjtGOeWOzwxpAAQP(CufLFODQDXTAPyRqYodN) , 
CqtPRJvHlGJFssiPzsOm(DCdZnqpcBnvXVgEsLBnz) , 
AdmgfUbRAfzFeYHxSnQr(AuqaQHxmPQSyYRemQvyX) , 
AwZQzUhuWVLGrWgLHRuM(AzTsxYcYjIEJsGQbeYHm) , 
fxxCPKTclxXPxrdMAkwi(eybNKlJCSDUvsznWynwK) , 
IbSWJNMuIiKbocfQKqXb((CTCbzQMDaLxINPbODdng != CLOUhPjbgggWoXHTtmjC) 
|| (CpMjJjtGOeWOzwxpAAQP != CqtPRJvHlGJFssiPzsOm)) {
#if (CUDNN_MAJOR < 6)
 throw std::runtime_error("Fused ConvReLU Layer only supported for cuDNN 6 or greater");
#else
 dMxIKDGTITyhdLqIHBLA = ntwk_impl; 
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&NZjOkZPwLzQsdEVkwMcX)); 
CUDNN_CALL(hipdnnCreateFilterDescriptor(&QhTesEEIHwhNmHSeYbRR)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&JwxFdqOKggeawILBfGgg));  
CUDNN_CALL(hipdnnCreateActivationDescriptor(&oJUVMnJggjhEdQLWzIUC)); 
MWTensorBase* ipTensor_conv = getLayer()->getInputTensor(0); int 
NbunkIVaMPVYgAQHXXYd = CTCbzQMDaLxINPbODdng; int 
NldNILHvuQqQPSAHXxdT = CpMjJjtGOeWOzwxpAAQP; if 
(IbSWJNMuIiKbocfQKqXb) { NbunkIVaMPVYgAQHXXYd = 0; 
NldNILHvuQqQPSAHXxdT = 0; URgvgDXnZskIYGdtimcU = new MWTensor<float>(-1, 
-1, -1, -1, -1, NULL, getLayer(), 0); if (!URgvgDXnZskIYGdtimcU) { 
MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&YCSvyQZBWMDYQXHtyVai)); } else { 
URgvgDXnZskIYGdtimcU = ipTensor_conv; } assert(URgvgDXnZskIYGdtimcU != NULL); 
bYBVtTnVUuGDUlaTmmHp = CTCbzQMDaLxINPbODdng; cQBKlCKXxecGPJrXBXdk = 
CpMjJjtGOeWOzwxpAAQP; 
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(NZjOkZPwLzQsdEVkwMcX, 
NbunkIVaMPVYgAQHXXYd, NldNILHvuQqQPSAHXxdT, FrpxvsDMwwgbpqHXWxmN, 
FwLnexHgxHRquTKmNpoa, AdmgfUbRAfzFeYHxSnQr, AwZQzUhuWVLGrWgLHRuM, 
HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
#if (FP16_ENABLED == 1 && ( CUDNN_MAJOR > 7 || (CUDNN_MAJOR == 7 && CUDNN_MINOR >= 2) ))
 CUDNN_CALL(hipdnnSetConvolutionMathType(NZjOkZPwLzQsdEVkwMcX, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
#endif
 if (BuyZFXzwOMxcePIbCLfl > 1){ 
CUDNN_CALL(hipdnnSetConvolutionGroupCount(NZjOkZPwLzQsdEVkwMcX, 
BuyZFXzwOMxcePIbCLfl)); } 
CUDNN_CALL(hipdnnSetActivationDescriptor(oJUVMnJggjhEdQLWzIUC, 
HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0)); int 
euppfEoiaoCTcVgRPVhA = BkwhtPQUCQKchmmimoXs*BuyZFXzwOMxcePIbCLfl; int 
fSbUUBgjKRbNXrHrlOLo = BlRIQPyqJZORKENzSdYf*BuyZFXzwOMxcePIbCLfl; 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(QhTesEEIHwhNmHSeYbRR, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, fSbUUBgjKRbNXrHrlOLo, 
euppfEoiaoCTcVgRPVhA/BuyZFXzwOMxcePIbCLfl, BHuHNDGoRwGRouCxeMbw, 
BLjrjqvCcCommiXWQLjs)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JwxFdqOKggeawILBfGgg, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, fSbUUBgjKRbNXrHrlOLo, 1, 1)); int weightSize = 
BkwhtPQUCQKchmmimoXs*fSbUUBgjKRbNXrHrlOLo*BHuHNDGoRwGRouCxeMbw*BLjrjqvCcCommiXWQLjs; 
CUDA_CALL(hipMalloc((void**)&vpXxoeEhdEosLSsYXkNG, sizeof(float)*weightSize)); 
CUDA_CALL(hipMalloc((void**)&JCaothdxtQBQxaaitLio, 
sizeof(float)*fSbUUBgjKRbNXrHrlOLo)); 
loadWeights(xHViLEwTujGGrPZZgmbF); loadBias(KCudOrFMfgCzUPMcdePX); createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex());
#endif
 } MWFusedConvReLULayerImpl::~MWFusedConvReLULayerImpl() { } void 
MWFusedConvReLULayerImpl::propagateSize() {
#if (CUDNN_MAJOR >= 6)
 MWTensorBase* ipTensor_conv = getLayer()->getInputTensor(0); int inputH; int 
inputW; if (IbSWJNMuIiKbocfQKqXb) { inputH = 
ipTensor_conv->getHeight() + CTCbzQMDaLxINPbODdng + CLOUhPjbgggWoXHTtmjC; 
inputW = ipTensor_conv->getWidth() + CpMjJjtGOeWOzwxpAAQP + 
CqtPRJvHlGJFssiPzsOm; } else { inputH = ipTensor_conv->getHeight(); inputW = 
ipTensor_conv->getWidth(); } URgvgDXnZskIYGdtimcU->setHeight(inputH); 
URgvgDXnZskIYGdtimcU->setWidth(inputW); 
URgvgDXnZskIYGdtimcU->setChannels(ipTensor_conv->getChannels()); 
URgvgDXnZskIYGdtimcU->setBatchSize(ipTensor_conv->getBatchSize()); 
URgvgDXnZskIYGdtimcU->setSequenceLength(ipTensor_conv->getSequenceLength()); 
assert(URgvgDXnZskIYGdtimcU->getSequenceLength() == 1); if 
(IbSWJNMuIiKbocfQKqXb) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(YCSvyQZBWMDYQXHtyVai, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, URgvgDXnZskIYGdtimcU->getBatchSize(), 
URgvgDXnZskIYGdtimcU->getChannels(), URgvgDXnZskIYGdtimcU->getHeight(), 
URgvgDXnZskIYGdtimcU->getWidth())); } else { YCSvyQZBWMDYQXHtyVai = 
MWCNNLayerImpl::getCuDNNDescriptor(URgvgDXnZskIYGdtimcU); } 
assert(BkwhtPQUCQKchmmimoXs == 
URgvgDXnZskIYGdtimcU->getChannels()/BuyZFXzwOMxcePIbCLfl); MWTensorBase* opTensor 
= getLayer()->getOutputTensor(0); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc); 
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensor));
#if (CUDNN_MAJOR < 7)
 { 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
YCSvyQZBWMDYQXHtyVai, QhTesEEIHwhNmHSeYbRR, NZjOkZPwLzQsdEVkwMcX, *desc, 
HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &NNhshzQGJHLSGjDiVerE)); }
#else
 { const int maxAlgoCount(3); int returnedAlgoCount(-1); 
hipdnnConvolutionFwdAlgoPerf_t perf_results[maxAlgoCount]; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
YCSvyQZBWMDYQXHtyVai, QhTesEEIHwhNmHSeYbRR, NZjOkZPwLzQsdEVkwMcX, *desc, 
maxAlgoCount, &returnedAlgoCount, perf_results)); NNhshzQGJHLSGjDiVerE = 
perf_results[0].algo; }
#endif
 if (CUDNN_VERSION < 7402) fixConvAlgo(); size_t tnTPxeDjBsqLAPkJcPJX = 0; 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
YCSvyQZBWMDYQXHtyVai, QhTesEEIHwhNmHSeYbRR, NZjOkZPwLzQsdEVkwMcX, *desc, 
NNhshzQGJHLSGjDiVerE, &tnTPxeDjBsqLAPkJcPJX)); if( tnTPxeDjBsqLAPkJcPJX > 
*dMxIKDGTITyhdLqIHBLA->getProposedWorkSpaceSize() ) { 
dMxIKDGTITyhdLqIHBLA->setProposedWorkSpaceSize(tnTPxeDjBsqLAPkJcPJX); }
#endif
 } void MWFusedConvReLULayerImpl::allocate() { MWTensorBase* ipTensor_conv = 
getLayer()->getInputTensor(0); if (IbSWJNMuIiKbocfQKqXb) { float* 
newInput; int inputH = ipTensor_conv->getHeight() + CTCbzQMDaLxINPbODdng + 
CLOUhPjbgggWoXHTtmjC; int inputW = ipTensor_conv->getWidth() + 
CpMjJjtGOeWOzwxpAAQP + CqtPRJvHlGJFssiPzsOm; int paddedSize = 
ipTensor_conv->getBatchSize() * ipTensor_conv->getChannels() * inputH * inputW; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float)*paddedSize)); 
CUDA_CALL(hipMemset(newInput, 0, sizeof(float)*paddedSize)); 
static_cast<MWTensor<float>*>(URgvgDXnZskIYGdtimcU)->setData(newInput); } 
XCnEVUzxqcNgsuUbRonz = 
static_cast<MWTensor<float>*>(getLayer()->getOutputTensor(0))->getData(); 
setalpha2Ptr(getZeroPtr()); int numInputs = getLayer()->getNumInputs(); if 
(numInputs == 2) { setalpha2Ptr(getOnePtr()); XCnEVUzxqcNgsuUbRonz = 
static_cast<MWTensor<float>*>(getLayer()->getInputTensor(1))->getData(); } if 
(static_cast<MWTensor<float>*>(URgvgDXnZskIYGdtimcU)->getData() == 
XCnEVUzxqcNgsuUbRonz){ int xInputTensorSize = 
getLayer()->getInputTensor(0)->getNumElements(); 
CUDA_CALL(hipMalloc((void**)&GFienSVKLlDQuZeqAdLC, sizeof(float) * 
xInputTensorSize)); } } void MWFusedConvReLULayerImpl::deallocate() { if 
(URgvgDXnZskIYGdtimcU != getLayer()->getInputTensor(0)) { 
assert(IbSWJNMuIiKbocfQKqXb); 
CUDA_FREE_CALL(static_cast<MWTensor<float>*>(URgvgDXnZskIYGdtimcU)->getData()); 
static_cast<MWTensor<float>*>(URgvgDXnZskIYGdtimcU)->setData((float*)NULL); } if 
(GFienSVKLlDQuZeqAdLC){ CUDA_FREE_CALL(GFienSVKLlDQuZeqAdLC); 
GFienSVKLlDQuZeqAdLC = NULL;  }  } void 
MWFusedConvReLULayerImpl::predict() { MWCNNLayer* fusedConvReluLayer = 
getLayer(); MWTensorBase* ipTensorBase = fusedConvReluLayer->getInputTensor(); 
MWTensorBase* opTensorBase = fusedConvReluLayer->getOutputTensor(); 
MWTensor<float>* ipTensor = static_cast<MWTensor<float>*>(ipTensorBase); 
MWTensor<float>* opTensor = static_cast<MWTensor<float>*>(opTensorBase); if 
(URgvgDXnZskIYGdtimcU != fusedConvReluLayer->getInputTensor()) { 
CUDA_CALL(hipMemset(static_cast<MWTensor<float>*>(URgvgDXnZskIYGdtimcU)->getData(), 
0, sizeof(float)*URgvgDXnZskIYGdtimcU->getNumElements())); 
MWCNNLayerImpl::padInput(ipTensor->getData(), ipTensor->getHeight(), 
ipTensor->getWidth(), ipTensor->getChannels(), URgvgDXnZskIYGdtimcU->getHeight(), 
URgvgDXnZskIYGdtimcU->getWidth(), bYBVtTnVUuGDUlaTmmHp, cQBKlCKXxecGPJrXBXdk, 
static_cast<MWTensor<float>*>(URgvgDXnZskIYGdtimcU)->getData(), 
ipTensor->getNumElements()); } hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensor->getSourcePortIndex()); assert(desc);
#if (CUDNN_MAJOR >= 6)
 assert(opTensor->getData() != 
static_cast<MWTensor<float>*>(URgvgDXnZskIYGdtimcU)->getData() || 
(getLayer()->getNumInputs() == 2)); float* rIcMzXptfYweLArNRnBw; if 
(static_cast<MWTensor<float>*>(URgvgDXnZskIYGdtimcU)->getData() == 
XCnEVUzxqcNgsuUbRonz){ CUDA_CALL(hipMemcpy(GFienSVKLlDQuZeqAdLC, 
static_cast<MWTensor<float>*>(URgvgDXnZskIYGdtimcU)->getData(), sizeof(float) * 
opTensorBase->getNumElements(), hipMemcpyDeviceToDevice)); 
rIcMzXptfYweLArNRnBw = GFienSVKLlDQuZeqAdLC; }else{ 
rIcMzXptfYweLArNRnBw = 
static_cast<MWTensor<float>*>(URgvgDXnZskIYGdtimcU)->getData(); } 
CUDNN_CALL(cudnnConvolutionBiasActivationForward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
getOnePtr(), YCSvyQZBWMDYQXHtyVai, rIcMzXptfYweLArNRnBw, 
QhTesEEIHwhNmHSeYbRR, vpXxoeEhdEosLSsYXkNG, NZjOkZPwLzQsdEVkwMcX, NNhshzQGJHLSGjDiVerE, 
dMxIKDGTITyhdLqIHBLA->getWorkSpace(), 
*dMxIKDGTITyhdLqIHBLA->getAllocatedWorkSpaceSize(), getalpha2Ptr(),  *desc,  
XCnEVUzxqcNgsuUbRonz,  JwxFdqOKggeawILBfGgg, JCaothdxtQBQxaaitLio, oJUVMnJggjhEdQLWzIUC, 
*desc, opTensor->getData()));
#endif
 } void MWFusedConvReLULayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(NZjOkZPwLzQsdEVkwMcX)); 
CUDNN_CALL(hipdnnDestroyFilterDescriptor(QhTesEEIHwhNmHSeYbRR)); 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(oJUVMnJggjhEdQLWzIUC)); if 
(vpXxoeEhdEosLSsYXkNG) { CUDA_FREE_CALL(vpXxoeEhdEosLSsYXkNG); vpXxoeEhdEosLSsYXkNG = NULL; } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(JwxFdqOKggeawILBfGgg)); if 
(JCaothdxtQBQxaaitLio) { CUDA_FREE_CALL(JCaothdxtQBQxaaitLio); JCaothdxtQBQxaaitLio = NULL; } if 
(URgvgDXnZskIYGdtimcU != getLayer()->getInputTensor(0)) { 
assert(IbSWJNMuIiKbocfQKqXb); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(YCSvyQZBWMDYQXHtyVai)); } } void 
MWFusedConvReLULayerImpl::loadWeights(const char* QVgVGfoCXYiYXzPhvVPX) { FILE* 
QwUuNuQNtlPXrIwRNiSZ = MWCNNLayer::openBinaryFile(QVgVGfoCXYiYXzPhvVPX); 
assert(QwUuNuQNtlPXrIwRNiSZ); int dkLDkRwCBjeybwDHbKiE = 
BkwhtPQUCQKchmmimoXs*BuyZFXzwOMxcePIbCLfl*BlRIQPyqJZORKENzSdYf*BHuHNDGoRwGRouCxeMbw*BLjrjqvCcCommiXWQLjs; 
 float* LgxABSJPBXdCozJkFqTg = MALLOC_CALL(sizeof(float)*dkLDkRwCBjeybwDHbKiE); 
MWCNNLayer::call_fread(LgxABSJPBXdCozJkFqTg, sizeof(float), dkLDkRwCBjeybwDHbKiE, 
QwUuNuQNtlPXrIwRNiSZ, QVgVGfoCXYiYXzPhvVPX); CUDA_CALL(hipMemcpy(vpXxoeEhdEosLSsYXkNG, 
LgxABSJPBXdCozJkFqTg, sizeof(float)*dkLDkRwCBjeybwDHbKiE, hipMemcpyHostToDevice));
#if 0
 printf("%s loaded. Size = %d. %f\n", QVgVGfoCXYiYXzPhvVPX, dkLDkRwCBjeybwDHbKiE, LgxABSJPBXdCozJkFqTg[0]);
#endif
 free(LgxABSJPBXdCozJkFqTg); fclose(QwUuNuQNtlPXrIwRNiSZ); return; } void 
MWFusedConvReLULayerImpl::loadBias(const char* QVgVGfoCXYiYXzPhvVPX) { FILE* 
QwUuNuQNtlPXrIwRNiSZ = MWCNNLayer::openBinaryFile(QVgVGfoCXYiYXzPhvVPX); 
assert(QwUuNuQNtlPXrIwRNiSZ); int dkLDkRwCBjeybwDHbKiE = 
BuyZFXzwOMxcePIbCLfl*BlRIQPyqJZORKENzSdYf;  float* LgxABSJPBXdCozJkFqTg = 
MALLOC_CALL(sizeof(float)*dkLDkRwCBjeybwDHbKiE); 
MWCNNLayer::call_fread(LgxABSJPBXdCozJkFqTg, sizeof(float), dkLDkRwCBjeybwDHbKiE, 
QwUuNuQNtlPXrIwRNiSZ, QVgVGfoCXYiYXzPhvVPX); CUDA_CALL(hipMemcpy(JCaothdxtQBQxaaitLio, 
LgxABSJPBXdCozJkFqTg, sizeof(float)*dkLDkRwCBjeybwDHbKiE, hipMemcpyHostToDevice)); 
free(LgxABSJPBXdCozJkFqTg); fclose(QwUuNuQNtlPXrIwRNiSZ); return; } void 
MWFusedConvReLULayerImpl::postSetup() { if (dMxIKDGTITyhdLqIHBLA->getAutoTune()) 
{ getConvAlgoTuned(); } else { getConvAlgoWorkSpaceLimit(); } } void 
MWFusedConvReLULayerImpl::getConvAlgoTuned() { MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); hipdnnConvolutionFwdAlgoPerf_t 
perf_results[3]; hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); 
assert(desc); int returnedAlgoCount; 
CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithmEx(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
YCSvyQZBWMDYQXHtyVai, 
static_cast<MWTensor<float>*>(URgvgDXnZskIYGdtimcU)->getData(), 
QhTesEEIHwhNmHSeYbRR, vpXxoeEhdEosLSsYXkNG, NZjOkZPwLzQsdEVkwMcX, *desc, 
opTensor->getData(), 3, &returnedAlgoCount, &perf_results[0], 
dMxIKDGTITyhdLqIHBLA->getWorkSpace(), 
*dMxIKDGTITyhdLqIHBLA->getAllocatedWorkSpaceSize())); NNhshzQGJHLSGjDiVerE = 
perf_results[0].algo; if (CUDNN_VERSION < 7402) fixConvAlgo(); } void 
MWFusedConvReLULayerImpl::getConvAlgoWorkSpaceLimit() { 
hipdnnTensorDescriptor_t* desc = 
getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex()); assert(desc);
#if (CUDNN_MAJOR < 8)
 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
YCSvyQZBWMDYQXHtyVai, QhTesEEIHwhNmHSeYbRR, NZjOkZPwLzQsdEVkwMcX, *desc, 
HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT, 
*dMxIKDGTITyhdLqIHBLA->getAllocatedWorkSpaceSize(), &NNhshzQGJHLSGjDiVerE));
#else
 int maxAlgoCount(-1); 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithmMaxCount(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
&maxAlgoCount)); int returnedAlgoCount(-1); 
std::vector<hipdnnConvolutionFwdAlgoPerf_t> perf_results(maxAlgoCount);  
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
YCSvyQZBWMDYQXHtyVai, QhTesEEIHwhNmHSeYbRR, NZjOkZPwLzQsdEVkwMcX, *desc, 
maxAlgoCount, &returnedAlgoCount, &perf_results[0])); 
hipdnnConvolutionFwdAlgoPerf_t nextFastest; bool algoFound(false); for (int i = 
0; i < returnedAlgoCount; ++i) { nextFastest = perf_results[i]; if 
(nextFastest.memory <= *dMxIKDGTITyhdLqIHBLA->getAllocatedWorkSpaceSize()) { 
NNhshzQGJHLSGjDiVerE = nextFastest.algo; algoFound = true; break; } } assert(algoFound);
#endif
 if (CUDNN_VERSION < 7402) fixConvAlgo(); } void 
MWFusedConvReLULayerImpl::fixConvAlgo() { int inputH = 
URgvgDXnZskIYGdtimcU->getHeight(); int inputW = URgvgDXnZskIYGdtimcU->getWidth(); 
if (NNhshzQGJHLSGjDiVerE == HIPDNN_CONVOLUTION_FWD_ALGO_FFT && (inputH > 64 || 
inputW > 64)) { NNhshzQGJHLSGjDiVerE = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM; 
} }