#include "hip/hip_runtime.h"
//
// File: predict.cu
//
// GPU Coder version                    : 2.1
// CUDA/C/C++ source code generated on  : 08-Apr-2021 15:32:32
//

// Include Files
#include "predict.h"
#include "DeepLearningNetwork.h"
#include "line_detection_internal_types.h"
#include "MWCudaDimUtility.hpp"

// Type Definitions
struct cell_wrap_13 {
  float f1[2];
};

struct cell_wrap_9 {
  float f1[172800];
};

struct cell_wrap_12 {
  float f1[172800];
};

// Function Declarations
static __global__ void
DeepLearningNetwork_predict_kernel40(const unsigned char varargin_1[172800],
                                     cell_wrap_9 dataInputsSingle[1]);

static __global__ void
DeepLearningNetwork_predict_kernel41(const cell_wrap_9 dataInputsSingle[1],
                                     cell_wrap_12 inMiniBatchGroup[1]);

static __global__ void
DeepLearningNetwork_predict_kernel42(const cell_wrap_13 outMiniBatchGroup[1],
                                     float varargout_1[2]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char varargin_1[172800]
//                cell_wrap_9 dataInputsSingle[1]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void DeepLearningNetwork_predict_kernel40(
        const unsigned char varargin_1[172800], cell_wrap_9 dataInputsSingle[1])
{
  unsigned long threadId;
  int i;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId);
  if (i < 172800) {
    dataInputsSingle[0].f1[i] = static_cast<float>(varargin_1[i]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const cell_wrap_9 dataInputsSingle[1]
//                cell_wrap_12 inMiniBatchGroup[1]
// Return Type  : void
//
static __global__
    __launch_bounds__(512, 1) void DeepLearningNetwork_predict_kernel41(
        const cell_wrap_9 dataInputsSingle[1], cell_wrap_12 inMiniBatchGroup[1])
{
  unsigned long threadId;
  int i;
  int i1;
  int p;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId % 320UL);
  threadId = (threadId - static_cast<unsigned long>(i)) / 320UL;
  i1 = static_cast<int>(threadId % 180UL);
  threadId = (threadId - static_cast<unsigned long>(i1)) / 180UL;
  p = static_cast<int>(threadId);
  if ((static_cast<int>((static_cast<int>(p < 3)) &&
                        (static_cast<int>(i1 < 180)))) &&
      (static_cast<int>(i < 320))) {
    inMiniBatchGroup[0].f1[(i + 320 * i1) + 57600 * p] =
        dataInputsSingle[0].f1[(i1 + 180 * i) + 57600 * p];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const cell_wrap_13 outMiniBatchGroup[1]
//                float varargout_1[2]
// Return Type  : void
//
static __global__
    __launch_bounds__(32, 1) void DeepLearningNetwork_predict_kernel42(
        const cell_wrap_13 outMiniBatchGroup[1], float varargout_1[2])
{
  unsigned long threadId;
  int i;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int>(threadId);
  if (i < 2) {
    varargout_1[i] = outMiniBatchGroup[0].f1[i];
  }
}

//
// Arguments    : mynet_new0_0 *obj
//                const unsigned char varargin_1[172800]
//                float varargout_1[2]
// Return Type  : void
//
namespace coder {
void DeepLearningNetwork_predict(mynet_new0_0 *obj,
                                 const unsigned char varargin_1[172800],
                                 float varargout_1[2])
{
  cell_wrap_12(*gpu_inMiniBatchGroup)[1];
  cell_wrap_13(*gpu_outMiniBatchGroup)[1];
  cell_wrap_9(*gpu_dataInputsSingle)[1];
  float(*gpu_varargout_1)[2];
  unsigned char(*gpu_varargin_1)[172800];
  hipMalloc(&gpu_varargout_1, 8UL);
  hipMalloc(&gpu_outMiniBatchGroup, 8UL);
  hipMalloc(&gpu_inMiniBatchGroup, 691200UL);
  hipMalloc(&gpu_dataInputsSingle, 691200UL);
  hipMalloc(&gpu_varargin_1, 172800UL);
  hipMemcpy(*gpu_varargin_1, varargin_1, 172800UL, hipMemcpyHostToDevice);
  DeepLearningNetwork_predict_kernel40<<<dim3(338U, 1U, 1U),
                                         dim3(512U, 1U, 1U)>>>(
      *gpu_varargin_1, *gpu_dataInputsSingle);
  DeepLearningNetwork_predict_kernel41<<<dim3(338U, 1U, 1U),
                                         dim3(512U, 1U, 1U)>>>(
      *gpu_dataInputsSingle, *gpu_inMiniBatchGroup);
  hipMemcpy(obj->getInputDataPointer(0), (*gpu_inMiniBatchGroup)[0].f1,
             obj->getLayerOutputSize(0, 0), hipMemcpyDeviceToDevice);
  obj->predict();
  hipMemcpy((*gpu_outMiniBatchGroup)[0].f1, obj->getLayerOutput(25, 0),
             obj->getLayerOutputSize(25, 0), hipMemcpyDeviceToDevice);
  DeepLearningNetwork_predict_kernel42<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      *gpu_outMiniBatchGroup, *gpu_varargout_1);
  hipMemcpy(varargout_1, *gpu_varargout_1, 8UL, hipMemcpyDeviceToHost);
  hipFree(*gpu_varargin_1);
  hipFree(*gpu_dataInputsSingle);
  hipFree(*gpu_inMiniBatchGroup);
  hipFree(*gpu_outMiniBatchGroup);
  hipFree(*gpu_varargout_1);
}

} // namespace coder

//
// File trailer for predict.cu
//
// [EOF]
//
