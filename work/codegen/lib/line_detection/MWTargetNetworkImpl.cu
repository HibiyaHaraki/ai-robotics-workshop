#include "MWTargetNetworkImpl.hpp"
#include "MWCNNLayerImpl.hpp"
#include <cassert>
#include <cmath>
#include <algorithm>
 void MWTargetNetworkImpl::allocate(int BufSize, int numBufsToAlloc) { numBufs 
= numBufsToAlloc; for(int i = 0; i < numBufs; i++) { float *memPtr = 0; 
CUDA_CALL(hipMalloc((void**)&memPtr, sizeof(float)*BufSize)); 
memBuffer.push_back(memPtr); }  } void 
MWTargetNetworkImpl::allocatePermuteBuffers(int bufSize, int numBufsToAlloc) { 
for (int i = 0; i < numBufsToAlloc; i++) { float* memPtr = 0; 
CUDA_CALL(hipMalloc((void**)&memPtr, sizeof(float) * bufSize)); 
mbKaFvmHqfBiTISNPGKJ.push_back(memPtr); } } void 
MWTargetNetworkImpl::preSetup() {  NmExSIssnXpisMKKatUq = new 
hipblasHandle_t; if(!NmExSIssnXpisMKKatUq) { 
MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); } 
CUBLAS_CALL(hipblasCreate(NmExSIssnXpisMKKatUq)); NnAKUXChhnRnQmWsknGy 
= new hipdnnHandle_t; if(!NnAKUXChhnRnQmWsknGy) { 
MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); } 
CUDNN_CALL(hipdnnCreate(NnAKUXChhnRnQmWsknGy));  } void 
MWTargetNetworkImpl::postSetup(MWCNNLayer* layers[],int numLayers) { if 
(*getProposedWorkSpaceSize() > *getAllocatedWorkSpaceSize()) { if 
(xcusoQxPPodcHwVviCWI) { destroyWorkSpace(xcusoQxPPodcHwVviCWI); } 
createWorkSpace(xcusoQxPPodcHwVviCWI); while ((!xcusoQxPPodcHwVviCWI) && 
(*getProposedWorkSpaceSize() > 0)) { 
setProposedWorkSpaceSize(MWTargetNetworkImpl::getNextProposedWorkSpaceSize(*getProposedWorkSpaceSize())); 
createWorkSpace(xcusoQxPPodcHwVviCWI); } } for (int i = 0; i < numLayers; i++) 
{ layers[i]->postSetup();  }  } size_t 
MWTargetNetworkImpl::getNextProposedWorkSpaceSize(size_t failedWorkSpaceSize) { 
assert(failedWorkSpaceSize > 0); return failedWorkSpaceSize/2; } void 
MWTargetNetworkImpl::createWorkSpace(float*& xkUNToJIgvoLoUQuzKRF) { 
hipError_t rlQsibXJSWJVnUVpdNeL = hipMalloc((void**)&xkUNToJIgvoLoUQuzKRF, 
*getProposedWorkSpaceSize()); if (rlQsibXJSWJVnUVpdNeL != hipSuccess) { 
xkUNToJIgvoLoUQuzKRF = NULL; setAllocatedWorkSpaceSize(0);  
rlQsibXJSWJVnUVpdNeL = hipGetLastError();  } else { 
setAllocatedWorkSpaceSize(*getProposedWorkSpaceSize()); } } void 
MWTargetNetworkImpl::destroyWorkSpace(float*& xkUNToJIgvoLoUQuzKRF) { 
CUDA_FREE_CALL(xkUNToJIgvoLoUQuzKRF); xkUNToJIgvoLoUQuzKRF = NULL; 
setAllocatedWorkSpaceSize(0);  } void 
MWTargetNetworkImpl::setProposedWorkSpaceSize(size_t wss) { 
nNULvWnBXnnWdpEkHPAH = wss;  } size_t* 
MWTargetNetworkImpl::getProposedWorkSpaceSize() { return 
&nNULvWnBXnnWdpEkHPAH; } void 
MWTargetNetworkImpl::setAllocatedWorkSpaceSize(size_t wss) { 
HJHXkKmgFxxIOsIvRRnF = wss;  } size_t* 
MWTargetNetworkImpl::getAllocatedWorkSpaceSize() { return 
&HJHXkKmgFxxIOsIvRRnF; } float* 
MWTargetNetworkImpl::getWorkSpace() { return xcusoQxPPodcHwVviCWI; } float* 
MWTargetNetworkImpl::getPermuteBuffer(int bufIndex) { return 
mbKaFvmHqfBiTISNPGKJ[bufIndex]; } hipblasHandle_t* 
MWTargetNetworkImpl::getCublasHandle() { return NmExSIssnXpisMKKatUq; } 
hipdnnHandle_t* MWTargetNetworkImpl::getCudnnHandle() { return 
NnAKUXChhnRnQmWsknGy; } void MWTargetNetworkImpl::setAutoTune(bool 
autotune) { MW_autoTune = autotune; } bool MWTargetNetworkImpl::getAutoTune() 
const { return MW_autoTune; } void MWTargetNetworkImpl::deallocate() { for(int 
i = 0; i < memBuffer.size(); i++) { float *memPtr = memBuffer[i]; if(memPtr) { 
CUDA_FREE_CALL(memPtr); }  } memBuffer.clear(); for(int i = 0; i < 
mbKaFvmHqfBiTISNPGKJ.size(); i++) { float *memPtr = 
mbKaFvmHqfBiTISNPGKJ[i]; if(memPtr) { CUDA_FREE_CALL(memPtr); } } 
mbKaFvmHqfBiTISNPGKJ.clear(); } void MWTargetNetworkImpl::cleanup() { if 
(xcusoQxPPodcHwVviCWI) { destroyWorkSpace(xcusoQxPPodcHwVviCWI); } if 
(NmExSIssnXpisMKKatUq) { 
CUBLAS_CALL(hipblasDestroy(*NmExSIssnXpisMKKatUq)); delete 
NmExSIssnXpisMKKatUq; } if (NnAKUXChhnRnQmWsknGy) {
#if (CUDNN_MAJOR < 8) 
 CUDNN_CALL(hipdnnDestroy(*NnAKUXChhnRnQmWsknGy));
#else
 hipError_t hipError_t = hipPeekAtLastError(); if (hipError_t != 
hipErrorDeinitialized) { 
CUDNN_CALL(hipdnnDestroy(*NnAKUXChhnRnQmWsknGy)); }
#endif
 delete NnAKUXChhnRnQmWsknGy; } } float * 
MWTargetNetworkImpl::getBufferPtr(int bufferIndex) { return 
memBuffer[bufferIndex];  } MWTargetNetworkImpl::~MWTargetNetworkImpl() { }