#include "hip/hip_runtime.h"
#include "MWKernelHeaders.hpp"
#include <math.h>
#include <stdio.h>
 void __global__ __launch_bounds__(1024) scale_scalar_kernel(float* 
inputBuffer, float* outputBuffer, float* pvpNsgGssdTxeVoFIkXI, long int 
YNmJhGSUszJKxsodxiuV) {  for (long int idx = blockDim.x * blockIdx.x + 
threadIdx.x; idx < YNmJhGSUszJKxsodxiuV; idx += blockDim.x * gridDim.x) {  
outputBuffer[idx] = pvpNsgGssdTxeVoFIkXI[0]*inputBuffer[idx]; } } void __global__ 
__launch_bounds__(1024) scale_vector_kernel(float* inputBuffer, float* 
outputBuffer, float* pvpNsgGssdTxeVoFIkXI, double YeIFysyIXePEVfpcANol, 
double YOWMnLKOMqAODXiVNoGy, long int YNmJhGSUszJKxsodxiuV) {  for 
(long int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < 
YNmJhGSUszJKxsodxiuV; idx += blockDim.x * gridDim.x) { double batchIdx = 
floor(idx / YOWMnLKOMqAODXiVNoGy); double i_batch = idx - (batchIdx * 
YOWMnLKOMqAODXiVNoGy); double channelIdx = floor(i_batch / 
YeIFysyIXePEVfpcANol); outputBuffer[idx] = 
pvpNsgGssdTxeVoFIkXI[static_cast<long int>(channelIdx)]*inputBuffer[idx]; } } void 
__global__ __launch_bounds__(1024) scale_matrix2d_kernel(float* inputBuffer, 
float* outputBuffer, float* pvpNsgGssdTxeVoFIkXI, double 
YeIFysyIXePEVfpcANol, long int YNmJhGSUszJKxsodxiuV) {  for (long int 
idx = blockDim.x * blockIdx.x + threadIdx.x; idx < YNmJhGSUszJKxsodxiuV; idx += 
blockDim.x * gridDim.x) { double totalChannelIdx = floor(idx / 
YeIFysyIXePEVfpcANol); double i_channel = idx - (totalChannelIdx * 
YeIFysyIXePEVfpcANol); outputBuffer[idx] = 
pvpNsgGssdTxeVoFIkXI[static_cast<long int>(i_channel)]*inputBuffer[idx]; } } void 
__global__ __launch_bounds__(1024) scale_tensor3d_kernel(float* inputBuffer, 
float* outputBuffer, float* pvpNsgGssdTxeVoFIkXI, double 
YOWMnLKOMqAODXiVNoGy, long int YNmJhGSUszJKxsodxiuV) {  for (long int 
idx = blockDim.x * blockIdx.x + threadIdx.x; idx < YNmJhGSUszJKxsodxiuV; idx += 
blockDim.x * gridDim.x) { double batchIdx = floor(idx / 
YOWMnLKOMqAODXiVNoGy); double i_batch = idx - (batchIdx * 
YOWMnLKOMqAODXiVNoGy); outputBuffer[idx] = 
pvpNsgGssdTxeVoFIkXI[static_cast<long int>(i_batch)]*inputBuffer[idx]; } }  void 
__global__ __launch_bounds__(1024) offset_scalar_kernel(float* inputBuffer, 
float* outputBuffer, float* gNROjwaqhxDPvBWUCUcQ, long int YNmJhGSUszJKxsodxiuV, 
bool ZinudJuZuGitiNTsJpBR, int bUVPfnrJhLfHzOLUUrKk, int 
veFyKKHbdqBIvQLYBqfF) {  for (long int idx = blockDim.x * blockIdx.x + 
threadIdx.x; idx < YNmJhGSUszJKxsodxiuV; idx += blockDim.x * gridDim.x) { float 
out = inputBuffer[idx] + gNROjwaqhxDPvBWUCUcQ[0]; if (ZinudJuZuGitiNTsJpBR){ out = 
out > veFyKKHbdqBIvQLYBqfF ? veFyKKHbdqBIvQLYBqfF : out; out = out < 
bUVPfnrJhLfHzOLUUrKk ? bUVPfnrJhLfHzOLUUrKk : out; } outputBuffer[idx] = out; 
} } void __global__ __launch_bounds__(1024) offset_vector_kernel(float* 
inputBuffer, float* outputBuffer, float* gNROjwaqhxDPvBWUCUcQ,  double 
YeIFysyIXePEVfpcANol, double YOWMnLKOMqAODXiVNoGy, long int 
YNmJhGSUszJKxsodxiuV, bool ZinudJuZuGitiNTsJpBR, int bUVPfnrJhLfHzOLUUrKk, int 
veFyKKHbdqBIvQLYBqfF) {  for (long int idx = blockDim.x * blockIdx.x + 
threadIdx.x; idx < YNmJhGSUszJKxsodxiuV; idx += blockDim.x * gridDim.x) { 
double batchIdx = floor(idx / YOWMnLKOMqAODXiVNoGy); double i_batch = 
idx - (batchIdx * YOWMnLKOMqAODXiVNoGy); double channelIdx = 
floor(i_batch / YeIFysyIXePEVfpcANol); float out = inputBuffer[idx] + 
gNROjwaqhxDPvBWUCUcQ[static_cast<long int>(channelIdx)]; if 
(ZinudJuZuGitiNTsJpBR){ out = out > veFyKKHbdqBIvQLYBqfF ? 
veFyKKHbdqBIvQLYBqfF : out; out = out < bUVPfnrJhLfHzOLUUrKk ? 
bUVPfnrJhLfHzOLUUrKk : out; } outputBuffer[idx] = out; } } void __global__ 
__launch_bounds__(1024) offset_matrix2d_kernel(float* inputBuffer, float* 
outputBuffer, float* gNROjwaqhxDPvBWUCUcQ, double YeIFysyIXePEVfpcANol, 
long int YNmJhGSUszJKxsodxiuV, bool ZinudJuZuGitiNTsJpBR, int 
bUVPfnrJhLfHzOLUUrKk, int veFyKKHbdqBIvQLYBqfF) {  for (long int idx = 
blockDim.x * blockIdx.x + threadIdx.x; idx < YNmJhGSUszJKxsodxiuV; idx += 
blockDim.x * gridDim.x) { double totalChannelIdx = floor(idx / 
YeIFysyIXePEVfpcANol); double i_channel = idx - (totalChannelIdx * 
YeIFysyIXePEVfpcANol); float out = inputBuffer[idx] + 
gNROjwaqhxDPvBWUCUcQ[static_cast<long int>(i_channel)]; if (ZinudJuZuGitiNTsJpBR){ 
out = out > veFyKKHbdqBIvQLYBqfF ? veFyKKHbdqBIvQLYBqfF : out; out = out < 
bUVPfnrJhLfHzOLUUrKk ? bUVPfnrJhLfHzOLUUrKk : out; } outputBuffer[idx] = out; 
} } void __global__ __launch_bounds__(1024) offset_tensor3d_kernel(float* 
inputBuffer, float* outputBuffer, float* gNROjwaqhxDPvBWUCUcQ, double 
YOWMnLKOMqAODXiVNoGy, long int YNmJhGSUszJKxsodxiuV, bool 
ZinudJuZuGitiNTsJpBR, int bUVPfnrJhLfHzOLUUrKk, int veFyKKHbdqBIvQLYBqfF) {  
for (long int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < 
YNmJhGSUszJKxsodxiuV; idx += blockDim.x * gridDim.x) { double batchIdx = 
floor(idx / YOWMnLKOMqAODXiVNoGy); double i_batch = idx - (batchIdx * 
YOWMnLKOMqAODXiVNoGy); float out = inputBuffer[idx] + 
gNROjwaqhxDPvBWUCUcQ[static_cast<long int>(i_batch)]; if (ZinudJuZuGitiNTsJpBR){ 
out = out > veFyKKHbdqBIvQLYBqfF ? veFyKKHbdqBIvQLYBqfF : out; out = out < 
bUVPfnrJhLfHzOLUUrKk ? bUVPfnrJhLfHzOLUUrKk : out; } outputBuffer[idx] = out; 
} } 