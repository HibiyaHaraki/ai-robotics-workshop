#include "hip/hip_runtime.h"
//
// File: line_detection.cu
//
// GPU Coder version                    : 2.1
// CUDA/C/C++ source code generated on  : 11-Nov-2021 11:08:16
//

// Include Files
#include "line_detection.h"
#include "DeepLearningNetwork.h"
#include "line_detection_data.h"
#include "line_detection_initialize.h"
#include "line_detection_internal_types.h"
#include "predict.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "insertShapeUtilsCore_api.hpp"
#include <cmath>

// Variable Definitions
static mynet_new0_0 lineDetObj;

static bool lineDetObj_not_empty;

// Function Declarations
static __global__ void
line_detection_kernel1(const unsigned char I_data[], const int I_size[3],
                       const int img_size[3], const int b_I_size,
                       const int plast, unsigned char img_data[2764800]);

static __global__ void line_detection_kernel10(const double aux1_data[1440],
                                               const int in_rows,
                                               const double scale_idx_0,
                                               const double kwidthRow,
                                               double ipRowIndices_data[2880],
                                               double rowWeights_data[2880]);

static __global__ void line_detection_kernel11(const int plast,
                                               double ipColIndices_data[9280]);

static __global__ void line_detection_kernel12(const int plast,
                                               double colWeights_data[9280]);

static __global__ void line_detection_kernel13(const double aux2_data[2560],
                                               const int in_cols,
                                               const double scale_idx_1,
                                               const double kwidthCol,
                                               double ipColIndices_data[9280],
                                               double colWeights_data[9280]);

static __global__ void
line_detection_kernel14(const double rowWeights_data[2880],
                        double rowWeightsTotal[180]);

static __global__ void
line_detection_kernel15(const double rowWeights_data[2880], const int in_rows,
                        double rowWeightsTotal[180]);

static __global__ void
line_detection_kernel16(const double colWeights_data[9280],
                        double colWeightsTotal[320]);

static __global__ void
line_detection_kernel17(const double colWeights_data[9280], const int in_rows,
                        double colWeightsTotal[320]);

static __global__ void line_detection_kernel18(const int img_size[3],
                                               int partialResize_size[3]);

static __global__ void line_detection_kernel19(
    const double colWeightsTotal[320], const double colWeights_data[9280],
    const unsigned char img_data[2764800], const double ipColIndices_data[9280],
    const int img_size[3], const int partialResize_size[3],
    const double kwidthCol, const int plast,
    unsigned char partialResize_data[691200]);

static __global__ void line_detection_kernel2(const unsigned char I_data[],
                                              const int I_size,
                                              unsigned char img_data[2764800]);

static __global__ void line_detection_kernel20(
    const double rowWeightsTotal[180], const double rowWeights_data[2880],
    const unsigned char partialResize_data[691200],
    const int partialResize_size[3], const double ipRowIndices_data[2880],
    const double kwidthRow, unsigned char out[172800]);

static __global__ void line_detection_kernel21(const int img_size[3],
                                               int partialResize_size[3]);

static __global__ void line_detection_kernel22(
    const double rowWeightsTotal[180], const double rowWeights_data[2880],
    const unsigned char img_data[2764800], const int img_size[3],
    const double ipRowIndices_data[2880], const int partialResize_size[3],
    const double kwidthRow, const int plast,
    unsigned char partialResize_data[691200]);

static __global__ void line_detection_kernel23(
    const double colWeightsTotal[320], const double colWeights_data[9280],
    const unsigned char partialResize_data[691200],
    const int partialResize_size[3], const double ipColIndices_data[9280],
    const double kwidthCol, unsigned char out[172800]);

static __global__ void line_detection_kernel24(const short sz[3],
                                               int positionOut_data[4]);

static __global__ void line_detection_kernel25(const short sz[3],
                                               int positionOut_data[4]);

static __global__ void line_detection_kernel26(const int plast,
                                               const float predictedPosNorm[2],
                                               int positionOut_data[4],
                                               float predictedPos[2]);

static __global__ void line_detection_kernel27(const int img_size[3],
                                               short sz[3]);

static __global__ void line_detection_kernel28(const int sz,
                                               unsigned char img_data[2764800]);

static __global__ void
line_detection_kernel29(const short pixCountSize,
                        unsigned char pixCount_data[1280]);

static __global__ void
line_detection_kernel3(const unsigned char img_data[2764800],
                       const int img_size[3], const int b_img_size[3],
                       const int plast, const int in_rows,
                       unsigned char b_img_data[2764800]);

static __global__ void line_detection_kernel30(const unsigned char color[3],
                                               unsigned char b_color[3]);

static __global__ void line_detection_kernel31(const float predictedPos[2],
                                               int positionOut[3]);

static __global__ void line_detection_kernel32(int positionOut[3]);

static __global__ void line_detection_kernel33(const short tmpRGB_size[3],
                                               short sz[3]);

static __global__ void line_detection_kernel34(const int sz,
                                               unsigned char img_data[2764800]);

static __global__ void
line_detection_kernel35(const short pixCountSize,
                        unsigned char pixCount_data[1280]);

static __global__ void line_detection_kernel36(const unsigned char color[3],
                                               unsigned char b_color[3]);

static __global__ void
line_detection_kernel37(const unsigned char img_data[2764800],
                        const short out_size[3], const int Iout_size[3],
                        const int b_out_size, const int c_out_size,
                        unsigned char Iout_data[]);

static __global__ void
line_detection_kernel38(const unsigned char img_data[2764800],
                        const short out_size[3], const short b_out_size[3],
                        const int plast, const int in_rows,
                        unsigned char b_img_data[2764800]);

static __global__ void
line_detection_kernel39(const unsigned char img_data[2764800],
                        const int tmpRGB_size, unsigned char Iout_data[]);

static __global__ void
line_detection_kernel4(const unsigned char img_data[2764800],
                       const int img_size, unsigned char b_img_data[2764800]);

static __global__ void line_detection_kernel5(const int img_size[3],
                                              short sz[3]);

static __global__ void line_detection_kernel6(const int in_rows,
                                              const int plast,
                                              double aux1_data[1440]);

static __global__ void line_detection_kernel7(const int in_cols,
                                              const int plast,
                                              double aux2_data[2560]);

static __global__ void line_detection_kernel8(const int plast,
                                              double ipRowIndices_data[2880]);

static __global__ void line_detection_kernel9(const int plast,
                                              double rowWeights_data[2880]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char I_data[]
//                const int I_size[3]
//                const int img_size[3]
//                const int b_I_size
//                const int plast
//                unsigned char img_data[2764800]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel1(
    const unsigned char I_data[], const int I_size[3], const int img_size[3],
    const int b_I_size, const int plast, unsigned char img_data[2764800])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = 3UL * ((static_cast<unsigned long>(b_I_size) + 1UL) *
                   (static_cast<unsigned long>(plast - 1) + 1UL)) -
            1UL;
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    unsigned long tmpIndex;
    int colIndices;
    int k;
    int rowIndices;
    colIndices = static_cast<int>(idx % 3UL);
    tmpIndex = (idx - static_cast<unsigned long>(colIndices)) / 3UL;
    rowIndices = static_cast<int>(tmpIndex %
                                  (static_cast<unsigned long>(b_I_size) + 1UL));
    tmpIndex = (tmpIndex - static_cast<unsigned long>(rowIndices)) /
               (static_cast<unsigned long>(b_I_size) + 1UL);
    k = static_cast<int>(tmpIndex);
    img_data[((static_cast<int>(static_cast<short>(k + 1)) +
               img_size[0] *
                   (static_cast<int>(static_cast<short>(rowIndices + 1)) - 1)) +
              img_size[0] * img_size[1] * colIndices) -
             1] = I_data
        [(colIndices +
          3 * (static_cast<int>(static_cast<short>(rowIndices + 1)) - 1)) +
         3 * I_size[1] * (static_cast<int>(static_cast<short>(k + 1)) - 1)];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double aux1_data[1440]
//                const int in_rows
//                const double scale_idx_0
//                const double kwidthRow
//                double ipRowIndices_data[2880]
//                double rowWeights_data[2880]
// Return Type  : void
//
static __global__ __launch_bounds__(192, 1) void line_detection_kernel10(
    const double aux1_data[1440], const int in_rows, const double scale_idx_0,
    const double kwidthRow, double ipRowIndices_data[2880],
    double rowWeights_data[2880])
{
  unsigned long threadId;
  int rowIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  rowIdx = static_cast<int>(threadId);
  if (rowIdx < 180) {
    int colIndices;
    colIndices = static_cast<int>(ceil(kwidthRow));
    for (int k{0}; k < colIndices; k++) {
      double absx;
      double absx2;
      double sumVal;
      int b_y;
      int l;
      int rowIndices;
      sumVal = (static_cast<double>(rowIdx) + 1.0) / scale_idx_0 +
               0.5 * (1.0 - 1.0 / scale_idx_0);
      rowIndices = static_cast<int>(floor(sumVal - kwidthRow / 2.0));
      sumVal -= static_cast<double>(rowIndices + k) + 1.0;
      if (scale_idx_0 < 1.0) {
        sumVal *= scale_idx_0;
      }
      absx = fabs(sumVal);
      absx2 = absx * absx;
      sumVal = pow(absx, 3.0);
      sumVal = ((1.5 * sumVal - 2.5 * absx2) + 1.0) *
                   static_cast<double>(absx <= 1.0) +
               (((-0.5 * sumVal + 2.5 * absx2) - 4.0 * absx) + 2.0) *
                   static_cast<double>((static_cast<int>(1.0 < absx)) &&
                                       (static_cast<int>(absx <= 2.0)));
      if (scale_idx_0 < 1.0) {
        rowWeights_data[rowIdx + 180 * k] = scale_idx_0 * sumVal;
      } else {
        rowWeights_data[rowIdx + 180 * k] = sumVal;
      }
      b_y = in_rows << 1;
      rowIndices = (rowIndices + k) + 1;
      l = rowIndices - 1;
      if (b_y == 0) {
        if (rowIndices - 1 == 0) {
          l = 0;
        }
      } else if (rowIndices - 1 == 0) {
        l = 0;
      } else {
        l = static_cast<int>(fmod(static_cast<double>(rowIndices) - 1.0,
                                  static_cast<double>(b_y)));
        if ((static_cast<int>(l != 0)) &&
            (static_cast<int>(rowIndices - 1 < 0))) {
          l += b_y;
        }
      }
      ipRowIndices_data[rowIdx + 180 * k] = aux1_data[l];
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int plast
//                double ipColIndices_data[9280]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel11(
    const int plast, double ipColIndices_data[9280])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(320 * plast - 1);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    ipColIndices_data[oldIdx] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int plast
//                double colWeights_data[9280]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel12(
    const int plast, double colWeights_data[9280])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(320 * plast - 1);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    colWeights_data[oldIdx] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double aux2_data[2560]
//                const int in_cols
//                const double scale_idx_1
//                const double kwidthCol
//                double ipColIndices_data[9280]
//                double colWeights_data[9280]
// Return Type  : void
//
static __global__ __launch_bounds__(320, 1) void line_detection_kernel13(
    const double aux2_data[2560], const int in_cols, const double scale_idx_1,
    const double kwidthCol, double ipColIndices_data[9280],
    double colWeights_data[9280])
{
  unsigned long threadId;
  int colIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  colIdx = static_cast<int>(threadId);
  if (colIdx < 320) {
    int rowIndices;
    rowIndices = static_cast<int>(ceil(kwidthCol));
    for (int k{0}; k < rowIndices; k++) {
      double absx;
      double absx2;
      double sumVal;
      int b_y;
      int colIndices;
      int l;
      sumVal = (static_cast<double>(colIdx) + 1.0) / scale_idx_1 +
               0.5 * (1.0 - 1.0 / scale_idx_1);
      colIndices = static_cast<int>(floor(sumVal - kwidthCol / 2.0));
      sumVal -= static_cast<double>(colIndices + k) + 1.0;
      if (scale_idx_1 < 1.0) {
        sumVal *= scale_idx_1;
      }
      absx = fabs(sumVal);
      absx2 = absx * absx;
      sumVal = pow(absx, 3.0);
      sumVal = ((1.5 * sumVal - 2.5 * absx2) + 1.0) *
                   static_cast<double>(absx <= 1.0) +
               (((-0.5 * sumVal + 2.5 * absx2) - 4.0 * absx) + 2.0) *
                   static_cast<double>((static_cast<int>(1.0 < absx)) &&
                                       (static_cast<int>(absx <= 2.0)));
      if (scale_idx_1 < 1.0) {
        colWeights_data[colIdx + 320 * k] = scale_idx_1 * sumVal;
      } else {
        colWeights_data[colIdx + 320 * k] = sumVal;
      }
      b_y = in_cols << 1;
      colIndices = (colIndices + k) + 1;
      l = colIndices - 1;
      if (b_y == 0) {
        if (colIndices - 1 == 0) {
          l = 0;
        }
      } else if (colIndices - 1 == 0) {
        l = 0;
      } else {
        l = static_cast<int>(fmod(static_cast<double>(colIndices) - 1.0,
                                  static_cast<double>(b_y)));
        if ((static_cast<int>(l != 0)) &&
            (static_cast<int>(colIndices - 1 < 0))) {
          l += b_y;
        }
      }
      ipColIndices_data[colIdx + 320 * k] = aux2_data[l];
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double rowWeights_data[2880]
//                double rowWeightsTotal[180]
// Return Type  : void
//
static __global__ __launch_bounds__(192, 1) void line_detection_kernel14(
    const double rowWeights_data[2880], double rowWeightsTotal[180])
{
  unsigned long threadId;
  int colIndices;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  colIndices = static_cast<int>(threadId);
  if (colIndices < 180) {
    rowWeightsTotal[colIndices] = rowWeights_data[colIndices];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double rowWeights_data[2880]
//                const int in_rows
//                double rowWeightsTotal[180]
// Return Type  : void
//
static __global__ __launch_bounds__(192, 1) void line_detection_kernel15(
    const double rowWeights_data[2880], const int in_rows,
    double rowWeightsTotal[180])
{
  unsigned long threadId;
  int colIndices;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  colIndices = static_cast<int>(threadId);
  if (colIndices < 180) {
    rowWeightsTotal[colIndices] += rowWeights_data[in_rows + colIndices];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double colWeights_data[9280]
//                double colWeightsTotal[320]
// Return Type  : void
//
static __global__ __launch_bounds__(320, 1) void line_detection_kernel16(
    const double colWeights_data[9280], double colWeightsTotal[320])
{
  unsigned long threadId;
  int colIndices;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  colIndices = static_cast<int>(threadId);
  if (colIndices < 320) {
    colWeightsTotal[colIndices] = colWeights_data[colIndices];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double colWeights_data[9280]
//                const int in_rows
//                double colWeightsTotal[320]
// Return Type  : void
//
static __global__ __launch_bounds__(320, 1) void line_detection_kernel17(
    const double colWeights_data[9280], const int in_rows,
    double colWeightsTotal[320])
{
  unsigned long threadId;
  int colIndices;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  colIndices = static_cast<int>(threadId);
  if (colIndices < 320) {
    colWeightsTotal[colIndices] += colWeights_data[in_rows + colIndices];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int img_size[3]
//                int partialResize_size[3]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void line_detection_kernel18(
    const int img_size[3], int partialResize_size[3])
{
  unsigned long threadId;
  int tmpIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    partialResize_size[0] = img_size[0];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double colWeightsTotal[320]
//                const double colWeights_data[9280]
//                const unsigned char img_data[2764800]
//                const double ipColIndices_data[9280]
//                const int img_size[3]
//                const int partialResize_size[3]
//                const double kwidthCol
//                const int plast
//                unsigned char partialResize_data[691200]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel19(
    const double colWeightsTotal[320], const double colWeights_data[9280],
    const unsigned char img_data[2764800], const double ipColIndices_data[9280],
    const int img_size[3], const int partialResize_size[3],
    const double kwidthCol, const int plast,
    unsigned char partialResize_data[691200])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = 3UL * (320UL * (static_cast<unsigned long>(plast - 1) + 1UL)) - 1UL;
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    double sumVal;
    unsigned long tmpIndex;
    int colIdx;
    int colIndices;
    int rowIdx;
    int rowIndices;
    unsigned char u;
    rowIndices = static_cast<int>(idx % 3UL);
    tmpIndex = (idx - static_cast<unsigned long>(rowIndices)) / 3UL;
    colIdx = static_cast<int>(tmpIndex % 320UL);
    tmpIndex = (tmpIndex - static_cast<unsigned long>(colIdx)) / 320UL;
    rowIdx = static_cast<int>(tmpIndex);
    sumVal = 0.0;
    colIndices = static_cast<int>(ceil(kwidthCol));
    for (int l{0}; l < colIndices; l++) {
      sumVal +=
          static_cast<double>(
              img_data[(rowIdx + img_size[0] *
                                     (static_cast<int>(
                                          ipColIndices_data[colIdx + 320 * l]) -
                                      1)) +
                       img_size[0] * img_size[1] * rowIndices]) *
          (colWeights_data[colIdx + 320 * l] / colWeightsTotal[colIdx]);
    }
    sumVal = round(sumVal);
    if (sumVal < 256.0) {
      if (sumVal >= 0.0) {
        u = static_cast<unsigned char>(sumVal);
      } else {
        u = static_cast<unsigned char>(0U);
      }
    } else if (sumVal >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }
    partialResize_data[(rowIdx + partialResize_size[0] * colIdx) +
                       partialResize_size[0] * 320 * rowIndices] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char I_data[]
//                const int I_size
//                unsigned char img_data[2764800]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel2(
    const unsigned char I_data[], const int I_size,
    unsigned char img_data[2764800])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(I_size);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    img_data[oldIdx] = I_data[oldIdx];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double rowWeightsTotal[180]
//                const double rowWeights_data[2880]
//                const unsigned char partialResize_data[691200]
//                const int partialResize_size[3]
//                const double ipRowIndices_data[2880]
//                const double kwidthRow
//                unsigned char out[172800]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void line_detection_kernel20(
    const double rowWeightsTotal[180], const double rowWeights_data[2880],
    const unsigned char partialResize_data[691200],
    const int partialResize_size[3], const double ipRowIndices_data[2880],
    const double kwidthRow, unsigned char out[172800])
{
  unsigned long threadId;
  int colIdx;
  int rowIdx;
  int rowIndices;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  rowIndices = static_cast<int>(threadId % 3UL);
  threadId = (threadId - static_cast<unsigned long>(rowIndices)) / 3UL;
  rowIdx = static_cast<int>(threadId % 180UL);
  threadId = (threadId - static_cast<unsigned long>(rowIdx)) / 180UL;
  colIdx = static_cast<int>(threadId);
  if ((static_cast<int>((static_cast<int>(colIdx < 320)) &&
                        (static_cast<int>(rowIdx < 180)))) &&
      (static_cast<int>(rowIndices < 3))) {
    double sumVal;
    int colIndices;
    unsigned char u;
    sumVal = 0.0;
    colIndices = static_cast<int>(ceil(kwidthRow));
    for (int l{0}; l < colIndices; l++) {
      sumVal +=
          static_cast<double>(
              partialResize_data[((static_cast<int>(
                                       ipRowIndices_data[rowIdx + 180 * l]) +
                                   partialResize_size[0] * colIdx) +
                                  partialResize_size[0] * 320 * rowIndices) -
                                 1]) *
          (rowWeights_data[rowIdx + 180 * l] / rowWeightsTotal[rowIdx]);
    }
    sumVal = round(sumVal);
    if (sumVal < 256.0) {
      if (sumVal >= 0.0) {
        u = static_cast<unsigned char>(sumVal);
      } else {
        u = static_cast<unsigned char>(0U);
      }
    } else if (sumVal >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }
    out[(rowIdx + 180 * colIdx) + 57600 * rowIndices] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int img_size[3]
//                int partialResize_size[3]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void line_detection_kernel21(
    const int img_size[3], int partialResize_size[3])
{
  unsigned long threadId;
  int tmpIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    partialResize_size[1] = img_size[1];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double rowWeightsTotal[180]
//                const double rowWeights_data[2880]
//                const unsigned char img_data[2764800]
//                const int img_size[3]
//                const double ipRowIndices_data[2880]
//                const int partialResize_size[3]
//                const double kwidthRow
//                const int plast
//                unsigned char partialResize_data[691200]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel22(
    const double rowWeightsTotal[180], const double rowWeights_data[2880],
    const unsigned char img_data[2764800], const int img_size[3],
    const double ipRowIndices_data[2880], const int partialResize_size[3],
    const double kwidthRow, const int plast,
    unsigned char partialResize_data[691200])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = 3UL * (180UL * (static_cast<unsigned long>(plast - 1) + 1UL)) - 1UL;
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    double sumVal;
    unsigned long tmpIndex;
    int colIdx;
    int colIndices;
    int rowIdx;
    int rowIndices;
    unsigned char u;
    rowIndices = static_cast<int>(idx % 3UL);
    tmpIndex = (idx - static_cast<unsigned long>(rowIndices)) / 3UL;
    rowIdx = static_cast<int>(tmpIndex % 180UL);
    tmpIndex = (tmpIndex - static_cast<unsigned long>(rowIdx)) / 180UL;
    colIdx = static_cast<int>(tmpIndex);
    sumVal = 0.0;
    colIndices = static_cast<int>(ceil(kwidthRow));
    for (int l{0}; l < colIndices; l++) {
      sumVal +=
          static_cast<double>(
              img_data[((static_cast<int>(ipRowIndices_data[rowIdx + 180 * l]) +
                         img_size[0] * colIdx) +
                        img_size[0] * img_size[1] * rowIndices) -
                       1]) *
          (rowWeights_data[rowIdx + 180 * l] / rowWeightsTotal[rowIdx]);
    }
    sumVal = round(sumVal);
    if (sumVal < 256.0) {
      if (sumVal >= 0.0) {
        u = static_cast<unsigned char>(sumVal);
      } else {
        u = static_cast<unsigned char>(0U);
      }
    } else if (sumVal >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }
    partialResize_data[(rowIdx + 180 * colIdx) +
                       180 * partialResize_size[1] * rowIndices] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double colWeightsTotal[320]
//                const double colWeights_data[9280]
//                const unsigned char partialResize_data[691200]
//                const int partialResize_size[3]
//                const double ipColIndices_data[9280]
//                const double kwidthCol
//                unsigned char out[172800]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void line_detection_kernel23(
    const double colWeightsTotal[320], const double colWeights_data[9280],
    const unsigned char partialResize_data[691200],
    const int partialResize_size[3], const double ipColIndices_data[9280],
    const double kwidthCol, unsigned char out[172800])
{
  unsigned long threadId;
  int colIdx;
  int rowIdx;
  int rowIndices;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  rowIndices = static_cast<int>(threadId % 3UL);
  threadId = (threadId - static_cast<unsigned long>(rowIndices)) / 3UL;
  rowIdx = static_cast<int>(threadId % 180UL);
  threadId = (threadId - static_cast<unsigned long>(rowIdx)) / 180UL;
  colIdx = static_cast<int>(threadId);
  if ((static_cast<int>((static_cast<int>(colIdx < 320)) &&
                        (static_cast<int>(rowIdx < 180)))) &&
      (static_cast<int>(rowIndices < 3))) {
    double sumVal;
    int colIndices;
    unsigned char u;
    sumVal = 0.0;
    colIndices = static_cast<int>(ceil(kwidthCol));
    for (int l{0}; l < colIndices; l++) {
      sumVal +=
          static_cast<double>(
              partialResize_data
                  [(rowIdx + 180 * (static_cast<int>(
                                        ipColIndices_data[colIdx + 320 * l]) -
                                    1)) +
                   180 * partialResize_size[1] * rowIndices]) *
          (colWeights_data[colIdx + 320 * l] / colWeightsTotal[colIdx]);
    }
    sumVal = round(sumVal);
    if (sumVal < 256.0) {
      if (sumVal >= 0.0) {
        u = static_cast<unsigned char>(sumVal);
      } else {
        u = static_cast<unsigned char>(0U);
      }
    } else if (sumVal >= 256.0) {
      u = MAX_uint8_T;
    } else {
      u = static_cast<unsigned char>(0U);
    }
    out[(rowIdx + 180 * colIdx) + 57600 * rowIndices] = u;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const short sz[3]
//                int positionOut_data[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void line_detection_kernel24(
    const short sz[3], int positionOut_data[4])
{
  unsigned long threadId;
  int tmpIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    positionOut_data[0] = static_cast<int>(
        roundf(static_cast<float>(static_cast<double>(sz[1]) / 2.0)));
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const short sz[3]
//                int positionOut_data[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void line_detection_kernel25(
    const short sz[3], int positionOut_data[4])
{
  unsigned long threadId;
  int tmpIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    positionOut_data[1] = static_cast<int>(sz[0]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int plast
//                const float predictedPosNorm[2]
//                int positionOut_data[4]
//                float predictedPos[2]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void line_detection_kernel26(
    const int plast, const float predictedPosNorm[2], int positionOut_data[4],
    float predictedPos[2])
{
  unsigned long threadId;
  int oldIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 2) {
    float f;
    int rowIndices;
    f = (predictedPosNorm[oldIdx] + 1.0F) / 2.0F * static_cast<float>(plast);
    predictedPos[oldIdx] = f;
    f = roundf(f);
    if (f < 2.14748365E+9F) {
      if (f >= -2.14748365E+9F) {
        rowIndices = static_cast<int>(f);
      } else {
        rowIndices = MIN_int32_T;
      }
    } else if (f >= 2.14748365E+9F) {
      rowIndices = MAX_int32_T;
    } else {
      rowIndices = 0;
    }
    positionOut_data[oldIdx + 2] = rowIndices;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int img_size[3]
//                short sz[3]
// Return Type  : void
//
static __global__
    __launch_bounds__(32, 1) void line_detection_kernel27(const int img_size[3],
                                                          short sz[3])
{
  unsigned long threadId;
  int oldIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 3) {
    sz[oldIdx] = static_cast<short>(img_size[oldIdx]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int sz
//                unsigned char img_data[2764800]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel28(
    const int sz, unsigned char img_data[2764800])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(sz);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    img_data[oldIdx] = static_cast<unsigned char>(0U);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const short pixCountSize
//                unsigned char pixCount_data[1280]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel29(
    const short pixCountSize, unsigned char pixCount_data[1280])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(static_cast<int>(pixCountSize) - 1);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    pixCount_data[oldIdx] = static_cast<unsigned char>(0U);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char img_data[2764800]
//                const int img_size[3]
//                const int b_img_size[3]
//                const int plast
//                const int in_rows
//                unsigned char b_img_data[2764800]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel3(
    const unsigned char img_data[2764800], const int img_size[3],
    const int b_img_size[3], const int plast, const int in_rows,
    unsigned char b_img_data[2764800])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<unsigned long>(plast) + 1UL) *
                ((static_cast<unsigned long>(in_rows) + 1UL) * 3UL) -
            1UL;
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    unsigned long tmpIndex;
    int colIndices;
    int oldIdx;
    int rowIndices;
    rowIndices =
        static_cast<int>(idx % (static_cast<unsigned long>(plast) + 1UL));
    tmpIndex = (idx - static_cast<unsigned long>(rowIndices)) /
               (static_cast<unsigned long>(plast) + 1UL);
    colIndices = static_cast<int>(tmpIndex %
                                  (static_cast<unsigned long>(in_rows) + 1UL));
    tmpIndex = (tmpIndex - static_cast<unsigned long>(colIndices)) /
               (static_cast<unsigned long>(in_rows) + 1UL);
    oldIdx = static_cast<int>(tmpIndex);
    b_img_data[(rowIndices + b_img_size[0] * colIndices) +
               b_img_size[0] * b_img_size[1] * oldIdx] =
        img_data[(rowIndices + img_size[0] * colIndices) +
                 img_size[0] * img_size[1] * (2 - oldIdx)];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char color[3]
//                unsigned char b_color[3]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void line_detection_kernel30(
    const unsigned char color[3], unsigned char b_color[3])
{
  unsigned long threadId;
  int oldIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 3) {
    b_color[oldIdx] = color[oldIdx];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float predictedPos[2]
//                int positionOut[3]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void line_detection_kernel31(
    const float predictedPos[2], int positionOut[3])
{
  unsigned long threadId;
  int oldIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 2) {
    float f;
    int rowIndices;
    f = roundf(predictedPos[oldIdx]);
    if (f < 2.14748365E+9F) {
      if (f >= -2.14748365E+9F) {
        rowIndices = static_cast<int>(f);
      } else {
        rowIndices = MIN_int32_T;
      }
    } else if (f >= 2.14748365E+9F) {
      rowIndices = MAX_int32_T;
    } else {
      rowIndices = 0;
    }
    positionOut[oldIdx] = rowIndices;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                int positionOut[3]
// Return Type  : void
//
static __global__
    __launch_bounds__(32, 1) void line_detection_kernel32(int positionOut[3])
{
  unsigned long threadId;
  int tmpIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    positionOut[2] = 5;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const short tmpRGB_size[3]
//                short sz[3]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void line_detection_kernel33(
    const short tmpRGB_size[3], short sz[3])
{
  unsigned long threadId;
  int oldIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 3) {
    sz[oldIdx] = tmpRGB_size[oldIdx];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int sz
//                unsigned char img_data[2764800]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel34(
    const int sz, unsigned char img_data[2764800])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(sz);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    img_data[oldIdx] = static_cast<unsigned char>(0U);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const short pixCountSize
//                unsigned char pixCount_data[1280]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel35(
    const short pixCountSize, unsigned char pixCount_data[1280])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(static_cast<int>(pixCountSize) - 1);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    pixCount_data[oldIdx] = static_cast<unsigned char>(0U);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char color[3]
//                unsigned char b_color[3]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void line_detection_kernel36(
    const unsigned char color[3], unsigned char b_color[3])
{
  unsigned long threadId;
  int oldIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 3) {
    b_color[oldIdx] = color[oldIdx];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char img_data[2764800]
//                const short out_size[3]
//                const int Iout_size[3]
//                const int b_out_size
//                const int c_out_size
//                unsigned char Iout_data[]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel37(
    const unsigned char img_data[2764800], const short out_size[3],
    const int Iout_size[3], const int b_out_size, const int c_out_size,
    unsigned char Iout_data[])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<unsigned long>(b_out_size) + 1UL) *
                ((static_cast<unsigned long>(c_out_size) + 1UL) * 3UL) -
            1UL;
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    unsigned long tmpIndex;
    int colIndices;
    int k;
    int rowIndices;
    colIndices =
        static_cast<int>(idx % (static_cast<unsigned long>(b_out_size) + 1UL));
    tmpIndex = (idx - static_cast<unsigned long>(colIndices)) /
               (static_cast<unsigned long>(b_out_size) + 1UL);
    rowIndices = static_cast<int>(
        tmpIndex % (static_cast<unsigned long>(c_out_size) + 1UL));
    tmpIndex = (tmpIndex - static_cast<unsigned long>(rowIndices)) /
               (static_cast<unsigned long>(c_out_size) + 1UL);
    k = static_cast<int>(tmpIndex);
    Iout_data[(k + 3 * rowIndices) + 3 * Iout_size[1] * colIndices] =
        img_data[(colIndices + static_cast<int>(out_size[0]) * rowIndices) +
                 static_cast<int>(out_size[0]) * static_cast<int>(out_size[1]) *
                     (2 - k)];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char img_data[2764800]
//                const short out_size[3]
//                const short b_out_size[3]
//                const int plast
//                const int in_rows
//                unsigned char b_img_data[2764800]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel38(
    const unsigned char img_data[2764800], const short out_size[3],
    const short b_out_size[3], const int plast, const int in_rows,
    unsigned char b_img_data[2764800])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<unsigned long>(plast) + 1UL) *
                ((static_cast<unsigned long>(in_rows) + 1UL) * 3UL) -
            1UL;
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    unsigned long tmpIndex;
    int colIndices;
    int oldIdx;
    int rowIndices;
    rowIndices =
        static_cast<int>(idx % (static_cast<unsigned long>(plast) + 1UL));
    tmpIndex = (idx - static_cast<unsigned long>(rowIndices)) /
               (static_cast<unsigned long>(plast) + 1UL);
    colIndices = static_cast<int>(tmpIndex %
                                  (static_cast<unsigned long>(in_rows) + 1UL));
    tmpIndex = (tmpIndex - static_cast<unsigned long>(colIndices)) /
               (static_cast<unsigned long>(in_rows) + 1UL);
    oldIdx = static_cast<int>(tmpIndex);
    b_img_data[(rowIndices + static_cast<int>(b_out_size[0]) * colIndices) +
               static_cast<int>(b_out_size[0]) *
                   static_cast<int>(b_out_size[1]) * oldIdx] =
        img_data[(rowIndices + static_cast<int>(out_size[0]) * colIndices) +
                 static_cast<int>(out_size[0]) * static_cast<int>(out_size[1]) *
                     (2 - oldIdx)];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char img_data[2764800]
//                const int tmpRGB_size
//                unsigned char Iout_data[]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel39(
    const unsigned char img_data[2764800], const int tmpRGB_size,
    unsigned char Iout_data[])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(tmpRGB_size);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    Iout_data[oldIdx] = img_data[oldIdx];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const unsigned char img_data[2764800]
//                const int img_size
//                unsigned char b_img_data[2764800]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel4(
    const unsigned char img_data[2764800], const int img_size,
    unsigned char b_img_data[2764800])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(img_size);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    b_img_data[oldIdx] = img_data[oldIdx];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int img_size[3]
//                short sz[3]
// Return Type  : void
//
static __global__
    __launch_bounds__(32, 1) void line_detection_kernel5(const int img_size[3],
                                                         short sz[3])
{
  unsigned long threadId;
  int oldIdx;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  oldIdx = static_cast<int>(threadId);
  if (oldIdx < 3) {
    // img = imrotate(img,180);
    sz[oldIdx] = static_cast<short>(img_size[oldIdx]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int in_rows
//                const int plast
//                double aux1_data[1440]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel6(
    const int in_rows, const int plast, double aux1_data[1440])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(plast - 1);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int colIndices;
    colIndices = static_cast<int>(idx);
    if (colIndices + 1 <= in_rows) {
      aux1_data[colIndices] = static_cast<double>(colIndices) + 1.0;
    } else {
      aux1_data[colIndices] =
          (static_cast<double>(plast - colIndices) - 1.0) + 1.0;
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int in_cols
//                const int plast
//                double aux2_data[2560]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel7(
    const int in_cols, const int plast, double aux2_data[2560])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(plast - 1);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int colIndices;
    colIndices = static_cast<int>(idx);
    if (colIndices + 1 <= in_cols) {
      aux2_data[colIndices] = static_cast<double>(colIndices) + 1.0;
    } else {
      aux2_data[colIndices] =
          (static_cast<double>(plast - colIndices) - 1.0) + 1.0;
    }
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int plast
//                double ipRowIndices_data[2880]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel8(
    const int plast, double ipRowIndices_data[2880])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(180 * plast - 1);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    ipRowIndices_data[oldIdx] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const int plast
//                double rowWeights_data[2880]
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void line_detection_kernel9(
    const int plast, double rowWeights_data[2880])
{
  unsigned long loopEnd;
  unsigned long threadId;
  unsigned long threadStride;
  threadId = static_cast<unsigned long>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<unsigned long>(180 * plast - 1);
  for (unsigned long idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int oldIdx;
    oldIdx = static_cast<int>(idx);
    rowWeights_data[oldIdx] = 0.0;
  }
}

//
// Copyright 2020 The MathWorks, Inc.
//
// Arguments    : const unsigned char I_data[]
//                const int I_size[3]
//                float predictedPosNorm[2]
//                unsigned char Iout_data[]
//                int Iout_size[3]
// Return Type  : void
//
void line_detection(const unsigned char I_data[], const int I_size[3],
                    float predictedPosNorm[2], unsigned char Iout_data[],
                    int Iout_size[3])
{
  static const unsigned char b_color[3]{MAX_uint8_T, 0U, 0U};
  static unsigned char b_img_data[2764800];
  static unsigned char img_data[2764800];
  void *b_colPtr;
  void *b_posPtr;
  void *b_ptrObj;
  void *colPtr;
  void *posPtr;
  void *ptrObj;
  dim3 block;
  dim3 grid;
  double(*gpu_colWeights_data)[9280];
  double(*gpu_ipColIndices_data)[9280];
  double(*gpu_ipRowIndices_data)[2880];
  double(*gpu_rowWeights_data)[2880];
  double(*gpu_aux2_data)[2560];
  double(*gpu_aux1_data)[1440];
  double(*gpu_colWeightsTotal)[320];
  double(*gpu_rowWeightsTotal)[180];
  double kwidthCol;
  double kwidthRow;
  double scale_idx_0;
  double scale_idx_1;
  float(*gpu_predictedPos)[2];
  float(*gpu_predictedPosNorm)[2];
  int positionOut_data[4];
  int(*gpu_positionOut_data)[4];
  int b_img_size[3];
  int img_size[3];
  int positionOut[3];
  int(*b_gpu_img_size)[3];
  int(*gpu_I_size)[3];
  int(*gpu_Iout_size)[3];
  int(*gpu_img_size)[3];
  int(*gpu_partialResize_size)[3];
  int(*gpu_positionOut)[3];
  int colWeights_size[2];
  int rowWeights_size[2];
  int in_cols;
  int plast;
  short b_out_size[3];
  short out_size[3];
  short sz[3];
  short tmpRGB_size[3];
  short(*b_gpu_out_size)[3];
  short(*gpu_out_size)[3];
  short(*gpu_sz)[3];
  short(*gpu_tmpRGB_size)[3];
  short numCol;
  short numRow;
  short pixCountSize;
  unsigned char(*b_gpu_img_data)[2764800];
  unsigned char(*gpu_img_data)[2764800];
  unsigned char(*gpu_partialResize_data)[691200];
  unsigned char out[172800];
  unsigned char(*gpu_out)[172800];
  unsigned char pixCount_data[1280];
  unsigned char(*gpu_pixCount_data)[1280];
  unsigned char color[3];
  unsigned char(*b_gpu_color)[3];
  unsigned char(*gpu_color)[3];
  unsigned char *gpu_I_data;
  unsigned char *gpu_Iout_data;
  bool b_img_data_dirtyOnCpu;
  bool b_img_data_dirtyOnGpu;
  bool exitg1;
  bool img_data_dirtyOnCpu;
  bool img_data_dirtyOnGpu;
  bool isInitialise;
  bool pixCount_data_dirtyOnCpu;
  bool pixCount_data_dirtyOnGpu;
  bool validLaunchParams;
  if (!isInitialized_line_detection) {
    line_detection_initialize();
  }
  hipMalloc(&gpu_Iout_data, 2764800U * sizeof(unsigned char));
  hipMalloc(&gpu_Iout_size, 12UL);
  hipMalloc(&b_gpu_out_size, 6UL);
  hipMalloc(&gpu_out_size, 6UL);
  hipMalloc(&gpu_tmpRGB_size, 6UL);
  hipMalloc(&gpu_positionOut, 12UL);
  hipMalloc(&b_gpu_color, 3UL);
  hipMalloc(&gpu_color, 3UL);
  hipMalloc(&gpu_pixCount_data, 1280UL);
  hipMalloc(&gpu_predictedPos, 8UL);
  hipMalloc(&gpu_predictedPosNorm, 8UL);
  hipMalloc(&gpu_positionOut_data, 16UL);
  hipMalloc(&gpu_out, 172800UL);
  hipMalloc(&gpu_partialResize_data, 691200UL);
  hipMalloc(&gpu_partialResize_size, 12UL);
  hipMalloc(&gpu_colWeightsTotal, 2560UL);
  hipMalloc(&gpu_rowWeightsTotal, 1440UL);
  hipMalloc(&gpu_colWeights_data, 74240UL);
  hipMalloc(&gpu_ipColIndices_data, 74240UL);
  hipMalloc(&gpu_rowWeights_data, 23040UL);
  hipMalloc(&gpu_ipRowIndices_data, 23040UL);
  hipMalloc(&gpu_aux2_data, 20480UL);
  hipMalloc(&gpu_aux1_data, 11520UL);
  hipMalloc(&gpu_sz, 6UL);
  hipMalloc(&b_gpu_img_data, 2764800UL);
  hipMalloc(&b_gpu_img_size, 12UL);
  hipMalloc(&gpu_img_size, 12UL);
  hipMalloc(&gpu_I_size, 12UL);
  hipMalloc(&gpu_img_data, 2764800UL);
  hipMalloc(&gpu_I_data, 2764800U * sizeof(unsigned char));
  pixCount_data_dirtyOnGpu = false;
  b_img_data_dirtyOnGpu = false;
  img_data_dirtyOnGpu = false;
  pixCount_data_dirtyOnCpu = false;
  b_img_data_dirtyOnCpu = false;
  img_data_dirtyOnCpu = false;
  if (!lineDetObj_not_empty) {
    coder::DeepLearningNetwork_setup(&lineDetObj);
    lineDetObj_not_empty = true;
  }
  //  Convert image data format OpenCV BGR to MATLAB compatible RGB image format
  //  Convert image data format OpenCV BGR to MATLAB compatible RGB image format
  //  Copyright 2020 The MathWorks, Inc.
  isInitialise = true;
  if ((I_size[1] != 0) && (I_size[2] != 0)) {
    plast = 0;
    in_cols = 1;
    exitg1 = false;
    while ((!exitg1) && (in_cols < 4)) {
      if (I_size[3 - in_cols] != 1) {
        if (plast > 4 - in_cols) {
          isInitialise = false;
          exitg1 = true;
        } else {
          plast = 4 - in_cols;
          in_cols++;
        }
      } else {
        in_cols++;
      }
    }
  }
  if (isInitialise) {
    img_size[0] = I_size[2];
    img_size[1] = I_size[1];
    isInitialise = true;
    in_cols = I_size[1] * I_size[2] * 3 - 1;
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<double>(in_cols + 1L), &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      hipMemcpy(gpu_I_data, I_data,
                 3 * (I_size[1] * I_size[2]) * sizeof(unsigned char),
                 hipMemcpyHostToDevice);
      line_detection_kernel2<<<grid, block>>>(gpu_I_data, in_cols,
                                              *gpu_img_data);
      img_data_dirtyOnGpu = true;
    }
  } else {
    img_size[0] = I_size[2];
    img_size[1] = I_size[1];
    isInitialise = true;
    plast = I_size[2];
    in_cols = I_size[1] - 1;
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<double>(3L *
                            (((I_size[1] - 1) + 1L) * ((I_size[2] - 1) + 1L))),
        &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      hipMemcpy(gpu_I_data, I_data,
                 3 * (I_size[1] * I_size[2]) * sizeof(unsigned char),
                 hipMemcpyHostToDevice);
      hipMemcpy(*gpu_I_size, I_size, 12UL, hipMemcpyHostToDevice);
      hipMemcpy(*gpu_img_size, img_size, 12UL, hipMemcpyHostToDevice);
      isInitialise = false;
      line_detection_kernel1<<<grid, block>>>(gpu_I_data, *gpu_I_size,
                                              *gpu_img_size, in_cols, plast,
                                              *gpu_img_data);
      img_data_dirtyOnGpu = true;
    }
  }
  plast = img_size[0] - 1;
  in_cols = img_size[1] - 1;
  b_img_size[0] = img_size[0];
  b_img_size[1] = img_size[1];
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<double>(((img_size[0] - 1) + 1L) *
                          (((img_size[1] - 1) + 1L) * 3L)),
      &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    if (isInitialise) {
      hipMemcpy(*gpu_img_size, img_size, 12UL, hipMemcpyHostToDevice);
    }
    hipMemcpy(*b_gpu_img_size, b_img_size, 12UL, hipMemcpyHostToDevice);
    line_detection_kernel3<<<grid, block>>>(*gpu_img_data, *gpu_img_size,
                                            *b_gpu_img_size, plast, in_cols,
                                            *b_gpu_img_data);
    b_img_data_dirtyOnGpu = true;
  }
  img_size[0] = b_img_size[0];
  img_size[1] = b_img_size[1];
  img_size[2] = 3;
  in_cols = b_img_size[0] * b_img_size[1] * 3 - 1;
  validLaunchParams = mwGetLaunchParameters1D(static_cast<double>(in_cols + 1L),
                                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    line_detection_kernel4<<<grid, block>>>(*b_gpu_img_data, in_cols,
                                            *gpu_img_data);
    img_data_dirtyOnGpu = true;
  }
  // img = imrotate(img,180);
  hipMemcpy(*gpu_img_size, img_size, 12UL, hipMemcpyHostToDevice);
  line_detection_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_img_size,
                                                                  *gpu_sz);
  //  Resize
  scale_idx_0 = 180.0 / static_cast<double>(img_size[0]);
  scale_idx_1 = 320.0 / static_cast<double>(img_size[1]);
  plast = img_size[0] << 1;
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<double>((plast - 1) + 1L), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    line_detection_kernel6<<<grid, block>>>(img_size[0], plast, *gpu_aux1_data);
  }
  plast = img_size[1] << 1;
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<double>((plast - 1) + 1L), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    line_detection_kernel7<<<grid, block>>>(img_size[1], plast, *gpu_aux2_data);
  }
  if (scale_idx_0 < 1.0) {
    kwidthRow = 4.0 / scale_idx_0;
  } else {
    kwidthRow = 4.0;
  }
  plast = static_cast<int>(std::ceil(kwidthRow));
  validLaunchParams =
      mwGetLaunchParameters1D(static_cast<double>((180 * plast - 1) + 1L),
                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    line_detection_kernel8<<<grid, block>>>(plast, *gpu_ipRowIndices_data);
  }
  plast = static_cast<int>(std::ceil(kwidthRow));
  rowWeights_size[1] = plast;
  validLaunchParams =
      mwGetLaunchParameters1D(static_cast<double>((180 * plast - 1) + 1L),
                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    line_detection_kernel9<<<grid, block>>>(plast, *gpu_rowWeights_data);
  }
  line_detection_kernel10<<<dim3(1U, 1U, 1U), dim3(192U, 1U, 1U)>>>(
      *gpu_aux1_data, img_size[0], scale_idx_0, kwidthRow,
      *gpu_ipRowIndices_data, *gpu_rowWeights_data);
  if (scale_idx_1 < 1.0) {
    kwidthCol = 4.0 / scale_idx_1;
  } else {
    kwidthCol = 4.0;
  }
  plast = static_cast<int>(std::ceil(kwidthCol));
  validLaunchParams =
      mwGetLaunchParameters1D(static_cast<double>((320 * plast - 1) + 1L),
                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    line_detection_kernel11<<<grid, block>>>(plast, *gpu_ipColIndices_data);
  }
  plast = static_cast<int>(std::ceil(kwidthCol));
  colWeights_size[1] = plast;
  validLaunchParams =
      mwGetLaunchParameters1D(static_cast<double>((320 * plast - 1) + 1L),
                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    line_detection_kernel12<<<grid, block>>>(plast, *gpu_colWeights_data);
  }
  line_detection_kernel13<<<dim3(1U, 1U, 1U), dim3(320U, 1U, 1U)>>>(
      *gpu_aux2_data, img_size[1], scale_idx_1, kwidthCol,
      *gpu_ipColIndices_data, *gpu_colWeights_data);
  plast = rowWeights_size[1];
  line_detection_kernel14<<<dim3(1U, 1U, 1U), dim3(192U, 1U, 1U)>>>(
      *gpu_rowWeights_data, *gpu_rowWeightsTotal);
  for (in_cols = 0; in_cols <= plast - 2; in_cols++) {
    line_detection_kernel15<<<dim3(1U, 1U, 1U), dim3(192U, 1U, 1U)>>>(
        *gpu_rowWeights_data, (in_cols + 1) * 180, *gpu_rowWeightsTotal);
  }
  plast = colWeights_size[1];
  line_detection_kernel16<<<dim3(1U, 1U, 1U), dim3(320U, 1U, 1U)>>>(
      *gpu_colWeights_data, *gpu_colWeightsTotal);
  for (in_cols = 0; in_cols <= plast - 2; in_cols++) {
    line_detection_kernel17<<<dim3(1U, 1U, 1U), dim3(320U, 1U, 1U)>>>(
        *gpu_colWeights_data, (in_cols + 1) * 320, *gpu_colWeightsTotal);
  }
  if (!(scale_idx_0 > scale_idx_1)) {
    line_detection_kernel21<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        *gpu_img_size, *gpu_partialResize_size);
    plast = img_size[1];
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<double>(3L * (180L * ((img_size[1] - 1) + 1L))), &grid,
        &block, 1024U, 65535U);
    if (validLaunchParams) {
      line_detection_kernel22<<<grid, block>>>(
          *gpu_rowWeightsTotal, *gpu_rowWeights_data, *gpu_img_data,
          *gpu_img_size, *gpu_ipRowIndices_data, *gpu_partialResize_size,
          kwidthRow, plast, *gpu_partialResize_data);
    }
    line_detection_kernel23<<<dim3(338U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_colWeightsTotal, *gpu_colWeights_data, *gpu_partialResize_data,
        *gpu_partialResize_size, *gpu_ipColIndices_data, kwidthCol, *gpu_out);
  } else {
    line_detection_kernel18<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        *gpu_img_size, *gpu_partialResize_size);
    plast = img_size[0];
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<double>(3L * (320L * ((img_size[0] - 1) + 1L))), &grid,
        &block, 1024U, 65535U);
    if (validLaunchParams) {
      line_detection_kernel19<<<grid, block>>>(
          *gpu_colWeightsTotal, *gpu_colWeights_data, *gpu_img_data,
          *gpu_ipColIndices_data, *gpu_img_size, *gpu_partialResize_size,
          kwidthCol, plast, *gpu_partialResize_data);
    }
    line_detection_kernel20<<<dim3(338U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_rowWeightsTotal, *gpu_rowWeights_data, *gpu_partialResize_data,
        *gpu_partialResize_size, *gpu_ipRowIndices_data, kwidthRow, *gpu_out);
  }
  //  pass in input
  hipMemcpy(out, *gpu_out, 172800UL, hipMemcpyDeviceToHost);
  coder::DeepLearningNetwork_predict(&lineDetObj, out, predictedPosNorm);
  hipMemcpy(sz, *gpu_sz, 6UL, hipMemcpyDeviceToHost);
  //  Visualize
  line_detection_kernel24<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      *gpu_sz, *gpu_positionOut_data);
  line_detection_kernel25<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      *gpu_sz, *gpu_positionOut_data);
  hipMemcpy(*gpu_predictedPosNorm, predictedPosNorm, 8UL,
             hipMemcpyHostToDevice);
  line_detection_kernel26<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      static_cast<int>(sz[1]), *gpu_predictedPosNorm, *gpu_positionOut_data,
      *gpu_predictedPos);
  numRow = static_cast<short>(img_size[0]);
  numCol = static_cast<short>(img_size[1]);
  line_detection_kernel27<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      *gpu_img_size, *gpu_sz);
  hipMemcpy(sz, *gpu_sz, 6UL, hipMemcpyDeviceToHost);
  tmpRGB_size[0] = sz[0];
  tmpRGB_size[1] = sz[1];
  tmpRGB_size[2] = 3;
  in_cols = sz[0] * sz[1] * 3 - 1;
  validLaunchParams = mwGetLaunchParameters1D(static_cast<double>(in_cols + 1L),
                                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    line_detection_kernel28<<<grid, block>>>(in_cols, *b_gpu_img_data);
    b_img_data_dirtyOnGpu = true;
  }
  if (img_size[0] > img_size[1]) {
    pixCountSize = static_cast<short>(img_size[0]);
  } else {
    pixCountSize = static_cast<short>(img_size[1]);
  }
  validLaunchParams =
      mwGetLaunchParameters1D(static_cast<double>((pixCountSize - 1) + 1L),
                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    line_detection_kernel29<<<grid, block>>>(pixCountSize, *gpu_pixCount_data);
    pixCount_data_dirtyOnGpu = true;
  }
  ptrObj = NULL;
  constructDrawBaseObjectShape(&ptrObj);
  posPtr = NULL;
  hipMemcpy(positionOut_data, *gpu_positionOut_data, 16UL,
             hipMemcpyDeviceToHost);
  getPositionDataPointer(&posPtr, &positionOut_data[0], 4U, 1U);
  hipMemcpy(*gpu_color, b_color, 3UL, hipMemcpyHostToDevice);
  line_detection_kernel30<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      *gpu_color, *b_gpu_color);
  colPtr = NULL;
  hipMemcpy(color, *b_gpu_color, 3UL, hipMemcpyDeviceToHost);
  getColorDataPointer_uint8(&colPtr, &color[0], 1U, 3U);
  isInitialise = initialiseDrawbaseShape(ptrObj, 0, 2);
  if (!isInitialise) {
    if (b_img_data_dirtyOnGpu) {
      hipMemcpy(b_img_data, *b_gpu_img_data, 2764800UL,
                 hipMemcpyDeviceToHost);
    }
    if (img_data_dirtyOnGpu) {
      hipMemcpy(img_data, *gpu_img_data, 2764800UL, hipMemcpyDeviceToHost);
    }
    if (pixCount_data_dirtyOnGpu) {
      hipMemcpy(pixCount_data, *gpu_pixCount_data, 1280UL,
                 hipMemcpyDeviceToHost);
    }
    instantiateDrawBaseShape_uint8(
        ptrObj, &b_img_data[0], &img_data[0], posPtr, colPtr, 0.6, 5, 2, true,
        numRow, numCol, 3, 2, 4, 1, 1, false, false, &pixCount_data[0], 0);
    pixCount_data_dirtyOnGpu = false;
    pixCount_data_dirtyOnCpu = true;
    img_data_dirtyOnGpu = false;
    img_data_dirtyOnCpu = true;
    b_img_data_dirtyOnGpu = false;
    b_img_data_dirtyOnCpu = true;
  }
  isInitialise = initialiseDrawbaseShape(ptrObj, 1, 2);
  if (!isInitialise) {
    if (b_img_data_dirtyOnGpu) {
      hipMemcpy(b_img_data, *b_gpu_img_data, 2764800UL,
                 hipMemcpyDeviceToHost);
    }
    if (img_data_dirtyOnGpu) {
      hipMemcpy(img_data, *gpu_img_data, 2764800UL, hipMemcpyDeviceToHost);
    }
    if (pixCount_data_dirtyOnGpu) {
      hipMemcpy(pixCount_data, *gpu_pixCount_data, 1280UL,
                 hipMemcpyDeviceToHost);
    }
    instantiateDrawBaseShape_uint8(
        ptrObj, &b_img_data[0], &img_data[0], posPtr, colPtr, 0.6, 5, 2, true,
        numRow, numCol, 3, 2, 4, 1, 1, false, true, &pixCount_data[0], 1);
    b_img_data_dirtyOnGpu = false;
    b_img_data_dirtyOnCpu = true;
    pixCount_data_dirtyOnGpu = false;
    pixCount_data_dirtyOnCpu = true;
    img_data_dirtyOnGpu = false;
    img_data_dirtyOnCpu = true;
  }
  mDrawShapes(ptrObj, false, true, 2, 5, static_cast<int>(numRow),
              static_cast<int>(numCol));
  deallocateMemoryShape(ptrObj);
  deletePositionDataPointer(posPtr);
  deleteColorDataPointer_uint8(colPtr);
  line_detection_kernel31<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      *gpu_predictedPos, *gpu_positionOut);
  line_detection_kernel32<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      *gpu_positionOut);
  numRow = tmpRGB_size[0];
  numCol = tmpRGB_size[1];
  hipMemcpy(*gpu_tmpRGB_size, tmpRGB_size, 6UL, hipMemcpyHostToDevice);
  line_detection_kernel33<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      *gpu_tmpRGB_size, *gpu_sz);
  hipMemcpy(sz, *gpu_sz, 6UL, hipMemcpyDeviceToHost);
  out_size[0] = sz[0];
  out_size[1] = sz[1];
  in_cols = sz[0] * sz[1] * 3 - 1;
  validLaunchParams = mwGetLaunchParameters1D(static_cast<double>(in_cols + 1L),
                                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    if (img_data_dirtyOnCpu) {
      hipMemcpy(*gpu_img_data, img_data, 2764800UL, hipMemcpyHostToDevice);
    }
    line_detection_kernel34<<<grid, block>>>(in_cols, *gpu_img_data);
    img_data_dirtyOnCpu = false;
    img_data_dirtyOnGpu = true;
  }
  if (tmpRGB_size[0] > tmpRGB_size[1]) {
    pixCountSize = tmpRGB_size[0];
  } else {
    pixCountSize = tmpRGB_size[1];
  }
  validLaunchParams =
      mwGetLaunchParameters1D(static_cast<double>((pixCountSize - 1) + 1L),
                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    if (pixCount_data_dirtyOnCpu) {
      hipMemcpy(*gpu_pixCount_data, pixCount_data, 1280UL,
                 hipMemcpyHostToDevice);
    }
    line_detection_kernel35<<<grid, block>>>(pixCountSize, *gpu_pixCount_data);
    pixCount_data_dirtyOnGpu = true;
  }
  b_ptrObj = NULL;
  constructDrawBaseObjectShape(&b_ptrObj);
  b_posPtr = NULL;
  hipMemcpy(positionOut, *gpu_positionOut, 12UL, hipMemcpyDeviceToHost);
  getPositionDataPointer(&b_posPtr, &positionOut[0], 1U, 3U);
  hipMemcpy(*b_gpu_color, color, 3UL, hipMemcpyHostToDevice);
  line_detection_kernel36<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      *gpu_color, *b_gpu_color);
  b_colPtr = NULL;
  hipMemcpy(color, *b_gpu_color, 3UL, hipMemcpyDeviceToHost);
  getColorDataPointer_uint8(&b_colPtr, &color[0], 1U, 3U);
  isInitialise = initialiseDrawbaseShape(b_ptrObj, 0, 4);
  if (!isInitialise) {
    if (img_data_dirtyOnGpu) {
      hipMemcpy(img_data, *gpu_img_data, 2764800UL, hipMemcpyDeviceToHost);
    }
    if (b_img_data_dirtyOnGpu) {
      hipMemcpy(b_img_data, *b_gpu_img_data, 2764800UL,
                 hipMemcpyDeviceToHost);
    }
    if (pixCount_data_dirtyOnGpu) {
      hipMemcpy(pixCount_data, *gpu_pixCount_data, 1280UL,
                 hipMemcpyDeviceToHost);
    }
    instantiateDrawBaseShape_uint8(
        b_ptrObj, &img_data[0], &b_img_data[0], b_posPtr, b_colPtr, 1.0, 1, 4,
        true, numRow, numCol, 3, 2, 1, 3, 1, true, false, &pixCount_data[0], 0);
    pixCount_data_dirtyOnGpu = false;
    b_img_data_dirtyOnGpu = false;
    b_img_data_dirtyOnCpu = true;
    img_data_dirtyOnGpu = false;
    img_data_dirtyOnCpu = true;
  }
  isInitialise = initialiseDrawbaseShape(b_ptrObj, 1, 4);
  if (!isInitialise) {
    if (img_data_dirtyOnGpu) {
      hipMemcpy(img_data, *gpu_img_data, 2764800UL, hipMemcpyDeviceToHost);
    }
    if (b_img_data_dirtyOnGpu) {
      hipMemcpy(b_img_data, *b_gpu_img_data, 2764800UL,
                 hipMemcpyDeviceToHost);
    }
    if (pixCount_data_dirtyOnGpu) {
      hipMemcpy(pixCount_data, *gpu_pixCount_data, 1280UL,
                 hipMemcpyDeviceToHost);
    }
    instantiateDrawBaseShape_uint8(
        b_ptrObj, &img_data[0], &b_img_data[0], b_posPtr, b_colPtr, 1.0, 1, 4,
        true, numRow, numCol, 3, 2, 1, 3, 1, true, true, &pixCount_data[0], 1);
    b_img_data_dirtyOnCpu = true;
    img_data_dirtyOnCpu = true;
  }
  mDrawShapes(b_ptrObj, true, true, 4, 1, static_cast<int>(numRow),
              static_cast<int>(numCol));
  deallocateMemoryShape(b_ptrObj);
  deletePositionDataPointer(b_posPtr);
  deleteColorDataPointer_uint8(b_colPtr);
  //  Convert image data format to OpenCV compatible image format
  //  Convert image data format to OpenCV compatible image format
  //  Copyright 2020 The MathWorks, Inc.
  isInitialise = true;
  if ((out_size[0] != 0) && (out_size[1] != 0)) {
    plast = 0;
    in_cols = 1;
    exitg1 = false;
    while ((!exitg1) && (in_cols < 4)) {
      tmpRGB_size[0] = out_size[0];
      tmpRGB_size[1] = out_size[1];
      tmpRGB_size[2] = 3;
      if (tmpRGB_size[3 - in_cols] != 1) {
        if (plast > 4 - in_cols) {
          isInitialise = false;
          exitg1 = true;
        } else {
          plast = 4 - in_cols;
          in_cols++;
        }
      } else {
        in_cols++;
      }
    }
  }
  if (isInitialise) {
    plast = out_size[0] - 1;
    in_cols = out_size[1] - 1;
    b_out_size[0] = out_size[0];
    b_out_size[1] = out_size[1];
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<double>(((out_size[0] - 1) + 1L) *
                            (((out_size[1] - 1) + 1L) * 3L)),
        &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (img_data_dirtyOnCpu) {
        hipMemcpy(*gpu_img_data, img_data, 2764800UL, hipMemcpyHostToDevice);
      }
      hipMemcpy(*gpu_out_size, out_size, 6UL, hipMemcpyHostToDevice);
      hipMemcpy(*b_gpu_out_size, b_out_size, 6UL, hipMemcpyHostToDevice);
      if (b_img_data_dirtyOnCpu) {
        hipMemcpy(*b_gpu_img_data, b_img_data, 2764800UL,
                   hipMemcpyHostToDevice);
      }
      line_detection_kernel38<<<grid, block>>>(*gpu_img_data, *gpu_out_size,
                                               *b_gpu_out_size, plast, in_cols,
                                               *b_gpu_img_data);
      b_img_data_dirtyOnCpu = false;
    }
    Iout_size[0] = 3;
    Iout_size[1] = out_size[1];
    Iout_size[2] = out_size[0];
    in_cols = 3 * out_size[1] * out_size[0] - 1;
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<double>(in_cols + 1L), &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (b_img_data_dirtyOnCpu) {
        hipMemcpy(*b_gpu_img_data, b_img_data, 2764800UL,
                   hipMemcpyHostToDevice);
      }
      line_detection_kernel39<<<grid, block>>>(*b_gpu_img_data, in_cols,
                                               gpu_Iout_data);
      hipMemcpy(Iout_data, gpu_Iout_data,
                 3 * (Iout_size[1] * Iout_size[2]) * sizeof(unsigned char),
                 hipMemcpyDeviceToHost);
    }
  } else {
    Iout_size[0] = 3;
    Iout_size[1] = out_size[1];
    Iout_size[2] = out_size[0];
    in_cols = out_size[1] - 1;
    plast = out_size[0] - 1;
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<double>(((out_size[0] - 1) + 1L) *
                            (((out_size[1] - 1) + 1L) * 3L)),
        &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (img_data_dirtyOnCpu) {
        hipMemcpy(*gpu_img_data, img_data, 2764800UL, hipMemcpyHostToDevice);
      }
      hipMemcpy(*gpu_out_size, out_size, 6UL, hipMemcpyHostToDevice);
      hipMemcpy(*gpu_Iout_size, Iout_size, 12UL, hipMemcpyHostToDevice);
      line_detection_kernel37<<<grid, block>>>(*gpu_img_data, *gpu_out_size,
                                               *gpu_Iout_size, plast, in_cols,
                                               gpu_Iout_data);
      hipMemcpy(Iout_data, gpu_Iout_data,
                 3 * (Iout_size[1] * Iout_size[2]) * sizeof(unsigned char),
                 hipMemcpyDeviceToHost);
    }
  }
  hipFree(gpu_I_data);
  hipFree(*gpu_img_data);
  hipFree(*gpu_I_size);
  hipFree(*gpu_img_size);
  hipFree(*b_gpu_img_size);
  hipFree(*b_gpu_img_data);
  hipFree(*gpu_sz);
  hipFree(*gpu_aux1_data);
  hipFree(*gpu_aux2_data);
  hipFree(*gpu_ipRowIndices_data);
  hipFree(*gpu_rowWeights_data);
  hipFree(*gpu_ipColIndices_data);
  hipFree(*gpu_colWeights_data);
  hipFree(*gpu_rowWeightsTotal);
  hipFree(*gpu_colWeightsTotal);
  hipFree(*gpu_partialResize_size);
  hipFree(*gpu_partialResize_data);
  hipFree(*gpu_out);
  hipFree(*gpu_positionOut_data);
  hipFree(*gpu_predictedPosNorm);
  hipFree(*gpu_predictedPos);
  hipFree(*gpu_pixCount_data);
  hipFree(*gpu_color);
  hipFree(*b_gpu_color);
  hipFree(*gpu_positionOut);
  hipFree(*gpu_tmpRGB_size);
  hipFree(*gpu_out_size);
  hipFree(*b_gpu_out_size);
  hipFree(*gpu_Iout_size);
  hipFree(gpu_Iout_data);
}

//
// Copyright 2020 The MathWorks, Inc.
//
// Arguments    : void
// Return Type  : void
//
void line_detection_init()
{
  lineDetObj_not_empty = false;
}

//
// File trailer for line_detection.cu
//
// [EOF]
//
