#include "MWElementwiseAffineLayerImpl.hpp"
#include "MWCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "MWKernelHeaders.hpp"
#include <math.h>
#include <cassert>
#include <cstdio>
 MWElementwiseAffineLayerImpl::MWElementwiseAffineLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int scale_H, int scale_W, int scale_C, int 
offset_H, int offset_W, int offset_C, bool isClipped, int lowerbound, int 
upperbound, const char* qWwjVYwfnvEnFKlgpqwA, const char* 
iADjqLChtuDbEWfMYFLp) : MWCNNLayerImpl(layer, ntwk_impl), 
pvpNsgGssdTxeVoFIkXI(NULL), gNROjwaqhxDPvBWUCUcQ(NULL), pzUAoBDvaKAtdsmkQuct(scale_H), 
qEXwbWWsnOADJeTXfRVa(scale_W), pxmnUEWGnfCxJNuDkXAo(scale_C), 
hljcfGWsvZXJZNrImpJB(offset_H), hvqKUzPqCuUJRfoNlbwW(offset_W), 
hKyfKjPACkOBDvLdESxH(offset_C), ZinudJuZuGitiNTsJpBR(isClipped), 
bUVPfnrJhLfHzOLUUrKk(lowerbound), veFyKKHbdqBIvQLYBqfF(upperbound) { 
CUDA_CALL(hipMalloc((void**)&pvpNsgGssdTxeVoFIkXI, 
sizeof(float)*pzUAoBDvaKAtdsmkQuct*qEXwbWWsnOADJeTXfRVa*pxmnUEWGnfCxJNuDkXAo)); 
CUDA_CALL(hipMalloc((void**)&gNROjwaqhxDPvBWUCUcQ, 
sizeof(float)*hljcfGWsvZXJZNrImpJB*hvqKUzPqCuUJRfoNlbwW*hKyfKjPACkOBDvLdESxH));  
loadScale(qWwjVYwfnvEnFKlgpqwA); loadOffset(iADjqLChtuDbEWfMYFLp); } 
MWElementwiseAffineLayerImpl::~MWElementwiseAffineLayerImpl() { } void 
MWElementwiseAffineLayerImpl::propagateSize() { } void 
MWElementwiseAffineLayerImpl::predict() { MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); MWTensor<float>* ipTensor = 
static_cast<MWTensor<float>*>(ipTensorBase); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); int WprSrhAStKGxyXeoxETy = 
ipTensor->getHeight(); int XCLDbxHBtWRStETWIkId = ipTensor->getWidth(); int 
WmXADZOqdcQvtBUvFerh = ipTensor->getChannels(); long int 
YeIFysyIXePEVfpcANol = WprSrhAStKGxyXeoxETy*XCLDbxHBtWRStETWIkId; long 
int YOWMnLKOMqAODXiVNoGy = 
YeIFysyIXePEVfpcANol*WmXADZOqdcQvtBUvFerh; long int 
YNmJhGSUszJKxsodxiuV = ipTensor->getNumElements(); long int sFIUeCwGDlfadqOrGZHC = 
((YNmJhGSUszJKxsodxiuV + 31) / 32) * 32; int tGsvtyAVkrDznETdweDC = 
(sFIUeCwGDlfadqOrGZHC < 1024) ? sFIUeCwGDlfadqOrGZHC : 1024; long int 
KZWeXiYFmdpQdsgidKeG = (YNmJhGSUszJKxsodxiuV + tGsvtyAVkrDznETdweDC - 
1) / tGsvtyAVkrDznETdweDC; long int qBTcAwVGZERyCjGYByPe = 
pzUAoBDvaKAtdsmkQuct * qEXwbWWsnOADJeTXfRVa * pxmnUEWGnfCxJNuDkXAo; long int 
hnewnpwgzKmOdualajhn = hljcfGWsvZXJZNrImpJB * hvqKUzPqCuUJRfoNlbwW * 
hKyfKjPACkOBDvLdESxH; assert(qBTcAwVGZERyCjGYByPe <= YNmJhGSUszJKxsodxiuV); 
assert(hnewnpwgzKmOdualajhn <= YNmJhGSUszJKxsodxiuV); if (qBTcAwVGZERyCjGYByPe == 
1) { scale_scalar_kernel<<<KZWeXiYFmdpQdsgidKeG, 
tGsvtyAVkrDznETdweDC>>>( ipTensor->getData(),  opTensor->getData(), 
pvpNsgGssdTxeVoFIkXI, YNmJhGSUszJKxsodxiuV); } else if (pzUAoBDvaKAtdsmkQuct == 1 && 
qEXwbWWsnOADJeTXfRVa == 1 && qBTcAwVGZERyCjGYByPe > 1) { 
scale_vector_kernel<<<KZWeXiYFmdpQdsgidKeG, tGsvtyAVkrDznETdweDC>>>( 
ipTensor->getData(),  opTensor->getData(), pvpNsgGssdTxeVoFIkXI, 
YeIFysyIXePEVfpcANol, YOWMnLKOMqAODXiVNoGy, 
YNmJhGSUszJKxsodxiuV); } else if (YOWMnLKOMqAODXiVNoGy == 
qBTcAwVGZERyCjGYByPe) {  scale_tensor3d_kernel<<<KZWeXiYFmdpQdsgidKeG, 
tGsvtyAVkrDznETdweDC>>>( ipTensor->getData(),  opTensor->getData(), 
pvpNsgGssdTxeVoFIkXI,  YOWMnLKOMqAODXiVNoGy, YNmJhGSUszJKxsodxiuV); } else 
{ scale_matrix2d_kernel<<<KZWeXiYFmdpQdsgidKeG, 
tGsvtyAVkrDznETdweDC>>>( ipTensor->getData(),  opTensor->getData(), 
pvpNsgGssdTxeVoFIkXI,  YeIFysyIXePEVfpcANol, YNmJhGSUszJKxsodxiuV); } if 
(hnewnpwgzKmOdualajhn == 1) { offset_scalar_kernel<<<KZWeXiYFmdpQdsgidKeG, 
tGsvtyAVkrDznETdweDC>>>( opTensor->getData(),  opTensor->getData(), 
gNROjwaqhxDPvBWUCUcQ, YNmJhGSUszJKxsodxiuV, ZinudJuZuGitiNTsJpBR, 
bUVPfnrJhLfHzOLUUrKk, veFyKKHbdqBIvQLYBqfF); } else if (hljcfGWsvZXJZNrImpJB 
== 1 && hvqKUzPqCuUJRfoNlbwW == 1 && hnewnpwgzKmOdualajhn > 1) { 
offset_vector_kernel<<<KZWeXiYFmdpQdsgidKeG, tGsvtyAVkrDznETdweDC>>>( 
opTensor->getData(),  opTensor->getData(), gNROjwaqhxDPvBWUCUcQ, 
YeIFysyIXePEVfpcANol, YOWMnLKOMqAODXiVNoGy, 
YNmJhGSUszJKxsodxiuV, ZinudJuZuGitiNTsJpBR, bUVPfnrJhLfHzOLUUrKk, 
veFyKKHbdqBIvQLYBqfF); } else if (YOWMnLKOMqAODXiVNoGy == 
hnewnpwgzKmOdualajhn) { offset_tensor3d_kernel<<<KZWeXiYFmdpQdsgidKeG, 
tGsvtyAVkrDznETdweDC>>>( opTensor->getData(),  opTensor->getData(), 
gNROjwaqhxDPvBWUCUcQ, YOWMnLKOMqAODXiVNoGy, YNmJhGSUszJKxsodxiuV, 
ZinudJuZuGitiNTsJpBR, bUVPfnrJhLfHzOLUUrKk, veFyKKHbdqBIvQLYBqfF); } else { 
offset_matrix2d_kernel<<<KZWeXiYFmdpQdsgidKeG, 
tGsvtyAVkrDznETdweDC>>>( opTensor->getData(),  opTensor->getData(), 
gNROjwaqhxDPvBWUCUcQ, YeIFysyIXePEVfpcANol, YNmJhGSUszJKxsodxiuV, 
ZinudJuZuGitiNTsJpBR, bUVPfnrJhLfHzOLUUrKk, veFyKKHbdqBIvQLYBqfF); } return; 
} void MWElementwiseAffineLayerImpl::cleanup() { if (pvpNsgGssdTxeVoFIkXI) { 
CUDA_FREE_CALL(pvpNsgGssdTxeVoFIkXI); pvpNsgGssdTxeVoFIkXI = NULL; } if 
(gNROjwaqhxDPvBWUCUcQ) { CUDA_FREE_CALL(gNROjwaqhxDPvBWUCUcQ); gNROjwaqhxDPvBWUCUcQ = 
NULL; }  } void MWElementwiseAffineLayerImpl::loadScale(const char* 
qWwjVYwfnvEnFKlgpqwA) { FILE* QwUuNuQNtlPXrIwRNiSZ = 
MWCNNLayer::openBinaryFile(qWwjVYwfnvEnFKlgpqwA); assert(QwUuNuQNtlPXrIwRNiSZ); long 
int dkLDkRwCBjeybwDHbKiE = pzUAoBDvaKAtdsmkQuct*qEXwbWWsnOADJeTXfRVa*pxmnUEWGnfCxJNuDkXAo; 
float* LgxABSJPBXdCozJkFqTg = MALLOC_CALL(sizeof(float)*dkLDkRwCBjeybwDHbKiE); 
MWCNNLayer::call_fread(LgxABSJPBXdCozJkFqTg, sizeof(float), dkLDkRwCBjeybwDHbKiE, 
QwUuNuQNtlPXrIwRNiSZ, qWwjVYwfnvEnFKlgpqwA); CUDA_CALL(hipMemcpy(pvpNsgGssdTxeVoFIkXI, 
LgxABSJPBXdCozJkFqTg, sizeof(float)*dkLDkRwCBjeybwDHbKiE, hipMemcpyHostToDevice)); 
free(LgxABSJPBXdCozJkFqTg); fclose(QwUuNuQNtlPXrIwRNiSZ);  } void 
MWElementwiseAffineLayerImpl::loadOffset(const char* iADjqLChtuDbEWfMYFLp) { 
FILE* QwUuNuQNtlPXrIwRNiSZ = MWCNNLayer::openBinaryFile(iADjqLChtuDbEWfMYFLp); 
assert(QwUuNuQNtlPXrIwRNiSZ); long int dkLDkRwCBjeybwDHbKiE = 
hljcfGWsvZXJZNrImpJB*hvqKUzPqCuUJRfoNlbwW*hKyfKjPACkOBDvLdESxH; float* 
LgxABSJPBXdCozJkFqTg = MALLOC_CALL(sizeof(float)*dkLDkRwCBjeybwDHbKiE); 
MWCNNLayer::call_fread(LgxABSJPBXdCozJkFqTg, sizeof(float), dkLDkRwCBjeybwDHbKiE, 
QwUuNuQNtlPXrIwRNiSZ, iADjqLChtuDbEWfMYFLp); CUDA_CALL(hipMemcpy(gNROjwaqhxDPvBWUCUcQ, 
LgxABSJPBXdCozJkFqTg, sizeof(float)*dkLDkRwCBjeybwDHbKiE, hipMemcpyHostToDevice)); 
free(LgxABSJPBXdCozJkFqTg); fclose(QwUuNuQNtlPXrIwRNiSZ);  }