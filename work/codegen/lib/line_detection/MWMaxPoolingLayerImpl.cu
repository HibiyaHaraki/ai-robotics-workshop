#include "hip/hip_runtime.h"
#include "MWMaxPoolingLayerImpl.hpp"
#include "MWCNNLayerImpl.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWCNNLayer.hpp"
#include "MWTargetNetworkImpl.hpp"
 void __global__ MWSetDyForBackPropImpl(float * PfisSEEWDaQFynnzlcin, const int 
fOpFYwKNwIfWjnPzNuob); void __global__ doMWMaxPoolingLayerImpl(float * 
UpnEytIWGokwbTFkBcSx, float * UdmcwaUkepxfZrpdpcAN, const int 
BdqURaHPmdnfzvtUvocl); MWMaxPoolingLayerImpl::MWMaxPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int DSsxcjIrUgZCKZovyNQf,  int 
EvebzoroiuKkIxwjkGnD,  int FshVHIJMRAhtQirYPlZd,  int 
GDRXdUDklKFEYEfifhIH, int ClEhcJFlvGCgiavziIag, int 
CZNYmBcNFSZWvaCklqeM,  int CufLFODQDXTAPyRqYodN, int 
DCdZnqpcBnvXVgEsLBnz, bool GLovsOhUpzOJhKgXUAJY, int 
nlIRrOJaFuVaywxOqOyb, const char* ABtNoHVrQOgivJIJagNR,  int 
fSKMHAqIghbYYgyIpNDw) : MWCNNLayerImpl(layer, ntwk_impl) , 
BRSPqxNffoBYKqpSVHne(GLovsOhUpzOJhKgXUAJY) , UpnEytIWGokwbTFkBcSx(0) 
, PfisSEEWDaQFynnzlcin(0) , DRzwhbNPpftRRIXXfHzd(DSsxcjIrUgZCKZovyNQf) , 
EfvWctmlsWAPsxXgdKWf(EvebzoroiuKkIxwjkGnD) , 
DGzdAcREJHGXjyRzNjJV(DSsxcjIrUgZCKZovyNQf) , 
ECTnqgWHyHCHCLBZlffd(EvebzoroiuKkIxwjkGnD) , 
CTCbzQMDaLxINPbODdng(ClEhcJFlvGCgiavziIag) , 
CLOUhPjbgggWoXHTtmjC(CZNYmBcNFSZWvaCklqeM) , 
CpMjJjtGOeWOzwxpAAQP(CufLFODQDXTAPyRqYodN) , 
CqtPRJvHlGJFssiPzsOm(DCdZnqpcBnvXVgEsLBnz) , 
FrpxvsDMwwgbpqHXWxmN(FshVHIJMRAhtQirYPlZd) , 
FwLnexHgxHRquTKmNpoa(GDRXdUDklKFEYEfifhIH) , 
pFoPPXxxFRbjXXxQWItv(std::pow(2, 
nlIRrOJaFuVaywxOqOyb)) , 
vjDFlBZzKvbpPseAtMBP(std::string{"INT8x4"}.compare(ABtNoHVrQOgivJIJagNR) 
== 0) , fXhhiexIRPLyKXApPmmy(fSKMHAqIghbYYgyIpNDw) {  
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&mtolGPkUMBYDlSSqrRzc)); 
createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex()); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&DqxLTLaJwwgQqmrtCDuu));  } 
MWMaxPoolingLayerImpl::~MWMaxPoolingLayerImpl() { } void 
MWMaxPoolingLayerImpl::propagateSize() {  MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); if ((DRzwhbNPpftRRIXXfHzd == -1) && 
(EfvWctmlsWAPsxXgdKWf == -1)) { DGzdAcREJHGXjyRzNjJV = 
ipTensorBase->getHeight(); ECTnqgWHyHCHCLBZlffd = ipTensorBase->getWidth(); } int 
muwRQxtWMMXAPxSuMYBw = CTCbzQMDaLxINPbODdng; int 
nDsbARncmIrIaLubvLVZ = CpMjJjtGOeWOzwxpAAQP; 
CUDNN_CALL(hipdnnSetPooling2dDescriptor(mtolGPkUMBYDlSSqrRzc, HIPDNN_POOLING_MAX, 
HIPDNN_NOT_PROPAGATE_NAN, DGzdAcREJHGXjyRzNjJV, ECTnqgWHyHCHCLBZlffd, 
muwRQxtWMMXAPxSuMYBw, nDsbARncmIrIaLubvLVZ, FrpxvsDMwwgbpqHXWxmN, 
FwLnexHgxHRquTKmNpoa)); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensorBase->getSourcePortIndex()); assert(desc); if 
(ipTensorBase->isInt8()) { if (vjDFlBZzKvbpPseAtMBP) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(DqxLTLaJwwgQqmrtCDuu, 
HIPDNN_TENSOR_NCHW_VECT_C,  HIPDNN_DATA_INT8x4,  ipTensorBase->getBatchSize(),  
ipTensorBase->getChannels(),  ipTensorBase->getHeight(),  
ipTensorBase->getWidth())  ); MWCNNLayerImpl::setDescriptorForINT8(*desc, 
static_cast<MWTensor<signed char>*>(opTensorBase), HIPDNN_DATA_INT8x4, 
HIPDNN_TENSOR_NCHW_VECT_C);  } else { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(DqxLTLaJwwgQqmrtCDuu, 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_INT8, ipTensorBase->getBatchSize(), 
ipTensorBase->getChannels(), ipTensorBase->getHeight(), 
ipTensorBase->getWidth())); MWCNNLayerImpl::setDescriptorForINT8(*desc, 
static_cast<MWTensor<signed char>*>(opTensorBase), HIPDNN_DATA_INT8, 
HIPDNN_TENSOR_NCHW); } } else { setDescriptor<float>(*desc, 
static_cast<MWTensor<float>*>(opTensorBase)); } } void 
MWMaxPoolingLayerImpl::allocate() { MWCNNLayer* maxpoolLayer = getLayer(); 
MWTensorBase* ipTensor = maxpoolLayer->getInputTensor(0); MWTensorBase* 
opTensor = maxpoolLayer->getOutputTensor(0); if (BRSPqxNffoBYKqpSVHne){ const 
int edQOkUJIZbwzEeIcCLzG = ipTensor->getNumElements(); 
CUDA_CALL(hipMalloc((void**)&UpnEytIWGokwbTFkBcSx, 
sizeof(float)*edQOkUJIZbwzEeIcCLzG)); const int fOpFYwKNwIfWjnPzNuob = 
opTensor->getNumElements(); CUDA_CALL(hipMalloc((void**)&PfisSEEWDaQFynnzlcin, 
sizeof(float)*fOpFYwKNwIfWjnPzNuob)); int tGsvtyAVkrDznETdweDC = 
(fOpFYwKNwIfWjnPzNuob < 1024) ? fOpFYwKNwIfWjnPzNuob : 1024; int 
KZWeXiYFmdpQdsgidKeG = (fOpFYwKNwIfWjnPzNuob + tGsvtyAVkrDznETdweDC - 
1)/tGsvtyAVkrDznETdweDC; 
MWSetDyForBackPropImpl<<<KZWeXiYFmdpQdsgidKeG, 
tGsvtyAVkrDznETdweDC>>>( PfisSEEWDaQFynnzlcin, fOpFYwKNwIfWjnPzNuob); } } void 
MWMaxPoolingLayerImpl::deallocate() { if (UpnEytIWGokwbTFkBcSx){ 
CUDA_FREE_CALL(UpnEytIWGokwbTFkBcSx); UpnEytIWGokwbTFkBcSx = 
NULL; } if (PfisSEEWDaQFynnzlcin){ CUDA_FREE_CALL(PfisSEEWDaQFynnzlcin); PfisSEEWDaQFynnzlcin = 
NULL; }  } void MWMaxPoolingLayerImpl::predict() { MWCNNLayer* maxpoolLayer = 
getLayer(); MWTensorBase* ipTensorBase = maxpoolLayer->getInputTensor(0); 
MWTensorBase* opTensorBase = maxpoolLayer->getOutputTensor(0); 
hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensorBase->getSourcePortIndex()); assert(desc); 
hipdnnTensorDescriptor_t YCSvyQZBWMDYQXHtyVai; if (opTensorBase->isInt8()) { 
YCSvyQZBWMDYQXHtyVai = DqxLTLaJwwgQqmrtCDuu; MWTensor<signed char>* ipTensor = 
static_cast<MWTensor<signed char>*>(ipTensorBase); MWTensor<signed char>* 
opTensor = static_cast<MWTensor<signed char>*>(opTensorBase); 
CUDNN_CALL(hipdnnPoolingForward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
mtolGPkUMBYDlSSqrRzc, &pFoPPXxxFRbjXXxQWItv, YCSvyQZBWMDYQXHtyVai, 
ipTensor->getData(), getZeroPtr(), *desc, opTensor->getData())); } else { 
YCSvyQZBWMDYQXHtyVai = MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase); 
MWTensor<float>* ipTensor = static_cast<MWTensor<float>*>(ipTensorBase); 
MWTensor<float>* opTensor = static_cast<MWTensor<float>*>(opTensorBase); 
CUDNN_CALL(hipdnnPoolingForward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
mtolGPkUMBYDlSSqrRzc, getOnePtr(), YCSvyQZBWMDYQXHtyVai, ipTensor->getData(), 
getZeroPtr(), *desc, opTensor->getData())); if (BRSPqxNffoBYKqpSVHne) { 
CUDNN_CALL(hipdnnPoolingBackward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
mtolGPkUMBYDlSSqrRzc, getOnePtr(), *desc, opTensor->getData(), *desc, 
PfisSEEWDaQFynnzlcin, YCSvyQZBWMDYQXHtyVai, ipTensor->getData(), getZeroPtr(), 
YCSvyQZBWMDYQXHtyVai, UpnEytIWGokwbTFkBcSx)); int edQOkUJIZbwzEeIcCLzG = 
ipTensor->getNumElements(); int tGsvtyAVkrDznETdweDC = 
(edQOkUJIZbwzEeIcCLzG < 1024) ? edQOkUJIZbwzEeIcCLzG : 1024; int 
KZWeXiYFmdpQdsgidKeG = (edQOkUJIZbwzEeIcCLzG + tGsvtyAVkrDznETdweDC - 
1)/tGsvtyAVkrDznETdweDC; 
doMWMaxPoolingLayerImpl<<<KZWeXiYFmdpQdsgidKeG, 
tGsvtyAVkrDznETdweDC>>>( UpnEytIWGokwbTFkBcSx, 
static_cast<MWTensor<float>*>(maxpoolLayer->getOutputTensor(1))->getData(), 
edQOkUJIZbwzEeIcCLzG); }  } return; } void MWMaxPoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(mtolGPkUMBYDlSSqrRzc)); MWTensorBase* 
opTensorBase = getLayer()->getOutputTensor(0);  if (opTensorBase->isInt8()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(DqxLTLaJwwgQqmrtCDuu)); } } float* 
MWMaxPoolingLayerImpl::getIndexData()  { return 
static_cast<MWTensor<float>*>(getLayer()->getOutputTensor(1))->getData(); } 
void __global__ __launch_bounds__(1024) MWSetDyForBackPropImpl(float * 
PfisSEEWDaQFynnzlcin, const int fOpFYwKNwIfWjnPzNuob) { for(int i = blockDim.x * 
blockIdx.x + threadIdx.x; i < fOpFYwKNwIfWjnPzNuob; i+= blockDim.x*gridDim.x) { 
PfisSEEWDaQFynnzlcin[i] = i+1; } } void __global__ __launch_bounds__(1024) 
doMWMaxPoolingLayerImpl(float * UpnEytIWGokwbTFkBcSx, float * 
UdmcwaUkepxfZrpdpcAN, const int BdqURaHPmdnfzvtUvocl) { for(int i = blockDim.x * 
blockIdx.x + threadIdx.x; i < BdqURaHPmdnfzvtUvocl; i+= blockDim.x*gridDim.x) { if 
(static_cast<int>(UpnEytIWGokwbTFkBcSx[i]) != 0){ 
UdmcwaUkepxfZrpdpcAN[static_cast<int>(UpnEytIWGokwbTFkBcSx[i])-1] = 
i; } } }