//
// File: line_detection_terminate.cu
//
// GPU Coder version                    : 2.1
// CUDA/C/C++ source code generated on  : 11-Nov-2021 11:08:16
//

// Include Files
#include "line_detection_terminate.h"
#include "line_detection_data.h"
#include <cstdio>

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void line_detection_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "ERR[%d] %s:%s\n", errCode, hipGetErrorName(errCode),
            hipGetErrorString(errCode));
    exit(errCode);
  }
  isInitialized_line_detection = false;
}

//
// File trailer for line_detection_terminate.cu
//
// [EOF]
//
