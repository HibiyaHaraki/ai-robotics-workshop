#include "hip/hip_runtime.h"
#include "MWAvgPoolingLayerImpl.hpp"
#include "MWCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include "MWTargetNetworkImpl.hpp"
 MWAvgPoolingLayerImpl::MWAvgPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int DSsxcjIrUgZCKZovyNQf, int 
EvebzoroiuKkIxwjkGnD, int FshVHIJMRAhtQirYPlZd, int GDRXdUDklKFEYEfifhIH, 
int ClEhcJFlvGCgiavziIag, int CZNYmBcNFSZWvaCklqeM, int 
CufLFODQDXTAPyRqYodN, int DCdZnqpcBnvXVgEsLBnz, int 
nlIRrOJaFuVaywxOqOyb, const char* ABtNoHVrQOgivJIJagNR) : 
MWCNNLayerImpl(layer, ntwk_impl) , TfsmDFpPPOscKZifVzSQ(NULL) , 
DRzwhbNPpftRRIXXfHzd(DSsxcjIrUgZCKZovyNQf) , 
EfvWctmlsWAPsxXgdKWf(EvebzoroiuKkIxwjkGnD) , 
DGzdAcREJHGXjyRzNjJV(DSsxcjIrUgZCKZovyNQf) , 
ECTnqgWHyHCHCLBZlffd(EvebzoroiuKkIxwjkGnD) , 
FrpxvsDMwwgbpqHXWxmN(FshVHIJMRAhtQirYPlZd) , 
FwLnexHgxHRquTKmNpoa(GDRXdUDklKFEYEfifhIH) , 
CTCbzQMDaLxINPbODdng(ClEhcJFlvGCgiavziIag) , 
CLOUhPjbgggWoXHTtmjC(CZNYmBcNFSZWvaCklqeM) , 
CpMjJjtGOeWOzwxpAAQP(CufLFODQDXTAPyRqYodN) , 
CqtPRJvHlGJFssiPzsOm(DCdZnqpcBnvXVgEsLBnz) , 
IbSWJNMuIiKbocfQKqXb((CTCbzQMDaLxINPbODdng != CLOUhPjbgggWoXHTtmjC) 
|| (CpMjJjtGOeWOzwxpAAQP != CqtPRJvHlGJFssiPzsOm)) , 
muwRQxtWMMXAPxSuMYBw(ClEhcJFlvGCgiavziIag) , 
nDsbARncmIrIaLubvLVZ(CufLFODQDXTAPyRqYodN) , 
pFoPPXxxFRbjXXxQWItv(std::pow(2, 
nlIRrOJaFuVaywxOqOyb)) , 
vjDFlBZzKvbpPseAtMBP(std::string{"INT8x4"}.compare(ABtNoHVrQOgivJIJagNR) 
== 0){ CUDNN_CALL(hipdnnCreatePoolingDescriptor(&mtolGPkUMBYDlSSqrRzc)); 
MWTensorBase* ipTensor = getLayer()->getInputTensor(0); if 
(IbSWJNMuIiKbocfQKqXb) { assert(ipTensor->isFloat()); 
muwRQxtWMMXAPxSuMYBw = 0;  nDsbARncmIrIaLubvLVZ = 0; 
TfsmDFpPPOscKZifVzSQ = new MWTensor<float>(-1, -1, -1, -1, -1, NULL, getLayer(), 0); 
if (!TfsmDFpPPOscKZifVzSQ) { MWCNNLayerImpl::throwAllocationError(__LINE__ , 
__FILE__); } CUDNN_CALL(hipdnnCreateTensorDescriptor(&YCSvyQZBWMDYQXHtyVai));  } 
else { TfsmDFpPPOscKZifVzSQ = ipTensor;  } assert(TfsmDFpPPOscKZifVzSQ != NULL); 
MWTensorBase* opTensorBase = getLayer()->getOutputTensor(0); 
createAndAddDescriptor(opTensorBase->getSourcePortIndex()); if 
(opTensorBase->isInt8()) { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&JABfZsGuaCAmcRcqOYEO)); } } 
MWAvgPoolingLayerImpl::~MWAvgPoolingLayerImpl() { } void 
MWAvgPoolingLayerImpl::propagateSize() { MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0);  if ((DRzwhbNPpftRRIXXfHzd == -1) && 
(EfvWctmlsWAPsxXgdKWf == -1)) { DGzdAcREJHGXjyRzNjJV = 
ipTensorBase->getHeight(); ECTnqgWHyHCHCLBZlffd = ipTensorBase->getWidth(); } int 
inputH; int inputW; if (IbSWJNMuIiKbocfQKqXb) { inputH = 
ipTensorBase->getHeight() + CTCbzQMDaLxINPbODdng + CLOUhPjbgggWoXHTtmjC;  
inputW = ipTensorBase->getWidth() + CpMjJjtGOeWOzwxpAAQP + 
CqtPRJvHlGJFssiPzsOm; } else { inputH = ipTensorBase->getHeight(); inputW = 
ipTensorBase->getWidth(); } TfsmDFpPPOscKZifVzSQ->setHeight(inputH); 
TfsmDFpPPOscKZifVzSQ->setWidth(inputW); 
TfsmDFpPPOscKZifVzSQ->setChannels(ipTensorBase->getChannels()); 
TfsmDFpPPOscKZifVzSQ->setBatchSize(ipTensorBase->getBatchSize()); 
TfsmDFpPPOscKZifVzSQ->setSequenceLength(ipTensorBase->getSequenceLength()); 
assert(TfsmDFpPPOscKZifVzSQ->getSequenceLength() == 1); 
CUDNN_CALL(hipdnnSetPooling2dDescriptor(mtolGPkUMBYDlSSqrRzc,  
HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,  HIPDNN_NOT_PROPAGATE_NAN,  
DGzdAcREJHGXjyRzNjJV,  ECTnqgWHyHCHCLBZlffd,  muwRQxtWMMXAPxSuMYBw,  
nDsbARncmIrIaLubvLVZ,  FrpxvsDMwwgbpqHXWxmN,  FwLnexHgxHRquTKmNpoa));  
hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensorBase->getSourcePortIndex()); assert(desc); if 
(opTensorBase->isInt8()) { if (vjDFlBZzKvbpPseAtMBP) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JABfZsGuaCAmcRcqOYEO, 
HIPDNN_TENSOR_NCHW_VECT_C,  HIPDNN_DATA_INT8x4,  ipTensorBase->getBatchSize(),  
ipTensorBase->getChannels(),  ipTensorBase->getHeight(),  
ipTensorBase->getWidth())  ); MWCNNLayerImpl::setDescriptorForINT8(*desc, 
static_cast<MWTensor<signed char>*>(opTensorBase), HIPDNN_DATA_INT8x4, 
HIPDNN_TENSOR_NCHW_VECT_C );  } else { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(JABfZsGuaCAmcRcqOYEO, 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_INT8, ipTensorBase->getBatchSize(), 
ipTensorBase->getChannels(), ipTensorBase->getHeight(), 
ipTensorBase->getWidth())); MWCNNLayerImpl::setDescriptorForINT8(*desc, 
static_cast<MWTensor<signed char>*>(opTensorBase), HIPDNN_DATA_INT8, 
HIPDNN_TENSOR_NCHW); } } else { if (IbSWJNMuIiKbocfQKqXb) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(YCSvyQZBWMDYQXHtyVai, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, TfsmDFpPPOscKZifVzSQ->getBatchSize(), TfsmDFpPPOscKZifVzSQ->getChannels(), 
TfsmDFpPPOscKZifVzSQ->getHeight(), TfsmDFpPPOscKZifVzSQ->getWidth())); } else { 
YCSvyQZBWMDYQXHtyVai = MWCNNLayerImpl::getCuDNNDescriptor(TfsmDFpPPOscKZifVzSQ); }  
setDescriptor<float>(*desc, static_cast<MWTensor<float>*>(opTensorBase)); } } 
void MWAvgPoolingLayerImpl::allocate() { MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); if (IbSWJNMuIiKbocfQKqXb) { float* 
newInput; int inputH = ipTensorBase->getHeight() + CTCbzQMDaLxINPbODdng + 
CLOUhPjbgggWoXHTtmjC;  int inputW = ipTensorBase->getWidth() + 
CpMjJjtGOeWOzwxpAAQP + CqtPRJvHlGJFssiPzsOm; int paddedSize = 
ipTensorBase->getBatchSize() * ipTensorBase->getChannels() * inputH * inputW; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float)*paddedSize)); 
CUDA_CALL(hipMemset(newInput, 0, sizeof(float)*paddedSize)); 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->setData(newInput); } } void 
MWAvgPoolingLayerImpl::deallocate() { if (TfsmDFpPPOscKZifVzSQ != 
getLayer()->getInputTensor(0)) { assert(IbSWJNMuIiKbocfQKqXb); 
CUDA_FREE_CALL(static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData()); 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->setData((float*)NULL); } } void 
MWAvgPoolingLayerImpl::predict() { MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(0); MWTensorBase* ipTensorBase = 
getLayer()->getInputTensor(0); hipdnnTensorDescriptor_t* desc = 
getDescriptor(opTensorBase->getSourcePortIndex()); assert(desc); if 
(opTensorBase->isInt8()) { assert(!IbSWJNMuIiKbocfQKqXb); 
MWTensor<signed char>* opTensor = static_cast<MWTensor<signed 
char>*>(opTensorBase); 
CUDNN_CALL(hipdnnPoolingForward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
mtolGPkUMBYDlSSqrRzc, &pFoPPXxxFRbjXXxQWItv, 
JABfZsGuaCAmcRcqOYEO, static_cast<MWTensor<signed 
char>*>(TfsmDFpPPOscKZifVzSQ)->getData(), getZeroPtr(), *desc, opTensor->getData())); 
}else{ MWTensor<float>* ipTensor = static_cast<MWTensor<float>*>(ipTensorBase); 
MWTensor<float>* opTensor = static_cast<MWTensor<float>*>(opTensorBase); if 
(TfsmDFpPPOscKZifVzSQ != ipTensorBase) { 
CUDA_CALL(hipMemset(static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), 
0, sizeof(float)*TfsmDFpPPOscKZifVzSQ->getNumElements())); 
MWCNNLayerImpl::padInput(ipTensor->getData(), ipTensor->getHeight(), 
ipTensor->getWidth(), ipTensor->getChannels(), TfsmDFpPPOscKZifVzSQ->getHeight(), 
TfsmDFpPPOscKZifVzSQ->getWidth(), CTCbzQMDaLxINPbODdng, CpMjJjtGOeWOzwxpAAQP, 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), 
ipTensor->getNumElements()); } assert(opTensor->getData() != 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData()); 
CUDNN_CALL(hipdnnPoolingForward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), 
mtolGPkUMBYDlSSqrRzc, getOnePtr(), YCSvyQZBWMDYQXHtyVai, 
static_cast<MWTensor<float>*>(TfsmDFpPPOscKZifVzSQ)->getData(), getZeroPtr(), *desc, 
opTensor->getData())); } } void MWAvgPoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(mtolGPkUMBYDlSSqrRzc)); if 
(TfsmDFpPPOscKZifVzSQ != getLayer()->getInputTensor(0)) { 
assert(IbSWJNMuIiKbocfQKqXb); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(YCSvyQZBWMDYQXHtyVai)); } MWTensorBase* 
opTensorBase = getLayer()->getOutputTensor(0); if (opTensorBase->isInt8()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(JABfZsGuaCAmcRcqOYEO)); }  }