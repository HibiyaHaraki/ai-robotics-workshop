#include "hip/hip_runtime.h"
#include "MWCNNLayerImpl.hpp"
#include "MWCNNLayer.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "MWTensorBase.hpp"
#include "MWTensor.hpp"
#include <cstdlib>
#include <cassert>
#include <cstdio>
#include <stdexcept>
#ifdef RANDOM
#include <hiprand.h>
 hiprandGenerator_t RQSttSyDKXCHDWSijmNk; void 
curand_call_line_file(hiprandStatus_t rlQsibXJSWJVnUVpdNeL, const int 
bERCRkGjpaKXMNComoYl, const char *QTXuPiGKeBUnmRzhlIDp) { if (rlQsibXJSWJVnUVpdNeL != 
HIPRAND_STATUS_SUCCESS) { char buffer[100]; int numElem = sprintf(buffer, 
"%d at line: %d, file: %s\n", rlQsibXJSWJVnUVpdNeL, bERCRkGjpaKXMNComoYl, 
QTXuPiGKeBUnmRzhlIDp); throw std::runtime_error(buffer); } }
#endif
 float* malloc_call_line_file(size_t msize, const int bERCRkGjpaKXMNComoYl, const 
char *QTXuPiGKeBUnmRzhlIDp) { float * mem = (float*)malloc(msize); if (!mem) { char 
buffer[100]; int numElem = sprintf(buffer, "%s at line: %d, file: %s\n", 
"Memory allocation failed. ", bERCRkGjpaKXMNComoYl, QTXuPiGKeBUnmRzhlIDp); throw 
std::runtime_error(buffer); } return mem; } void 
cuda_call_line_file(hipError_t rlQsibXJSWJVnUVpdNeL, const int bERCRkGjpaKXMNComoYl, 
const char *QTXuPiGKeBUnmRzhlIDp) { if (rlQsibXJSWJVnUVpdNeL != hipSuccess) { 
throw_cuda_error(rlQsibXJSWJVnUVpdNeL, bERCRkGjpaKXMNComoYl, QTXuPiGKeBUnmRzhlIDp);  } } 
void throw_cuda_error(hipError_t rlQsibXJSWJVnUVpdNeL, const int bERCRkGjpaKXMNComoYl, 
const char *QTXuPiGKeBUnmRzhlIDp) { char buffer[100]; int numElem = sprintf(buffer, 
"Cuda Error %d(%s) at line: %d, file: %s\n", rlQsibXJSWJVnUVpdNeL, 
hipGetErrorString(rlQsibXJSWJVnUVpdNeL), bERCRkGjpaKXMNComoYl, QTXuPiGKeBUnmRzhlIDp); 
rlQsibXJSWJVnUVpdNeL = hipGetLastError();  throw std::runtime_error(buffer);  } 
void cudnn_call_line_file(hipdnnStatus_t rlQsibXJSWJVnUVpdNeL, const int 
bERCRkGjpaKXMNComoYl, const char *QTXuPiGKeBUnmRzhlIDp) { if (rlQsibXJSWJVnUVpdNeL != 
HIPDNN_STATUS_SUCCESS) { char buffer[100]; int numElem = sprintf(buffer, 
"CuDNN Error %d(%s) at line: %d, file: %s\n", rlQsibXJSWJVnUVpdNeL, 
hipdnnGetErrorString(rlQsibXJSWJVnUVpdNeL), bERCRkGjpaKXMNComoYl, QTXuPiGKeBUnmRzhlIDp); 
throw std::runtime_error(buffer); } } const char* 
cublasGetErrorString(hipblasStatus_t rlQsibXJSWJVnUVpdNeL) { 
switch(rlQsibXJSWJVnUVpdNeL) { case HIPBLAS_STATUS_SUCCESS: return 
"HIPBLAS_STATUS_SUCCESS"; case HIPBLAS_STATUS_NOT_INITIALIZED: return 
"HIPBLAS_STATUS_NOT_INITIALIZED"; case HIPBLAS_STATUS_ALLOC_FAILED: return 
"HIPBLAS_STATUS_ALLOC_FAILED"; case HIPBLAS_STATUS_INVALID_VALUE: return 
"HIPBLAS_STATUS_INVALID_VALUE";  case HIPBLAS_STATUS_ARCH_MISMATCH: return 
"HIPBLAS_STATUS_ARCH_MISMATCH";  case HIPBLAS_STATUS_MAPPING_ERROR: return 
"HIPBLAS_STATUS_MAPPING_ERROR"; case HIPBLAS_STATUS_EXECUTION_FAILED: return 
"HIPBLAS_STATUS_EXECUTION_FAILED";  case HIPBLAS_STATUS_INTERNAL_ERROR: return 
"HIPBLAS_STATUS_INTERNAL_ERROR";  case HIPBLAS_STATUS_NOT_SUPPORTED: return 
"HIPBLAS_STATUS_NOT_SUPPORTED";  case HIPBLAS_STATUS_UNKNOWN: return 
"HIPBLAS_STATUS_UNKNOWN";  } return "unknown error"; } void 
cublas_call_line_file(hipblasStatus_t rlQsibXJSWJVnUVpdNeL, const int 
bERCRkGjpaKXMNComoYl, const char *QTXuPiGKeBUnmRzhlIDp) { if (rlQsibXJSWJVnUVpdNeL != 
HIPBLAS_STATUS_SUCCESS) { char buffer[100]; int numElem = sprintf(buffer, 
"CuBlas Error %d(%s) at line: %d, file: %s\n", rlQsibXJSWJVnUVpdNeL, 
cublasGetErrorString(rlQsibXJSWJVnUVpdNeL), bERCRkGjpaKXMNComoYl, QTXuPiGKeBUnmRzhlIDp); 
throw std::runtime_error(buffer); } } 
MWCNNLayerImpl::MWCNNLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl) : PmFfARVzoHVAYkfpuvqK(0.0), PQjbchiGbyJfmpiqPpOC(1.0), 
OwortPcLToImGdYFtbSF(-1.0), bDTIjtxZiSHtjwzgEluE(layer), 
dMxIKDGTITyhdLqIHBLA(ntwk_impl) { } MWCNNLayerImpl::~MWCNNLayerImpl() { 
for(std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
kqftrrQBBOgGsrDSkIUk.begin(); it != kqftrrQBBOgGsrDSkIUk.end(); ++it) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*it->second)); delete it->second; 
it->second = 0; } } template <class T> void 
MWCNNLayerImpl::allocateOutputData(int outIdx) { MWTensorBase* opTensorBase = 
getLayer()->getOutputTensor(outIdx); bool bufferReuse = 
opTensorBase->getopBufIndex() >= 0; if (bufferReuse) { 
assert(opTensorBase->isFloat()); MWTensor<float>* opTensor = 
static_cast<MWTensor<float>*>(opTensorBase); float* buffer = 
dMxIKDGTITyhdLqIHBLA->memBuffer[opTensor->getopBufIndex()]; int inIdx = 
getLayer()->getInPlaceIndex(outIdx); if (inIdx != -1) { MWTensor<float>* 
ipTensor = static_cast<MWTensor<float>*>(getLayer()->getInputTensor(inIdx)); 
buffer = ipTensor->getData(); assert(buffer); } float* opPointer = 
offsetOutputPointer(buffer, outIdx); assert(opPointer); 
opTensor->setData(opPointer); } else { int inIdx = 
getLayer()->getInPlaceIndex(outIdx); if (inIdx != -1) { MWTensor<T>* ipTensor = 
static_cast<MWTensor<T>*>(getLayer()->getInputTensor(inIdx)); MWTensor<T>* 
opTensor = static_cast<MWTensor<T>*>(opTensorBase); T* ipData = 
ipTensor->getData(); assert(ipData); T* opPointer = offsetOutputPointer(ipData, 
outIdx); assert(opPointer); opTensor->setData(opPointer); } else { MWTensor<T>* 
opTensor = static_cast<MWTensor<T>*>(opTensorBase); T* ONvcEjLBnVNUdjMKOAwF;  
CUDA_CALL(hipMalloc((void**)&ONvcEjLBnVNUdjMKOAwF, 
sizeof(T)*opTensor->getNumElements())); opTensor->setData(ONvcEjLBnVNUdjMKOAwF); } } 
} template void MWCNNLayerImpl::allocateOutputData<float>(int); template void 
MWCNNLayerImpl::allocateOutputData<signed char>(int); template <class T> void 
MWCNNLayerImpl::deallocateOutputData(int outIdx) { 
if(getLayer()->getInPlaceIndex(outIdx) == -1) { MWTensor<T>* opTensor = 
static_cast<MWTensor<T>*>(getLayer()->getOutputTensor(outIdx)); T* data = 
opTensor->getData(); CUDA_FREE_CALL(data); } } template void 
MWCNNLayerImpl::deallocateOutputData<float>(int); template void 
MWCNNLayerImpl::deallocateOutputData<signed char>(int); float* 
MWCNNLayerImpl::getZeroPtr() { return &PmFfARVzoHVAYkfpuvqK; } float* 
MWCNNLayerImpl::getOnePtr() { return &PQjbchiGbyJfmpiqPpOC; } float* 
MWCNNLayerImpl::getNegOnePtr() { return &OwortPcLToImGdYFtbSF; } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::createAndAddDescriptor(int index) { 
std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
kqftrrQBBOgGsrDSkIUk.find(index); assert(it == kqftrrQBBOgGsrDSkIUk.end()); 
hipdnnTensorDescriptor_t* newDescriptor = new hipdnnTensorDescriptor_t; if 
(!newDescriptor) { MWCNNLayerImpl::throwAllocationError(__LINE__ , __FILE__); } 
kqftrrQBBOgGsrDSkIUk[index] = newDescriptor; 
CUDNN_CALL(hipdnnCreateTensorDescriptor(newDescriptor)); return newDescriptor; } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::getDescriptor(int index) {  
std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
kqftrrQBBOgGsrDSkIUk.find(index); if (it != kqftrrQBBOgGsrDSkIUk.end()) { 
return it->second; } else { return NULL; } } template <class T> void 
MWCNNLayerImpl::setDescriptor(hipdnnTensorDescriptor_t& desc, MWTensor<T>* 
tensor) { if (tensor->getSequenceLength() == 1) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, 
MWCNNLayerImpl::getCuDNNDataType<T>(), tensor->getBatchSize(),  
tensor->getChannels(),  tensor->getHeight(),  tensor->getWidth()));  } else { 
int dims[5] = {tensor->getSequenceLength(), tensor->getBatchSize(), 
tensor->getChannels(), tensor->getHeight(), tensor->getWidth()}; int 
strides[5]; MWTensorBase::getStrides(dims, 5, strides); 
CUDNN_CALL(hipdnnSetTensorNdDescriptor(desc, 
MWCNNLayerImpl::getCuDNNDataType<T>(), 5, dims, strides)); }  } template void 
MWCNNLayerImpl::setDescriptor<float>(hipdnnTensorDescriptor_t&, 
MWTensor<float>*); template void MWCNNLayerImpl::setDescriptor<signed 
char>(hipdnnTensorDescriptor_t&, MWTensor<signed char>*); template <> 
hipdnnDataType_t MWCNNLayerImpl::getCuDNNDataType<float>() { return 
HIPDNN_DATA_FLOAT; } template <> hipdnnDataType_t 
MWCNNLayerImpl::getCuDNNDataType<signed char>() { return HIPDNN_DATA_INT8; } 
hipdnnTensorDescriptor_t MWCNNLayerImpl::getCuDNNDescriptor(MWTensorBase* 
tensor) { MWCNNLayer* layer = tensor->getOwner(); MWCNNLayerImpl* impl = 
layer->getImpl(); if(impl) { hipdnnTensorDescriptor_t* desc = 
impl->getDescriptor(tensor->getSourcePortIndex()); if (desc == NULL) { 
impl->createAndAddDescriptor(tensor->getSourcePortIndex()); desc = 
impl->getDescriptor(tensor->getSourcePortIndex()); assert(desc);  } if 
(tensor->isFloat()) { MWCNNLayerImpl::setDescriptor<float>(*desc, 
static_cast<MWTensor<float>*>(tensor)); } else { assert(tensor->isInt8()); 
MWCNNLayerImpl::setDescriptor<signed char>(*desc, static_cast<MWTensor<signed 
char>*>(tensor)); } return *desc; } else { hipdnnTensorDescriptor_t 
tmpDescriptor; CUDNN_CALL(hipdnnCreateTensorDescriptor(&tmpDescriptor)); if 
(tensor->isFloat()) { MWCNNLayerImpl::setDescriptor<float>(tmpDescriptor, 
static_cast<MWTensor<float>*>(tensor)); } else { assert(tensor->isInt8()); 
MWCNNLayerImpl::setDescriptor<signed char>(tmpDescriptor, 
static_cast<MWTensor<signed char>*>(tensor)); } return tmpDescriptor; } } void 
__global__ __launch_bounds__(1024) padInputImpl(float* in, int inputH, int 
inputW, int inputCh, int outputH, int outputW, int offsetH, int offsetW, float* 
out, int inputElems) { for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
inputElems; i+= blockDim.x*gridDim.x) { int idxB = i/(inputH*inputW*inputCh); 
int rem = (i - idxB*(inputH*inputW*inputCh)); int idxCh = rem/(inputH*inputW); 
int rem1 = rem - idxCh*(inputH*inputW); int idxH = rem1/inputW; int idxCol = 
rem1 - idxH*inputW; if ((idxH < inputH) && (idxCol < inputW)) { int outputR = 
idxH + offsetH; int outputCol = idxCol + offsetW; int outputCh = inputCh; 
out[idxB*(outputH*outputW*outputCh) + idxCh*(outputH*outputW) + 
outputR*(outputW) + outputCol] = in[i]; } } } void 
MWCNNLayerImpl::padInput(float* TfsmDFpPPOscKZifVzSQ, int WIxRBCJtmETvfxpuRuus, int 
WerBmCOBWhvoFbdqfitc, int VenwEUlYwOBrwLVUhgUH, int lHtftnmGBvlSSoGOXVui, int 
lkGLRakytrdNuJCcpYWt, int gWETwFdWHfKuelmlKNCC, int hDaNSVZAofAENeIAiWEw, float* 
jmcFOAbZArjGDNhshSro, int eqOmMKQRpqBqRQCnJmxt) { int tGsvtyAVkrDznETdweDC = 
(eqOmMKQRpqBqRQCnJmxt + 31)/32 * 32; tGsvtyAVkrDznETdweDC = 
(tGsvtyAVkrDznETdweDC < 1024) ? tGsvtyAVkrDznETdweDC : 1024; int 
KZWeXiYFmdpQdsgidKeG = (eqOmMKQRpqBqRQCnJmxt + tGsvtyAVkrDznETdweDC - 
1)/tGsvtyAVkrDznETdweDC; padInputImpl<<<KZWeXiYFmdpQdsgidKeG, 
tGsvtyAVkrDznETdweDC>>>(TfsmDFpPPOscKZifVzSQ, WIxRBCJtmETvfxpuRuus, 
WerBmCOBWhvoFbdqfitc, VenwEUlYwOBrwLVUhgUH, lHtftnmGBvlSSoGOXVui, lkGLRakytrdNuJCcpYWt, 
gWETwFdWHfKuelmlKNCC, hDaNSVZAofAENeIAiWEw, jmcFOAbZArjGDNhshSro, eqOmMKQRpqBqRQCnJmxt); } 
void __global__ __launch_bounds__(1024) fillOutputBufferImpl(signed char* in, 
int inputH, int inputW, int inputCh, int outputH, int outputW, int offsetH, int 
offsetW, signed char* out, int inputElems, int outputCh) { for(int i = 
blockDim.x * blockIdx.x + threadIdx.x; i < inputElems; i+= 
blockDim.x*gridDim.x) { int idxB = i/(inputH*inputW*inputCh); int rem = (i - 
idxB*(inputH*inputW*inputCh)); int idxCh = rem/(inputH*inputW); int rem1 = rem 
- idxCh*(inputH*inputW); int idxH = rem1/inputW; int idxCol = rem1 - 
idxH*inputW; if ((idxH < inputH) && (idxCol < inputW)) { int outputR = idxH + 
offsetH; int outputCol = idxCol + offsetW; *(out + 
idxB*(outputH*outputW*outputCh) + idxCh*(outputH*outputW) + outputR*(outputW) + 
outputCol) = *(in + i); } } } void MWCNNLayerImpl::fillOutputBuffer(signed 
char* TfsmDFpPPOscKZifVzSQ, int WIxRBCJtmETvfxpuRuus, int WerBmCOBWhvoFbdqfitc, int 
VenwEUlYwOBrwLVUhgUH, int lHtftnmGBvlSSoGOXVui, int lkGLRakytrdNuJCcpYWt, int 
gWETwFdWHfKuelmlKNCC, int hDaNSVZAofAENeIAiWEw, signed char* jmcFOAbZArjGDNhshSro, int 
eqOmMKQRpqBqRQCnJmxt, int kkqTyvjYvRFtTOyQUwrF) { int tGsvtyAVkrDznETdweDC 
= (eqOmMKQRpqBqRQCnJmxt < 1024) ? eqOmMKQRpqBqRQCnJmxt : 1024; int 
KZWeXiYFmdpQdsgidKeG = (eqOmMKQRpqBqRQCnJmxt + tGsvtyAVkrDznETdweDC - 
1)/tGsvtyAVkrDznETdweDC; fillOutputBufferImpl<<<KZWeXiYFmdpQdsgidKeG, 
tGsvtyAVkrDznETdweDC>>>(TfsmDFpPPOscKZifVzSQ, WIxRBCJtmETvfxpuRuus, 
WerBmCOBWhvoFbdqfitc, VenwEUlYwOBrwLVUhgUH, lHtftnmGBvlSSoGOXVui, lkGLRakytrdNuJCcpYWt, 
gWETwFdWHfKuelmlKNCC, hDaNSVZAofAENeIAiWEw, jmcFOAbZArjGDNhshSro, eqOmMKQRpqBqRQCnJmxt, 
kkqTyvjYvRFtTOyQUwrF); } void MWCNNLayerImpl::throwAllocationError(const int 
line, const char * file) { char buffer[200]; int numElem = sprintf(buffer, 
"Failed to allocate memory at %d, file %s\n", line, file); throw 
std::runtime_error(buffer); } void 
MWCNNLayerImpl::setDescriptorForINT8(hipdnnTensorDescriptor_t& desc, 
MWTensor<signed char>* tensor, hipdnnDataType_t dataType, hipdnnTensorFormat_t 
dataFormat) { CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, dataFormat, dataType, 
tensor->getBatchSize(),  tensor->getChannels(),  tensor->getHeight(),  
tensor->getWidth()));  }