//
// File: line_detection_initialize.cu
//
// GPU Coder version                    : 2.1
// CUDA/C/C++ source code generated on  : 11-Nov-2021 11:08:16
//

// Include Files
#include "line_detection_initialize.h"
#include "line_detection.h"
#include "line_detection_data.h"

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void line_detection_initialize()
{
  line_detection_init();
  hipGetLastError();
  isInitialized_line_detection = true;
}

//
// File trailer for line_detection_initialize.cu
//
// [EOF]
//
