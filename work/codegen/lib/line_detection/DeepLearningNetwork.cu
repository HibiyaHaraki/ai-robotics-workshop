//
// File: DeepLearningNetwork.cu
//
// GPU Coder version                    : 2.1
// CUDA/C/C++ source code generated on  : 11-Nov-2021 11:08:16
//

// Include Files
#include "DeepLearningNetwork.h"
#include "line_detection_internal_types.h"
#include "MWAvgPoolingLayer.hpp"
#include "MWCNNLayer.hpp"
#include "MWConvLayer.hpp"
#include "MWElementwiseAffineLayer.hpp"
#include "MWFCLayer.hpp"
#include "MWFusedConvReLULayer.hpp"
#include "MWInputLayer.hpp"
#include "MWMaxPoolingLayer.hpp"
#include "MWOutputLayer.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "MWTensor.hpp"
#include "MWTensorBase.hpp"
#include <cstdio>

// Named Constants
const char *errorString{
    "Abnormal termination due to: %s.\nError in %s (line %d)."};

// Function Declarations
static void checkCleanupCudaError(hipError_t errCode, const char *file,
                                  unsigned int line);

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void mynet_new0_0::allocate()
{
  this->targetImpl->allocate(921600, 3);
  for (int idx{0}; idx < 26; idx++) {
    this->layers[idx]->allocate();
  }
  (static_cast<MWTensor<float> *>(this->inputTensors[0]))
      ->setData(this->layers[0]->getLayerOutput(0));
}

//
// Arguments    : void
// Return Type  : void
//
void mynet_new0_0::cleanup()
{
  this->deallocate();
  for (int idx{0}; idx < 26; idx++) {
    this->layers[idx]->cleanup();
  }
  if (this->targetImpl) {
    this->targetImpl->cleanup();
  }
}

//
// Arguments    : void
// Return Type  : void
//
void mynet_new0_0::deallocate()
{
  this->targetImpl->deallocate();
  for (int idx{0}; idx < 26; idx++) {
    this->layers[idx]->deallocate();
  }
}

//
// Arguments    : void
// Return Type  : void
//
void mynet_new0_0::postsetup()
{
  this->targetImpl->postSetup(this->layers, this->numLayers);
}

//
// Arguments    : void
// Return Type  : void
//
void mynet_new0_0::resetState()
{
}

//
// Arguments    : void
// Return Type  : void
//
void mynet_new0_0::setSize()
{
  for (int idx{0}; idx < 26; idx++) {
    this->layers[idx]->propagateSize();
  }
  this->allocate();
  this->postsetup();
}

//
// Arguments    : void
// Return Type  : void
//
void mynet_new0_0::setup()
{
  if (this->isInitialized) {
    this->resetState();
  } else {
    this->isInitialized = true;
    this->targetImpl->preSetup();
    this->targetImpl->setAutoTune(true);
    (static_cast<MWInputLayer *>(this->layers[0]))
        ->createInputLayer(this->targetImpl, this->inputTensors[0], 180, 320, 3,
                           0, "", 0);
    (static_cast<MWElementwiseAffineLayer *>(this->layers[1]))
        ->createElementwiseAffineLayer(
            this->targetImpl, this->layers[0]->getOutputTensor(0), 1, 1, 3, 1,
            1, 3, false, 1, 1,
            "./codegen/lib/line_detection/cnn_mynet_new0_0_data_scale.bin",
            "./codegen/lib/line_detection/cnn_mynet_new0_0_data_offset.bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[2]))
        ->createFusedConvReLULayer(
            this->targetImpl, 1, this->layers[1]->getOutputTensor(0), 7, 7, 3,
            64, 2, 2, 3, 3, 3, 3, 1, 1, 1,
            "./codegen/lib/line_detection/cnn_mynet_new0_0_conv1_w.bin",
            "./codegen/lib/line_detection/cnn_mynet_new0_0_conv1_b.bin", 1);
    (static_cast<MWMaxPoolingLayer *>(this->layers[3]))
        ->createMaxPoolingLayer<float, float>(
            this->targetImpl, this->layers[2]->getOutputTensor(0), 3, 3, 2, 2,
            1, 1, 1, 1, 0, 0, "FLOAT", 1, 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[4]))
        ->createFusedConvReLULayer(this->targetImpl, 1,
                                   this->layers[3]->getOutputTensor(0), 3, 3,
                                   64, 64, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res2a_branch2a_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res2a_branch2a_b.bin",
                                   1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[5]))
        ->createFusedConvReLULayer(this->targetImpl, 2,
                                   this->layers[4]->getOutputTensor(0),
                                   this->layers[3]->getOutputTensor(0), 3, 3,
                                   64, 64, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res2a_branch2b_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res2a_branch2b_b.bin",
                                   2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[6]))
        ->createFusedConvReLULayer(this->targetImpl, 1,
                                   this->layers[5]->getOutputTensor(0), 3, 3,
                                   64, 64, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res2b_branch2a_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res2b_branch2a_b.bin",
                                   0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[7]))
        ->createFusedConvReLULayer(this->targetImpl, 2,
                                   this->layers[6]->getOutputTensor(0),
                                   this->layers[5]->getOutputTensor(0), 3, 3,
                                   64, 64, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res2b_branch2b_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res2b_branch2b_b.bin",
                                   1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[8]))
        ->createFusedConvReLULayer(this->targetImpl, 1,
                                   this->layers[7]->getOutputTensor(0), 3, 3,
                                   64, 128, 2, 2, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res3a_branch2a_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res3a_branch2a_b.bin",
                                   0);
    (static_cast<MWConvLayer *>(this->layers[9]))
        ->createConvLayer(
            this->targetImpl, this->layers[7]->getOutputTensor(0), 1, 1, 64,
            128, 2, 2, 0, 0, 0, 0, 1, 1, 1,
            "./codegen/lib/line_detection/cnn_mynet_new0_0_res3a_branch1_w.bin",
            "./codegen/lib/line_detection/cnn_mynet_new0_0_res3a_branch1_b.bin",
            2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[10]))
        ->createFusedConvReLULayer(this->targetImpl, 2,
                                   this->layers[8]->getOutputTensor(0),
                                   this->layers[9]->getOutputTensor(0), 3, 3,
                                   128, 128, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res3a_branch2b_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res3a_branch2b_b.bin",
                                   1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[11]))
        ->createFusedConvReLULayer(this->targetImpl, 1,
                                   this->layers[10]->getOutputTensor(0), 3, 3,
                                   128, 128, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res3b_branch2a_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res3b_branch2a_b.bin",
                                   0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[12]))
        ->createFusedConvReLULayer(this->targetImpl, 2,
                                   this->layers[11]->getOutputTensor(0),
                                   this->layers[10]->getOutputTensor(0), 3, 3,
                                   128, 128, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res3b_branch2b_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res3b_branch2b_b.bin",
                                   2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[13]))
        ->createFusedConvReLULayer(this->targetImpl, 1,
                                   this->layers[12]->getOutputTensor(0), 3, 3,
                                   128, 256, 2, 2, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res4a_branch2a_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res4a_branch2a_b.bin",
                                   0);
    (static_cast<MWConvLayer *>(this->layers[14]))
        ->createConvLayer(
            this->targetImpl, this->layers[12]->getOutputTensor(0), 1, 1, 128,
            256, 2, 2, 0, 0, 0, 0, 1, 1, 1,
            "./codegen/lib/line_detection/cnn_mynet_new0_0_res4a_branch1_w.bin",
            "./codegen/lib/line_detection/cnn_mynet_new0_0_res4a_branch1_b.bin",
            1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[15]))
        ->createFusedConvReLULayer(this->targetImpl, 2,
                                   this->layers[13]->getOutputTensor(0),
                                   this->layers[14]->getOutputTensor(0), 3, 3,
                                   256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res4a_branch2b_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res4a_branch2b_b.bin",
                                   2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[16]))
        ->createFusedConvReLULayer(this->targetImpl, 1,
                                   this->layers[15]->getOutputTensor(0), 3, 3,
                                   256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res4b_branch2a_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res4b_branch2a_b.bin",
                                   0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[17]))
        ->createFusedConvReLULayer(this->targetImpl, 2,
                                   this->layers[16]->getOutputTensor(0),
                                   this->layers[15]->getOutputTensor(0), 3, 3,
                                   256, 256, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res4b_branch2b_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res4b_branch2b_b.bin",
                                   1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[18]))
        ->createFusedConvReLULayer(this->targetImpl, 1,
                                   this->layers[17]->getOutputTensor(0), 3, 3,
                                   256, 512, 2, 2, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res5a_branch2a_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res5a_branch2a_b.bin",
                                   0);
    (static_cast<MWConvLayer *>(this->layers[19]))
        ->createConvLayer(
            this->targetImpl, this->layers[17]->getOutputTensor(0), 1, 1, 256,
            512, 2, 2, 0, 0, 0, 0, 1, 1, 1,
            "./codegen/lib/line_detection/cnn_mynet_new0_0_res5a_branch1_w.bin",
            "./codegen/lib/line_detection/cnn_mynet_new0_0_res5a_branch1_b.bin",
            2);
    (static_cast<MWFusedConvReLULayer *>(this->layers[20]))
        ->createFusedConvReLULayer(this->targetImpl, 2,
                                   this->layers[18]->getOutputTensor(0),
                                   this->layers[19]->getOutputTensor(0), 3, 3,
                                   512, 512, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res5a_branch2b_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res5a_branch2b_b.bin",
                                   1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[21]))
        ->createFusedConvReLULayer(this->targetImpl, 1,
                                   this->layers[20]->getOutputTensor(0), 3, 3,
                                   512, 512, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res5b_branch2a_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res5b_branch2a_b.bin",
                                   0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[22]))
        ->createFusedConvReLULayer(this->targetImpl, 2,
                                   this->layers[21]->getOutputTensor(0),
                                   this->layers[20]->getOutputTensor(0), 3, 3,
                                   512, 512, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res5b_branch2b_w.bin",
                                   "./codegen/lib/line_detection/"
                                   "cnn_mynet_new0_0_res5b_branch2b_b.bin",
                                   2);
    (static_cast<MWAvgPoolingLayer *>(this->layers[23]))
        ->createAvgPoolingLayer<float, float>(
            this->targetImpl, this->layers[22]->getOutputTensor(0), -1, -1, 1,
            1, 0, 0, 0, 0, 0, "FLOAT", 1, 0);
    (static_cast<MWFCLayer *>(this->layers[24]))
        ->createFCLayer(
            this->targetImpl, this->layers[23]->getOutputTensor(0), 512, 2,
            "./codegen/lib/line_detection/cnn_mynet_new0_0_new_fc_w.bin",
            "./codegen/lib/line_detection/cnn_mynet_new0_0_new_fc_b.bin", 1);
    (static_cast<MWOutputLayer *>(this->layers[25]))
        ->createOutputLayer(this->targetImpl,
                            this->layers[24]->getOutputTensor(0), 1);
    this->outputTensors[0] = this->layers[25]->getOutputTensor(0);
    this->setSize();
  }
}

//
// Arguments    : hipError_t errCode
//                const char *file
//                unsigned int line
// Return Type  : void
//
static void checkCleanupCudaError(hipError_t errCode, const char *file,
                                  unsigned int line)
{
  if ((errCode != hipSuccess) && (errCode != hipErrorDeinitialized)) {
    printf(errorString, hipGetErrorString(errCode), file, line);
  }
}

//
// Arguments    : void
// Return Type  : int
//
int mynet_new0_0::getBatchSize()
{
  return this->inputTensors[0]->getBatchSize();
}

//
// Arguments    : int b_index
// Return Type  : float *
//
float *mynet_new0_0::getInputDataPointer(int b_index)
{
  return (static_cast<MWTensor<float> *>(this->inputTensors[b_index]))
      ->getData();
}

//
// Arguments    : void
// Return Type  : float *
//
float *mynet_new0_0::getInputDataPointer()
{
  return (static_cast<MWTensor<float> *>(this->inputTensors[0]))->getData();
}

//
// Arguments    : int layerIndex
//                int portIndex
// Return Type  : float *
//
float *mynet_new0_0::getLayerOutput(int layerIndex, int portIndex)
{
  return this->layers[layerIndex]->getLayerOutput(portIndex);
}

//
// Arguments    : int layerIndex
//                int portIndex
// Return Type  : int
//
int mynet_new0_0::getLayerOutputSize(int layerIndex, int portIndex)
{
  return this->layers[layerIndex]
             ->getOutputTensor(portIndex)
             ->getNumElements() *
         sizeof(float);
}

//
// Arguments    : int b_index
// Return Type  : float *
//
float *mynet_new0_0::getOutputDataPointer(int b_index)
{
  return (static_cast<MWTensor<float> *>(this->outputTensors[b_index]))
      ->getData();
}

//
// Arguments    : void
// Return Type  : float *
//
float *mynet_new0_0::getOutputDataPointer()
{
  return (static_cast<MWTensor<float> *>(this->outputTensors[0]))->getData();
}

//
// Arguments    : void
// Return Type  : ::mynet_new0_0
//
mynet_new0_0::mynet_new0_0()
{
  this->numLayers = 26;
  this->isInitialized = false;
  this->targetImpl = 0;
  this->layers[0] = new MWInputLayer;
  this->layers[0]->setName("data");
  this->layers[1] = new MWElementwiseAffineLayer;
  this->layers[1]->setName("data_normalization");
  this->layers[1]->setInPlaceIndex(0, 0);
  this->layers[2] = new MWFusedConvReLULayer;
  this->layers[2]->setName("conv1_conv1_relu");
  this->layers[3] = new MWMaxPoolingLayer;
  this->layers[3]->setName("pool1");
  this->layers[4] = new MWFusedConvReLULayer;
  this->layers[4]->setName("res2a_branch2a_res2a_branch2a_relu");
  this->layers[5] = new MWFusedConvReLULayer;
  this->layers[5]->setName("res2a_branch2b_res2a_relu");
  this->layers[6] = new MWFusedConvReLULayer;
  this->layers[6]->setName("res2b_branch2a_res2b_branch2a_relu");
  this->layers[7] = new MWFusedConvReLULayer;
  this->layers[7]->setName("res2b_branch2b_res2b_relu");
  this->layers[8] = new MWFusedConvReLULayer;
  this->layers[8]->setName("res3a_branch2a_res3a_branch2a_relu");
  this->layers[9] = new MWConvLayer;
  this->layers[9]->setName("res3a_branch1");
  this->layers[10] = new MWFusedConvReLULayer;
  this->layers[10]->setName("res3a_branch2b_res3a_relu");
  this->layers[11] = new MWFusedConvReLULayer;
  this->layers[11]->setName("res3b_branch2a_res3b_branch2a_relu");
  this->layers[12] = new MWFusedConvReLULayer;
  this->layers[12]->setName("res3b_branch2b_res3b_relu");
  this->layers[13] = new MWFusedConvReLULayer;
  this->layers[13]->setName("res4a_branch2a_res4a_branch2a_relu");
  this->layers[14] = new MWConvLayer;
  this->layers[14]->setName("res4a_branch1");
  this->layers[15] = new MWFusedConvReLULayer;
  this->layers[15]->setName("res4a_branch2b_res4a_relu");
  this->layers[16] = new MWFusedConvReLULayer;
  this->layers[16]->setName("res4b_branch2a_res4b_branch2a_relu");
  this->layers[17] = new MWFusedConvReLULayer;
  this->layers[17]->setName("res4b_branch2b_res4b_relu");
  this->layers[18] = new MWFusedConvReLULayer;
  this->layers[18]->setName("res5a_branch2a_res5a_branch2a_relu");
  this->layers[19] = new MWConvLayer;
  this->layers[19]->setName("res5a_branch1");
  this->layers[20] = new MWFusedConvReLULayer;
  this->layers[20]->setName("res5a_branch2b_res5a_relu");
  this->layers[21] = new MWFusedConvReLULayer;
  this->layers[21]->setName("res5b_branch2a_res5b_branch2a_relu");
  this->layers[22] = new MWFusedConvReLULayer;
  this->layers[22]->setName("res5b_branch2b_res5b_relu");
  this->layers[23] = new MWAvgPoolingLayer;
  this->layers[23]->setName("pool5");
  this->layers[24] = new MWFCLayer;
  this->layers[24]->setName("new_fc");
  this->layers[25] = new MWOutputLayer;
  this->layers[25]->setName("regressionoutput");
  this->layers[25]->setInPlaceIndex(0, 0);
  this->targetImpl = new MWTargetNetworkImpl;
  this->inputTensors[0] = new MWTensor<float>;
  this->inputTensors[0]->setHeight(180);
  this->inputTensors[0]->setWidth(320);
  this->inputTensors[0]->setChannels(3);
  this->inputTensors[0]->setBatchSize(1);
  this->inputTensors[0]->setSequenceLength(1);
}

//
// Arguments    : void
// Return Type  : void
//
mynet_new0_0::~mynet_new0_0()
{
  this->cleanup();
  checkCleanupCudaError(hipGetLastError(), __FILE__, __LINE__);
  for (int idx{0}; idx < 26; idx++) {
    delete this->layers[idx];
  }
  if (this->targetImpl) {
    delete this->targetImpl;
  }
  delete this->inputTensors[0];
}

//
// Arguments    : void
// Return Type  : void
//
void mynet_new0_0::predict()
{
  for (int idx{0}; idx < 26; idx++) {
    this->layers[idx]->predict();
  }
}

//
// Arguments    : mynet_new0_0 *obj
// Return Type  : void
//
namespace coder {
void DeepLearningNetwork_setup(mynet_new0_0 *obj)
{
  obj->setup();
}

} // namespace coder

//
// File trailer for DeepLearningNetwork.cu
//
// [EOF]
//
